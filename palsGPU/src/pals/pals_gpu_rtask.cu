#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <limits.h>
#include <assert.h>

#include "../config.h"
#include "../utils.h"
#include "../random/RNG_rand48.h"

#include "pals_gpu_rtask.h"

#define PALS_GPU_RTASK__BLOCKS 			128
#define PALS_GPU_RTASK__THREADS 		128

__global__ void pals_rtask_kernel(
	int machines_count, int tasks_count, float current_makespan,
	float *gpu_etc_matrix, int *gpu_task_assignment, float *gpu_machine_compute_time, 
	int *gpu_random_numbers, ushort *gpu_best_swaps, float *gpu_best_swaps_delta)
{
	unsigned int thread_idx = threadIdx.x;
	unsigned int block_idx = blockIdx.x;
	
	//unsigned int mov_type = block_idx & 0x1;
	unsigned int mov_type = 0;

	__shared__ ushort block_swaps[PALS_GPU_RTASK__THREADS];
	__shared__ float block_swaps_delta[PALS_GPU_RTASK__THREADS];

	__shared__ int random1, random2;
	
	if (threadIdx.x == 0) {
		random1 = gpu_random_numbers[block_idx];
		random2 = gpu_random_numbers[block_idx + 1];
	}
	
	__syncthreads();
							
	// Tipo de movimiento.	
	if (mov_type == 0) { // Comparación a nivel de bit para saber si es par o impar.
		// Si es impar... 
		// Movimiento SWAP.
		
		int task_x, task_y;
		int machine_a, machine_b;
		
		float machine_a_ct_old, machine_b_ct_old;
		float machine_a_ct_new, machine_b_ct_new;
		
		float eval;
		eval = 0.0;
		
		// ================= Obtengo las tareas sorteadas.
		// TODO: OPTIMIZAR MODULOS!!!
		task_x = random1 % tasks_count;
				
		task_y = (random2 % (tasks_count - 1 - PALS_GPU_RTASK__THREADS)) + thread_idx;	
		
		if (task_y >= task_x) task_y++;
		task_y = task_y % tasks_count;
		
		// ================= Obtengo las máquinas a las que estan asignadas las tareas.
		machine_a = gpu_task_assignment[task_x]; // Máquina a.	
		machine_b = gpu_task_assignment[task_y]; // Máquina b.	

		if (machine_a != machine_b) {
			// Calculo el delta del swap sorteado.
			
			// Máquina 1.
			machine_a_ct_old = gpu_machine_compute_time[task_x];
					
			machine_a_ct_new = machine_a_ct_old - gpu_etc_matrix[(machine_a * tasks_count) + task_x]; // Resto del ETC de x en a.
			machine_a_ct_new = machine_a_ct_new + gpu_etc_matrix[(machine_a * tasks_count) + task_y]; // Sumo el ETC de y en a.
			
			// Máquina 2.
			machine_b_ct_old = gpu_machine_compute_time[task_y];

			machine_b_ct_new = machine_b_ct_old - gpu_etc_matrix[(machine_b * tasks_count) + task_y]; // Resto el ETC de y en b.
			machine_b_ct_new = machine_b_ct_new + gpu_etc_matrix[(machine_b * tasks_count) + task_x]; // Sumo el ETC de x en b.

			if ((machine_a_ct_new > current_makespan) || (machine_b_ct_new > current_makespan)) {
				if (machine_a_ct_new > current_makespan) eval = eval + (machine_a_ct_new - current_makespan);
				if (machine_b_ct_new > current_makespan) eval = eval + (machine_b_ct_new - current_makespan);
			} else {
				eval = eval + (machine_a_ct_new - current_makespan);
				eval = eval + (machine_b_ct_new - current_makespan);
			}
		}

		block_swaps[thread_idx] = (ushort)((PALS_GPU_RTASK_SWAP * PALS_GPU_RTASK__THREADS) + thread_idx);
		block_swaps_delta[thread_idx] = eval;
	} else {
		// Si es par...
		// Movimiento MOVE.
		
		// ================= Obtengo la tarea sorteada, la máquina a la que esta asignada,
		// ================= y el compute time de la máquina.
		/*raux1 = raux1 % tasks_count;
		int_aux1 = gpu_task_assignment[raux1]; // Máquina a.
		float_aux1 = gpu_machine_compute_time[int_aux1];	
							
		// ================= Obtengo la máquina destino sorteada.
		raux2 = raux2 % (machines_count - 1 - PALS_GPU_RTASK__THREADS);
		raux2 = raux2 + thread_idx;	
		if (raux2 >= int_aux1) raux2 = raux2 + 1;

		int_aux2 = raux2 % machines_count;
		float_aux2 = gpu_machine_compute_time[int_aux2];
		
		// Calculo el delta del swap sorteado.
		delta = float_aux1 - gpu_etc_matrix[(int_aux1 * tasks_count) + raux1]; // Resto del ETC de x en a.
		
		// Obtengo la diferencia en la máquina A.
		if (delta > current_makespan) {
			float_aux1 = delta - current_makespan;
		} else if ((float_aux1 + 1 >= current_makespan) && (delta < current_makespan)) { // sumo 1 por problemas de redondeo... funciona? 
			float_aux1 = delta - current_makespan;
		} else {
			float_aux1 = 0.0;
		}
			
		delta = float_aux2 + gpu_etc_matrix[(int_aux2 * tasks_count) + raux1]; // Sumo el ETC de x en b.

		// Obtengo la diferencia en la máquina B.
		if (delta > current_makespan) {
			float_aux2 = delta - current_makespan;
		} else if ((float_aux2 + 1 >= current_makespan) && (delta < current_makespan)) { // sumo 1 por problemas de redondeo... funciona? 
			float_aux2 = delta - current_makespan;	
		} else {
			float_aux2 = 0.0;
		}

		// Calculo la mejora de ambos movimientos combinados.
		if ((float_aux1 != 0.0) && (float_aux2 != 0.0)) {
			if (float_aux1 > float_aux2) {
				delta = float_aux1;
			} else {
				delta = float_aux2;
			}
		} else if ((float_aux1 != 0.0) && (float_aux2 == 0.0)) {
			delta = float_aux1;
		} else if ((float_aux1 == 0.0) && (float_aux2 != 0.0)) {
			delta = float_aux2;
		} else {
			delta = 0.0;
		}

		block_swaps[thread_idx] = (ushort)((PALS_GPU_RTASK_MOVE * PALS_GPU_RTASK__THREADS) + thread_idx);
		block_swaps_delta[thread_idx] = delta;*/
	}
	
	__syncthreads();

	// Aplico reduce para quedarme con el mejor delta.
	for (int i = 1; i < PALS_GPU_RTASK__THREADS; i *= 2) {
		int pos;
		pos = 2 * i * thread_idx;
	
		if (pos < PALS_GPU_RTASK__THREADS) {
			if (block_swaps_delta[pos] > block_swaps_delta[pos + i]) {
				block_swaps_delta[pos] = block_swaps_delta[pos + i];
				block_swaps[pos] = block_swaps[pos + i];
			}
		}
	
		__syncthreads();
	}
	
	if (thread_idx == 0) {
		gpu_best_swaps[block_idx] = block_swaps[0]; //best_swap;
		gpu_best_swaps_delta[block_idx] = block_swaps_delta[0]; //best_swap_delta;
	}
}

void pals_gpu_rtask_init(struct matrix *etc_matrix, struct solution *s, 
	struct pals_gpu_rtask_instance &instance, struct pals_gpu_rtask_result &result) {
	
	// Asignación del paralelismo del algoritmo.
	instance.number_of_blocks = PALS_GPU_RTASK__BLOCKS;
	instance.threads_per_block = PALS_GPU_RTASK__THREADS;
	
	// Cantidad total de movimientos a evaluar.
	instance.total_tasks = PALS_GPU_RTASK__BLOCKS * PALS_GPU_RTASK__THREADS;
	
	if (DEBUG) {
		fprintf(stdout, "[INFO] Number of blocks (grid size)   : %d\n", instance.number_of_blocks);
		fprintf(stdout, "[INFO] Threads per block (block size) : %d\n", instance.threads_per_block);	
		fprintf(stdout, "[INFO] Total tasks                    : %d\n", instance.total_tasks);
	}

	// =========================================================================

	// Pedido de memoria en el dispositivo y copiado de datos.
	timespec ts_1;
	timming_start(ts_1);
	
	// Pido memoria para guardar el resultado.
	int best_swaps_size = sizeof(ushort) * instance.number_of_blocks;	
	if (hipMalloc((void**)&(instance.gpu_best_swaps), best_swaps_size) != hipSuccess) {
		fprintf(stderr, "[ERROR] Solicitando memoria gpu_best_swaps (%d bytes).\n", best_swaps_size);
		exit(EXIT_FAILURE);
	}
		
	int best_swaps_delta_size = sizeof(float) * instance.number_of_blocks;	
	if (hipMalloc((void**)&(instance.gpu_best_swaps_delta), best_swaps_delta_size) != hipSuccess) {
		fprintf(stderr, "[ERROR] Solicitando memoria gpu_best_swaps_delta (%d bytes).\n", best_swaps_delta_size);
		exit(EXIT_FAILURE);
	}
	
	timming_end(".. gpu_best_swaps", ts_1);
		
	// =========================================================================
		
	timespec ts_2;
	timming_start(ts_2);
	
	// Copio la matriz de ETC.
	int etc_matrix_size = sizeof(float) * etc_matrix->tasks_count * etc_matrix->machines_count;
	if (hipMalloc((void**)&(instance.gpu_etc_matrix), etc_matrix_size) != hipSuccess) {
		fprintf(stderr, "[ERROR] Solicitando memoria etc_matrix (%d bytes).\n", etc_matrix_size);
		exit(EXIT_FAILURE);
	}
	
	if (hipMemcpy(instance.gpu_etc_matrix, etc_matrix->data, etc_matrix_size, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "[ERROR] Copiando etc_matrix al dispositivo (%d bytes).\n", etc_matrix_size);
		exit(EXIT_FAILURE);
	}

	timming_end(".. gpu_etc_matrix", ts_2);

	// =========================================================================
	
	timespec ts_3;
	timming_start(ts_3);
		
	// Copio la asignación de tareas a máquinas actuales.
	int task_assignment_size = sizeof(int) * etc_matrix->tasks_count;	
	if (hipMalloc((void**)&(instance.gpu_task_assignment), task_assignment_size) != hipSuccess) {
		fprintf(stderr, "[ERROR] Solicitando memoria task_assignment (%d bytes).\n", task_assignment_size);
		exit(EXIT_FAILURE);
	}
	
	if (hipMemcpy(instance.gpu_task_assignment, s->task_assignment, task_assignment_size, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "[ERROR] Copiando task_assignment al dispositivo (%d bytes).\n", task_assignment_size);
		exit(EXIT_FAILURE);
	}

	timming_end(".. gpu_task_assignment", ts_3);

	// =========================================================================
	
	timespec ts_4;
	timming_start(ts_4);
		
	// Copio el compute time de las máquinas en la solución actual.
	int machine_compute_time_size = sizeof(float) * etc_matrix->machines_count;	
	if (hipMalloc((void**)&(instance.gpu_machine_compute_time), machine_compute_time_size) != hipSuccess) {
		fprintf(stderr, "[ERROR] Solicitando memoria machine_compute_time (%d bytes).\n", machine_compute_time_size);
		exit(EXIT_FAILURE);
	}
	
	if (hipMemcpy(instance.gpu_machine_compute_time, s->machine_compute_time, machine_compute_time_size, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "[ERROR] Copiando machine_compute_time al dispositivo (%d bytes).\n", machine_compute_time_size);
		exit(EXIT_FAILURE);
	}

	timming_end(".. gpu_machine_compute_time", ts_4);
	
	// =========================================================================
	
	if (instance.result_count > instance.number_of_blocks) instance.result_count = instance.number_of_blocks;
	
	result.move_count = instance.result_count;
	result.move_type = (char*)malloc(sizeof(char) * instance.result_count);
	result.origin = (int*)malloc(sizeof(int) * instance.result_count);
	result.destination = (int*)malloc(sizeof(int) * instance.result_count);
	result.delta = (float*)malloc(sizeof(float) * instance.result_count);
}

void pals_gpu_rtask_finalize(struct pals_gpu_rtask_instance &instance) {
	if (hipFree(instance.gpu_etc_matrix) != hipSuccess) {
		fprintf(stderr, "[ERROR] Liberando la memoria solicitada para etc_matrix.\n");
		exit(EXIT_FAILURE);
	}
	
	if (hipFree(instance.gpu_task_assignment) != hipSuccess) {
		fprintf(stderr, "[ERROR] Liberando la memoria solicitada para task_assignment.\n");
		exit(EXIT_FAILURE);
	}

	if (hipFree(instance.gpu_machine_compute_time) != hipSuccess) {
		fprintf(stderr, "[ERROR] Liberando la memoria solicitada para machine_compute_time.\n");
		exit(EXIT_FAILURE);
	}
	
	if (hipFree(instance.gpu_best_swaps) != hipSuccess) {
		fprintf(stderr, "[ERROR] Liberando la memoria solicitada para best_swaps.\n");
		exit(EXIT_FAILURE);
	}
}

void pals_gpu_rtask_clean_result(struct pals_gpu_rtask_result &result) {
	free(result.move_type);
	free(result.origin);
	free(result.destination);
	free(result.delta);
}

void pals_gpu_rtask_wrapper(struct matrix *etc_matrix, struct solution *s, 
	struct pals_gpu_rtask_instance &instance, int *gpu_random_numbers, 
	struct pals_gpu_rtask_result &result) {

	// Timming -----------------------------------------------------
	timespec ts_pals_pre;
	timming_start(ts_pals_pre);
	// Timming -----------------------------------------------------
	
	// Timming -----------------------------------------------------
	timming_end(".. pals_gpu_rtask_pals_pre", ts_pals_pre);
	// Timming -----------------------------------------------------
	
	// ==============================================================================
	// Ejecución del algoritmo.
	// ==============================================================================	
	
	// Timming -----------------------------------------------------
	timespec ts_pals;
	timming_start(ts_pals);
	// Timming -----------------------------------------------------
	
	dim3 grid(instance.number_of_blocks, 1, 1);
	dim3 threads(instance.threads_per_block, 1, 1);

	pals_rtask_kernel<<< grid, threads >>>(
		etc_matrix->machines_count,
		etc_matrix->tasks_count,
		s->makespan,
		instance.gpu_etc_matrix, 
		instance.gpu_task_assignment, 
		instance.gpu_machine_compute_time, 
		gpu_random_numbers,
		instance.gpu_best_swaps, 
		instance.gpu_best_swaps_delta);

	// Pido el espacio de memoria para obtener los resultados desde la gpu.
	ushort *best_swaps = (ushort*)malloc(sizeof(ushort) * instance.number_of_blocks);
	float *best_swaps_delta = (float*)malloc(sizeof(float) * instance.number_of_blocks);
	int *rands_nums = (int*)malloc(sizeof(int) * instance.number_of_blocks * 2);

	// Copio los mejores movimientos desde el dispositivo.
	if (hipMemcpyAsync(best_swaps, instance.gpu_best_swaps, 
		sizeof(ushort) * instance.number_of_blocks, 
		hipMemcpyDeviceToHost, 0) != hipSuccess) {
		
		fprintf(stderr, "[ERROR] Copiando los mejores movimientos al host (best_swaps).\n");
		exit(EXIT_FAILURE);
	}
	
	if (hipMemcpyAsync(best_swaps_delta, instance.gpu_best_swaps_delta, 
		sizeof(float) * instance.number_of_blocks, 
		hipMemcpyDeviceToHost, 0) != hipSuccess) {
		
		fprintf(stderr, "[ERROR] Copiando los mejores movimientos al host (best_swaps_delta).\n");
		exit(EXIT_FAILURE);
	}

	if (hipMemcpyAsync(rands_nums, gpu_random_numbers, 
		sizeof(int) * instance.number_of_blocks * 2, 
		hipMemcpyDeviceToHost, 0) != hipSuccess) {
		
		fprintf(stderr, "[ERROR] Copiando al host los números aleatorios sorteados.\n");
		exit(EXIT_FAILURE);
	}

	hipDeviceSynchronize();

	// Timming -----------------------------------------------------
	timming_end(".. pals_gpu_rtask_pals", ts_pals);
	// Timming -----------------------------------------------------

	// =====================================================================
	// Se cargan los resultados a la respuesta.
	// (lo mejor sería usar la GPU para generar el resultado).
	// =====================================================================

	// Timming -----------------------------------------------------
	timespec ts_pals_post;
	timming_start(ts_pals_post);
	// Timming -----------------------------------------------------
	
	// Busco el block que encontró el mejor movimiento.
	int best_block_idx = 0;
	for (int i = 1; i < instance.number_of_blocks; i++) {
		if (best_swaps_delta[i] < best_swaps_delta[best_block_idx]) {
			best_block_idx = i;
		}
	}
	
	int block_idx = (best_block_idx);

	// Calculo cuales fueron los elementos modificados en ese mejor movimiento.	
	int swap = best_swaps[block_idx];

	// TODO: OPTIMIZAR!!! (pasar todo como struct? intentar meter todo dentro de un mismo array?)
	int move_type = swap / PALS_GPU_RTASK__THREADS;
	int thread_idx = swap % PALS_GPU_RTASK__THREADS;

	if (move_type == PALS_GPU_RTASK_SWAP) { // Movement type: SWAP
		int task_x = rands_nums[block_idx] % etc_matrix->tasks_count;

		int random_2 = rands_nums[block_idx + 1];
		int task_y = random_2 % (etc_matrix->tasks_count - 1 - PALS_GPU_RTASK__THREADS);
		task_y = task_y + thread_idx;

		if (task_y >= task_x) task_y = task_y + 1;
		if (task_y >= etc_matrix->tasks_count) task_y = task_y % etc_matrix->tasks_count;

		result.move_type[0] = move_type; // SWAP
		result.origin[0] = task_x;
		result.destination[0] = task_y;
		result.delta[0] = best_swaps_delta[block_idx];
		
		// =======> DEBUG
		if (DEBUG) { 
			int machine_a = s->task_assignment[task_x];
			int machine_b = s->task_assignment[task_y];

			fprintf(stdout, "[DEBUG] Task %d in %d swaps with task %d in %d. Delta %f.\n",
				task_x, machine_a, task_y, machine_b, best_swaps_delta[block_idx]);
		}
		// <======= DEBUG
	} else if (move_type == PALS_GPU_RTASK_MOVE) { // Movement type: MOVE
		int random_1 = rands_nums[block_idx] % etc_matrix->tasks_count;
		int task_x = random_1;
		int machine_a = s->task_assignment[task_x];

		int random_2 = rands_nums[block_idx + 1];
		int machine_b = (random_2 % (etc_matrix->machines_count - 1 - PALS_GPU_RTASK__THREADS)) + thread_idx;
		
		if (machine_b >= machine_a) machine_b = machine_b + 1;
		if (machine_b >= etc_matrix->machines_count) machine_b = machine_b % etc_matrix->machines_count;

		result.move_type[0] = move_type; // MOVE
		result.origin[0] = task_x;
		result.destination[0] = machine_b;
		result.delta[0] = best_swaps_delta[block_idx];
		
		// =======> DEBUG
		if (DEBUG) {
			fprintf(stdout, "[DEBUG] Task %d in %d is moved to machine %d. Delta %f.\n",
				task_x, machine_a, machine_b, best_swaps_delta[block_idx]);
		}
		// <======= DEBUG
	}
	
	// Timming -----------------------------------------------------
	timming_end(".. pals_gpu_rtask_pals_post", ts_pals_post);
	// Timming -----------------------------------------------------
}

void pals_gpu_rtask_move(struct pals_gpu_rtask_instance &instance, int task, int to_machine) {
	if (hipMemcpy(&(instance.gpu_task_assignment[task]), &to_machine, sizeof(int), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "[ERROR] Error moviendo la task %d a la máquina %d.\n", task, to_machine);
		exit(EXIT_FAILURE);
	}
}

void pals_gpu_rtask_update_machine(struct pals_gpu_rtask_instance &instance, int machine, float compute_time) {
	if (hipMemcpy(&(instance.gpu_machine_compute_time[machine]), &compute_time, sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "[ERROR] Error actualizando el compute time de la máquina %d.\n", machine);
		exit(EXIT_FAILURE);
	}
}
