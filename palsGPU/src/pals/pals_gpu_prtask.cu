#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <limits.h>
#include <assert.h>

#include "../config.h"
#include "../utils.h"

#include "../random/cpu_rand.h"
#include "../random/RNG_rand48.h"

#include "pals_gpu_prtask.h"

#define PALS_PRTASK_RANDS 6144*20
#define PALS_PRTASK_RESULT_COUNT 1

#define PALS_GPU_PRTASK__BLOCKS 		6
#define PALS_GPU_PRTASK__THREADS 		96
#define PALS_GPU_PRTASK__LOOPS	 		1

__global__ void pals_prtask_kernel(int machines_count, int tasks_count, float *gpu_etc_matrix, 
	int *gpu_task_assignment, float *gpu_machine_compute_time, int *gpu_random_numbers) {
	
	unsigned int thread_idx = threadIdx.x;
	unsigned int block_idx = blockIdx.x;

	__shared__ unsigned short block_op[PALS_GPU_PRTASK__THREADS];
	__shared__ unsigned short block_task_x[PALS_GPU_PRTASK__THREADS];
	__shared__ unsigned short block_task_y[PALS_GPU_PRTASK__THREADS];
	__shared__ unsigned short block_machine_a[PALS_GPU_PRTASK__THREADS];
	__shared__ unsigned short block_machine_b[PALS_GPU_PRTASK__THREADS];
	__shared__ float block_machine_a_ct_new[PALS_GPU_PRTASK__THREADS];
	__shared__ float block_machine_b_ct_new[PALS_GPU_PRTASK__THREADS];
	__shared__ float block_delta[PALS_GPU_PRTASK__THREADS];
	
	unsigned int machine_compute_time_offset = block_idx * machines_count;
	unsigned int task_assignment_offset = block_idx * tasks_count;
	
	for (short loop = 0; loop < PALS_GPU_PRTASK__LOOPS; loop++) {
		int random1, random2;

		random1 = gpu_random_numbers[(block_idx * PALS_GPU_PRTASK__LOOPS * 2) + (loop * 2)];
		random2 = gpu_random_numbers[(block_idx * PALS_GPU_PRTASK__LOOPS * 2) + (loop * 2) + 1];

		short mov_type = (short)((random1 & 0x1) ^ (random2 & 0x1));
	
		// Tipo de movimiento.
		if (mov_type == 0) { // Comparación a nivel de bit para saber si es par o impar.
			// Si es impar... 
			// Movimiento SWAP.
		
			int task_x, task_y;
			int machine_a, machine_b;
		
			float machine_a_ct_old, machine_b_ct_old;
			float machine_a_ct_new, machine_b_ct_new;
		
			float delta;
			delta = 0.0;
		
			// ================= Obtengo las tareas sorteadas.
			task_x = random1 % tasks_count;
				
			task_y = ((random2 >> 1) + thread_idx) % (tasks_count - 1);	
			if (task_y >= task_x) task_y++;
		
			// ================= Obtengo las máquinas a las que estan asignadas las tareas.
			machine_a = gpu_task_assignment[task_assignment_offset + task_x]; // Máquina a.	
			machine_b = gpu_task_assignment[task_assignment_offset + task_y]; // Máquina b.	

			if (machine_a != machine_b) {
				// Calculo el delta del swap sorteado.
			
				// Máquina 1.
				machine_a_ct_old = gpu_machine_compute_time[machine_compute_time_offset + machine_a];
					
				machine_a_ct_new = machine_a_ct_old;
				machine_a_ct_new = machine_a_ct_new - gpu_etc_matrix[(machine_a * tasks_count) + task_x]; // Resto del ETC de x en a.
				machine_a_ct_new = machine_a_ct_new + gpu_etc_matrix[(machine_a * tasks_count) + task_y]; // Sumo el ETC de y en a.
			
				// Máquina 2.
				machine_b_ct_old = gpu_machine_compute_time[machine_compute_time_offset + machine_b];

				machine_b_ct_new = machine_b_ct_old;
				machine_b_ct_new = machine_b_ct_new - gpu_etc_matrix[(machine_b * tasks_count) + task_y]; // Resto el ETC de y en b.
				machine_b_ct_new = machine_b_ct_new + gpu_etc_matrix[(machine_b * tasks_count) + task_x]; // Sumo el ETC de x en b.

				if (machine_b_ct_new > machine_a_ct_new) {
					delta = machine_b_ct_new;
				} else {
					delta = machine_a_ct_new;
				}

				if (machine_b_ct_old > machine_a_ct_old) {
					delta = delta - machine_b_ct_old;
				} else {
					delta = delta - machine_a_ct_old;
				}

				/*
				if ((machine_a_ct_new > current_makespan) || (machine_b_ct_new > current_makespan)) {
					// Luego del movimiento aumenta el makespan. Intento desestimularlo lo más posible.
					if (machine_a_ct_new > current_makespan) delta = delta + (machine_a_ct_new - current_makespan);
					if (machine_b_ct_new > current_makespan) delta = delta + (machine_b_ct_new - current_makespan);
				} else if ((machine_a_ct_old+1 >= current_makespan) || (machine_b_ct_old+1 >= current_makespan)) {	
					// Antes del movimiento una las de máquinas definía el makespan. Estos son los mejores movimientos.
				
					if (machine_a_ct_old+1 >= current_makespan) {
						delta = delta + (machine_a_ct_new - machine_a_ct_old);
					} else {
						delta = delta + 1/(machine_a_ct_new - machine_a_ct_old);
					}
				
					if (machine_b_ct_old+1 >= current_makespan) {
						delta = delta + (machine_b_ct_new - machine_b_ct_old);
					} else {
						delta = delta + 1/(machine_b_ct_new - machine_b_ct_old);
					}
				} else {
					// Ninguna de las máquinas intervenía en el makespan. Intento favorecer lo otros movimientos.
					delta = delta + (machine_a_ct_new - machine_a_ct_old);
					delta = delta + (machine_b_ct_new - machine_b_ct_old);
					delta = 1 / delta;
				}
				*/
			}

			block_op[thread_idx] = (short)PALS_GPU_PRTASK_SWAP;
			block_task_x[thread_idx] = (unsigned short)task_x;
			block_task_y[thread_idx] = (unsigned short)task_y;
			block_machine_a[thread_idx] = (unsigned short)machine_a;
			block_machine_b[thread_idx] = (unsigned short)machine_b;
			block_machine_a_ct_new[thread_idx] = machine_a_ct_new;
			block_machine_b_ct_new[thread_idx] = machine_b_ct_new;			
			block_delta[thread_idx] = delta;
		} else {
			// Si es par...
			// Movimiento MOVE.
		
			int task_x;
			int machine_a, machine_b;
		
			float machine_a_ct_old, machine_b_ct_old;
			float machine_a_ct_new, machine_b_ct_new;

			float delta;
			delta = 0.0;
		
			// ================= Obtengo la tarea sorteada, la máquina a la que esta asignada,
			// ================= y el compute time de la máquina.
			task_x = random1 % tasks_count;
			machine_a = gpu_task_assignment[task_assignment_offset + task_x]; // Máquina a.
			
			machine_a_ct_old = gpu_machine_compute_time[machine_compute_time_offset + machine_a];	
							
			// ================= Obtengo la máquina destino sorteada.
			machine_b = ((random2 >> 1) + thread_idx) % (machines_count - 1);
			if (machine_b >= machine_a) machine_b++;
		
			machine_b_ct_old = gpu_machine_compute_time[machine_compute_time_offset + machine_b];
		
			// Calculo el delta del swap sorteado.
			machine_a_ct_new = machine_a_ct_old - gpu_etc_matrix[(machine_a * tasks_count) + task_x]; // Resto del ETC de x en a.		
			machine_b_ct_new = machine_b_ct_old + gpu_etc_matrix[(machine_b * tasks_count) + task_x]; // Sumo el ETC de x en b.

			if (machine_b_ct_new > machine_a_ct_new) {
				delta = machine_b_ct_new;
			} else {
				delta = machine_a_ct_new;
			}

			if (machine_b_ct_old > machine_a_ct_old) {
				delta = delta - machine_b_ct_old;
			} else {
				delta = delta - machine_a_ct_old;
			}

			/*
			if (machine_b_ct_new > current_makespan) {
				// Luego del movimiento aumenta el makespan. Intento desestimularlo lo más posible.
				delta = delta + (machine_b_ct_new - current_makespan);
			} else if (machine_a_ct_old+1 >= current_makespan) {	
				// Antes del movimiento una las de máquinas definía el makespan. Estos son los mejores movimientos.
				delta = delta + (machine_a_ct_new - machine_a_ct_old);
				delta = delta + 1/(machine_b_ct_new - machine_b_ct_old);
			} else {
				// Ninguna de las máquinas intervenía en el makespan. Intento favorecer lo otros movimientos.
				delta = delta + (machine_a_ct_new - machine_a_ct_old);
				delta = delta + (machine_b_ct_new - machine_b_ct_old);
				delta = 1 / delta;
			}
			*/
			
			block_op[thread_idx] = (short)PALS_GPU_PRTASK_MOVE;
			block_task_x[thread_idx] = (unsigned short)task_x;
			block_task_y[thread_idx] = 0;
			block_machine_a[thread_idx] = (unsigned short)machine_a;
			block_machine_b[thread_idx] = (unsigned short)machine_b;
			block_machine_a_ct_new[thread_idx] = machine_a_ct_new;
			block_machine_b_ct_new[thread_idx] = machine_b_ct_new;			
			block_delta[thread_idx] = delta;
		}
		
		__syncthreads();

		// Aplico reduce para quedarme con el mejor movimiento.
		int pos;
		for (int i = 1; i < PALS_GPU_PRTASK__THREADS; i *= 2) {
			pos = 2 * i * thread_idx;
	
			if (pos < PALS_GPU_PRTASK__THREADS) {
				if (block_delta[pos] > block_delta[pos + i]) {			
					block_op[pos] = block_op[pos + i];
					block_task_x[pos] = block_task_x[pos + i];
					block_task_y[pos] = block_task_y[pos + i];
					block_machine_a[pos] = block_machine_a[pos + i];
					block_machine_b[pos] = block_machine_b[pos + i];
					block_machine_a_ct_new[pos] = block_machine_a_ct_new[pos + i];
					block_machine_b_ct_new[pos] = block_machine_b_ct_new[pos + i];
					block_delta[pos] = block_delta[pos + i];
				}
			}
	
			__syncthreads();
		}
		
		// Aplico el mejor movimiento encontrado en la iteración a la solución del bloque.
		/*
		if (thread_idx == 0) {
			if (block_op[0] == PALS_GPU_PRTASK_SWAP) {
				// SWAP
				gpu_task_assignment[task_assignment_offset + block_task_x[0]] = block_machine_b[0];
				gpu_task_assignment[task_assignment_offset + block_task_y[0]] = block_machine_a[0];
		
				gpu_machine_compute_time[machine_compute_time_offset + block_machine_a[0]] = block_machine_a_ct_new[0];
				gpu_machine_compute_time[machine_compute_time_offset + block_machine_b[0]] = block_machine_b_ct_new[0];
			} else {
				// MOVE
				gpu_task_assignment[task_assignment_offset + block_task_x[0]] = block_machine_b[0];
		
				gpu_machine_compute_time[machine_compute_time_offset + block_machine_a[0]] = block_machine_a_ct_new[0];
				gpu_machine_compute_time[machine_compute_time_offset + block_machine_b[0]] = block_machine_b_ct_new[0];
			}
		}
		*/
	}
}

void pals_gpu_prtask_init(struct matrix *etc_matrix, struct solution *s, struct pals_gpu_prtask_instance &instance) {
	// Asignación del paralelismo del algoritmo.
	instance.blocks = PALS_GPU_PRTASK__BLOCKS;
	instance.threads = PALS_GPU_PRTASK__THREADS;
	instance.loops = PALS_GPU_PRTASK__LOOPS;
	
	// Cantidad total de movimientos a evaluar.
	instance.total_tasks = PALS_GPU_PRTASK__BLOCKS * PALS_GPU_PRTASK__THREADS * PALS_GPU_PRTASK__LOOPS;
	
	if (DEBUG) {
		fprintf(stdout, "[INFO] Number of blocks (grid size)   : %d\n", instance.blocks);
		fprintf(stdout, "[INFO] Threads per block (block size) : %d\n", instance.threads);
		fprintf(stdout, "[INFO] Loops per thread               : %d\n", instance.loops);
		fprintf(stdout, "[INFO] Total tasks                    : %ld\n", instance.total_tasks);
	}
	
	// =========================================================================
		
	timespec ts_2;
	timming_start(ts_2);
	
	// Copio la matriz de ETC.
	int etc_matrix_size = sizeof(float) * etc_matrix->tasks_count * etc_matrix->machines_count;
	if (hipMalloc((void**)&(instance.gpu_etc_matrix), etc_matrix_size) != hipSuccess) {
		fprintf(stderr, "[ERROR] Solicitando memoria etc_matrix (%d bytes).\n", etc_matrix_size);
		exit(EXIT_FAILURE);
	}
	
	if (hipMemcpy(instance.gpu_etc_matrix, etc_matrix->data, etc_matrix_size, hipMemcpyHostToDevice) != hipSuccess) {
		fprintf(stderr, "[ERROR] Copiando etc_matrix al dispositivo (%d bytes).\n", etc_matrix_size);
		exit(EXIT_FAILURE);
	}

	timming_end(".. gpu_etc_matrix", ts_2);

	// =========================================================================
	
	timespec ts_3;
	timming_start(ts_3);
	
	// Copio la asignación de tareas a máquinas actuales.
	int task_assignment_size = sizeof(int) * etc_matrix->tasks_count * PALS_GPU_PRTASK__BLOCKS;	
	
	if (hipMalloc((void**)&(instance.gpu_task_assignment), task_assignment_size) != hipSuccess) {
		fprintf(stderr, "[ERROR] Solicitando memoria task_assignment (%d bytes).\n", task_assignment_size);
		exit(EXIT_FAILURE);
	} else {
		if (DEBUG) fprintf(stdout, "[DEBUG] Se solicitaron %d bytes de memoria para task_assignment.\n", task_assignment_size);
	}
	
	if (DEBUG) fprintf(stdout, "[DEBUG] task_assignment_size = %d\n", task_assignment_size);
	int *aux_host_task_assignment = (int*)(malloc(task_assignment_size));

	for (int i = 0; i < PALS_GPU_PRTASK__BLOCKS; i++) {
		int aux_size;
		aux_size = sizeof(int) * etc_matrix->tasks_count;

		int offset;
		offset = i * etc_matrix->tasks_count;
	
		if (DEBUG) fprintf(stdout, "[DEBUG] size en paso %d = %d. from %d\n", i, aux_size, offset);

		if (!memcpy(&(aux_host_task_assignment[offset]), s->task_assignment, aux_size)) {
			fprintf(stdout, "[ERROR] Copiando task_assignment\n");
			exit(EXIT_FAILURE);
		}
	}
	
	// Copio la asignación de tareas de la primer solución desde el HUESPED al DISPOSITIVO.
	if (hipMemcpy(instance.gpu_task_assignment, aux_host_task_assignment, task_assignment_size, 
		hipMemcpyHostToDevice) != hipSuccess) {
		
		fprintf(stderr, "[ERROR] Copiando task_assignment al dispositivo (%d bytes).\n", task_assignment_size);
		exit(EXIT_FAILURE);
	}

	free(aux_host_task_assignment);

	timming_end(".. gpu_task_assignment", ts_3);

	// =========================================================================
	
	timespec ts_4;
	timming_start(ts_4);
		
	// Copio el compute time de las máquinas en la solución actual.
	int machine_compute_time_size = sizeof(float) * etc_matrix->machines_count * PALS_GPU_PRTASK__BLOCKS;
	
	if (hipMalloc((void**)&(instance.gpu_machine_compute_time), machine_compute_time_size) != hipSuccess) {
		fprintf(stderr, "[ERROR] Solicitando memoria machine_compute_time (%d bytes).\n", machine_compute_time_size);
		exit(EXIT_FAILURE);
	}
	
	if (DEBUG) fprintf(stdout, "[DEBUG] machine_compute_time_size = %d\n", machine_compute_time_size);
	float *aux_host_machine_compute_time = (float*)(malloc(machine_compute_time_size));

	for (int i = 0; i < PALS_GPU_PRTASK__BLOCKS; i++) {
                int aux_size;
                aux_size = sizeof(float) * etc_matrix->machines_count;

                int offset;
                offset = i * etc_matrix->machines_count;

                if (DEBUG) fprintf(stdout, "[DEBUG] size en paso %d = %d. from %d\n", i, aux_size, offset);

		if (!memcpy(&(aux_host_machine_compute_time[offset]), s->machine_compute_time, aux_size)) {
			fprintf(stdout, "[ERROR] Copiando machine_compute_time\n");
			exit(EXIT_FAILURE);
		}
	}

	if (hipMemcpy(instance.gpu_machine_compute_time, aux_host_machine_compute_time, machine_compute_time_size, 
		hipMemcpyHostToDevice) != hipSuccess) {
		
		fprintf(stderr, "[ERROR] Copiando machine_compute_time al dispositivo (%d bytes).\n", machine_compute_time_size);
		exit(EXIT_FAILURE);
	}

	free(aux_host_machine_compute_time);

	timming_end(".. gpu_machine_compute_time", ts_4);
	
	// =========================================================================
}

void pals_gpu_prtask_finalize(struct pals_gpu_prtask_instance &instance) {
	if (hipFree(instance.gpu_etc_matrix) != hipSuccess) {
		fprintf(stderr, "[ERROR] Liberando la memoria solicitada para etc_matrix.\n");
		exit(EXIT_FAILURE);
	}
	
	if (hipFree(instance.gpu_task_assignment) != hipSuccess) {
		fprintf(stderr, "[ERROR] Liberando la memoria solicitada para task_assignment.\n");
		exit(EXIT_FAILURE);
	}

	if (hipFree(instance.gpu_machine_compute_time) != hipSuccess) {
		fprintf(stderr, "[ERROR] Liberando la memoria solicitada para machine_compute_time.\n");
		exit(EXIT_FAILURE);
	}
}

void pals_gpu_prtask_wrapper(struct matrix *etc_matrix, struct solution *s, 
	struct pals_gpu_prtask_instance &instance, int *gpu_random_numbers) {
	
	// ==============================================================================
	// Ejecución del algoritmo.
	// ==============================================================================	
	
	// Timming -----------------------------------------------------
	timespec ts_pals;
	timming_start(ts_pals);
	// Timming -----------------------------------------------------
	
	dim3 grid(instance.blocks, 1, 1);
	dim3 threads(instance.threads, 1, 1);

	pals_prtask_kernel<<< grid, threads >>>(
		etc_matrix->machines_count,
		etc_matrix->tasks_count,
		instance.gpu_etc_matrix, 
		instance.gpu_task_assignment, 
		instance.gpu_machine_compute_time, 
		gpu_random_numbers);

	if (TIMMING) hipDeviceSynchronize();

	// Timming -----------------------------------------------------
	timming_end(".. pals_gpu_prtask_pals", ts_pals);
	// Timming -----------------------------------------------------
}

void pals_gpu_prtask_get_solutions(struct matrix *etc_matrix, struct pals_gpu_prtask_instance &instance,
	int *task_assignment, float *machine_compute_time) {

	// Timming -----------------------------------------------------
	timespec ts_get;
	timming_start(ts_get);
	// Timming -----------------------------------------------------

	int machine_compute_time_size = sizeof(float) * etc_matrix->machines_count * PALS_GPU_PRTASK__BLOCKS;	
	if (hipMemcpy(machine_compute_time, instance.gpu_machine_compute_time, machine_compute_time_size, hipMemcpyDeviceToHost) != hipSuccess) {
		fprintf(stderr, "[ERROR] Copiando los compute time de las máquinas desde el dispositivo hacia el huesped (%d bytes).\n", machine_compute_time_size);
		exit(EXIT_FAILURE);
	}

	int task_assignment_size = sizeof(int) * etc_matrix->tasks_count * PALS_GPU_PRTASK__BLOCKS;	
	if (hipMemcpy(task_assignment, instance.gpu_task_assignment, task_assignment_size, hipMemcpyDeviceToHost) != hipSuccess) {	
		fprintf(stderr, "[ERROR] Copiando la asignación de tareas desde el dispositivo hacia el huesped (%d bytes).\n", task_assignment_size);
		exit(EXIT_FAILURE);
	}

	// Timming -----------------------------------------------------
	timming_end(".. pals_gpu_rtask_get", ts_get);
	// Timming -----------------------------------------------------
}

void pals_gpu_prtask_join_solutions(struct pals_gpu_prtask_instance &instance, struct matrix *etc_matrix) {
	// Timming -----------------------------------------------------
	timespec ts_join;
	timming_start(ts_join);
	// Timming -----------------------------------------------------

	// TODO: pasar todo este procesamiento a la GPU!!!
	// Pido el espacio de memoria para obtener los resultados desde la gpu.
	float *machine_compute_time = (float*)malloc(sizeof(float) * etc_matrix->machines_count);
	int best_solution = 0;
	float best_solution_makespan = 0.0;

	for (int i = 0; i < PALS_GPU_PRTASK__BLOCKS; i++) {
		// Copio los mejores movimientos desde el dispositivo.
		if (hipMemcpy(machine_compute_time, 
			instance.gpu_machine_compute_time + (i * sizeof(float) * etc_matrix->machines_count), 
			sizeof(float) * etc_matrix->machines_count, hipMemcpyDeviceToHost) != hipSuccess) {
		
			fprintf(stderr, "[ERROR] Copiando los mejores movimientos al host (best_swaps).\n");
			exit(EXIT_FAILURE);
		}
		
		float makespan;
		makespan = machine_compute_time[0];
		
		for (int j = 1; j < etc_matrix->machines_count; j++) {
			if (machine_compute_time[j] > makespan) {
				makespan = machine_compute_time[j];
			}
		}
		
		if (DEBUG) fprintf(stdout, "[DEBUG] Solution %d, makespan %f.\n", i, makespan);

		if (i == 0) {
			best_solution = 0;
			best_solution_makespan = makespan;
		} else {
			if (makespan < best_solution_makespan) {
				best_solution = i;
				best_solution_makespan = makespan;
			}
		}
	}

	for (int i = 0; i < PALS_GPU_PRTASK__BLOCKS; i++) {
		if (i != best_solution) {
			if (hipMemcpy(
				instance.gpu_machine_compute_time + (i * sizeof(float) * etc_matrix->machines_count), 
				instance.gpu_machine_compute_time + (best_solution * sizeof(float) * etc_matrix->machines_count), 
				sizeof(float) * etc_matrix->machines_count, 
				hipMemcpyDeviceToDevice) != hipSuccess) {
			
				fprintf(stderr, "[ERROR] Copiando machine_compute_time al dispositivo (%ld bytes).\n", 
					sizeof(float) * etc_matrix->machines_count);
				exit(EXIT_FAILURE);
			}
		}
	}

	// Timming -----------------------------------------------------
	timming_end(".. pals_gpu_rtask_join", ts_join);
	// Timming -----------------------------------------------------
}

void pals_gpu_prtask(struct params &input, struct matrix *etc_matrix, struct solution *current_solution) {	
	// ==============================================================================
	// PALS aleatorio por tarea.
	// ==============================================================================
	
	// Timming -----------------------------------------------------
	timespec ts_init;
	timming_start(ts_init);
	// Timming -----------------------------------------------------

	struct pals_gpu_prtask_instance instance;
	
	pals_gpu_prtask_init(etc_matrix, current_solution, instance);

	// Timming -----------------------------------------------------
	timming_end(">> pals_gpu_prtask_init", ts_init);
	// Timming -----------------------------------------------------

	// ===========> DEBUG
	/*if (DEBUG) {
		validate_solution(etc_matrix, current_solution);
	}*/
	// <=========== DEBUG
	
	float makespan_inicial = current_solution->makespan;
	
	// Ejecuto GPUPALS.
	int seed = input.seed;
	
	RNG_rand48 r48;
	RNG_rand48_init(r48, PALS_PRTASK_RANDS);	// Debe ser múltiplo de 6144

	// Cantidad de números aleatorios por invocación.
	const unsigned int size = instance.blocks * (2 * instance.loops); // 2 random numbers por block x loop.
	const short cant_iter_generadas = PALS_PRTASK_RANDS / size;
	fprintf(stdout, "[INFO] Cantidad de iteraciones por generación de numeros aleatorios: %d.\n", cant_iter_generadas);
	
	for (int i = 0; i < PALS_COUNT; i++) {
		if (DEBUG) fprintf(stdout, "[INFO] Iteracion %d =====================\n", i);

		// ==============================================================================
		// Sorteo de numeros aleatorios.
		// ==============================================================================

		// Timming -----------------------------------------------------	
		timespec ts_rand;
		timming_start(ts_rand);
		// Timming -----------------------------------------------------
			
		if (i % cant_iter_generadas == 0) {
			if (DEBUG) fprintf(stdout, "[INFO] Generando %d números aleatorios...\n", PALS_PRTASK_RANDS);
			RNG_rand48_generate(r48, seed);
		}
	
		timming_end(">> RNG_rand48", ts_rand);

		// ==============================================================================
		// PALS.
		// ==============================================================================
	
		// Timming -----------------------------------------------------
		timespec ts_wrapper;
		timming_start(ts_wrapper);
		// Timming -----------------------------------------------------

		pals_gpu_prtask_wrapper(etc_matrix, current_solution, instance, 
			&(r48.res[(i % cant_iter_generadas) * size]));

		// Timming -----------------------------------------------------
		timming_end(">> pals_gpu_prtask_wrapper", ts_wrapper);
		// Timming -----------------------------------------------------

		// ==============================================================================
		// Punto de sincronización.
		// ==============================================================================

		// Timming -----------------------------------------------------
		timespec ts_post;
		timming_start(ts_post);
		// Timming -----------------------------------------------------

		//pals_gpu_prtask_join_solutions(instance);

		// Timming -----------------------------------------------------
		timming_end(">> pals_gpu_prtask_post", ts_post);
		// Timming -----------------------------------------------------

		// Nuevo seed.		
		seed++;
	}
	
	// Timming -----------------------------------------------------
	timespec ts_finalize;
	timming_start(ts_finalize);
	// Timming -----------------------------------------------------
	
	// Libera la memoria del dispositivo con los números aleatorios.
	RNG_rand48_cleanup(r48);
	
	// ==============================================================================
	// Obtengo las soluciones desde el dispositivo.
	// ==============================================================================
	
	int machine_compute_time_size = sizeof(float) * etc_matrix->machines_count * PALS_GPU_PRTASK__BLOCKS;	
	float *machine_compute_time;
	
	if (DEBUG) fprintf(stdout, "[DEBUG] machine_compute_time_size = %d.\n", machine_compute_time_size);

	if (!(machine_compute_time = (float*)malloc(machine_compute_time_size))) {
		fprintf(stderr, "[ERROR] Solicitando memoria para los compute time de las máquinas (%d bytes).\n", machine_compute_time_size);
		exit(EXIT_FAILURE);
	}

	int task_assignment_size = sizeof(int) * etc_matrix->tasks_count * PALS_GPU_PRTASK__BLOCKS;
	int *task_assignment;

	if (DEBUG) fprintf(stdout, "[DEBUG] task_assignment_size = %d.\n", task_assignment_size);
	
	if (!(task_assignment = (int*)malloc(task_assignment_size))) {
		fprintf(stderr, "[ERROR] Solicitando memoria para la asignación de tarea (%d bytes).\n", task_assignment_size);
		exit(EXIT_FAILURE);
	}
	
	pals_gpu_prtask_get_solutions(etc_matrix, instance, task_assignment, machine_compute_time);
	
	// ==============================================================================
	// Actualizo la solución del host con la mejor del dispositivo.
	// ==============================================================================

	int machine_compute_time_offset;
	
	int best_solution = 0;
	float best_solution_makespan = 0.0;

	for (int i = 0; i < PALS_GPU_PRTASK__BLOCKS; i++) {
		machine_compute_time_offset = i * etc_matrix->machines_count;
	
		float makespan;
		makespan = machine_compute_time[machine_compute_time_offset + 0];
		
		for (int j = 1; j < etc_matrix->machines_count; j++) {
			if (machine_compute_time[machine_compute_time_offset + j] > makespan) {
				makespan = machine_compute_time[machine_compute_time_offset + j];
			}
		}
	
		if (DEBUG) fprintf(stdout, "[DEBUG] Solution %d, makespan %f.\n", i, makespan);
	
		if (i == 0) {
			best_solution = 0;
			best_solution_makespan = makespan;
		} else {
			if (makespan < best_solution_makespan) {
				best_solution = i;
				best_solution_makespan = makespan;
			}
		}
	}
	
	if (DEBUG) fprintf(stdout, "[DEBUG] best_solution = %d.\n", best_solution);

	memcpy(current_solution->task_assignment, &(task_assignment[best_solution * etc_matrix->tasks_count]), etc_matrix->tasks_count * sizeof(int));
	memcpy(current_solution->machine_compute_time, &(machine_compute_time[best_solution * etc_matrix->machines_count]), etc_matrix->machines_count * sizeof(float));
	current_solution->makespan = best_solution_makespan;

	free(task_assignment);
	free(machine_compute_time);

	if (DEBUG) {
		for (int i = 0; i < etc_matrix->tasks_count; i++) {
			fprintf(stdout, "[DEBUG] task %d on machine %d.\n", i, current_solution->task_assignment[i]);
		}

		for (int i = 0; i < etc_matrix->machines_count; i++) {
			fprintf(stdout, "[DEBUG] machine %d compute time %f.\n", i, current_solution->machine_compute_time[i]);
		}
	}

	/*
	if (DEBUG) {
		// Validación de la memoria del dispositivo.
		fprintf(stdout, ">> VALIDANDO MEMORIA GPU\n");

		int aux_task_assignment[etc_matrix->tasks_count];
	
		if (hipMemcpy(aux_task_assignment, instance.gpu_task_assignment, (int)(etc_matrix->tasks_count * sizeof(int)), 
			hipMemcpyDeviceToHost) != hipSuccess) {
			
			fprintf(stderr, "[ERROR] Copiando task_assignment al host (%d bytes).\n", (int)(etc_matrix->tasks_count * sizeof(int)));
			exit(EXIT_FAILURE);
		}

		for (int i = 0; i < etc_matrix->tasks_count; i++) {
			if (current_solution->task_assignment[i] != aux_task_assignment[i]) {
				fprintf(stdout, "[INFO] task assignment diff => task %d on host: %d, on device: %d\n",
					i, current_solution->task_assignment[i], aux_task_assignment[i]);
			}
		}

		float aux_machine_compute_time[etc_matrix->machines_count];
	
		if (hipMemcpy(aux_machine_compute_time, instance.gpu_machine_compute_time, (int)(etc_matrix->machines_count * sizeof(float)), 
			hipMemcpyDeviceToHost) != hipSuccess) {
			
			fprintf(stderr, "[ERROR] Copiando machine_compute_time al host (%d bytes).\n", (int)(etc_matrix->machines_count * sizeof(float)));
			exit(EXIT_FAILURE);
		}

		for (int i = 0; i < etc_matrix->machines_count; i++) {
			if (current_solution->machine_compute_time[i] != aux_machine_compute_time[i]) {
				fprintf(stdout, "[INFO] machine CT diff => machine %d on host: %f, on device: %f\n",
					i, current_solution->machine_compute_time[i], aux_machine_compute_time[i]);
			}
		}
	}
	*/

	// Reconstruye el compute time de cada máquina.
	// NOTA: tengo que hacer esto cada tanto por errores acumulados en el redondeo.
	/*for (int i = 0; i < etc_matrix->machines_count; i++) {
		current_solution->machine_compute_time[i] = 0.0;
	}
	
	for (int i = 0; i < etc_matrix->tasks_count; i++) {
		int assigned_machine = current_solution->task_assignment[i];
	
		current_solution->machine_compute_time[assigned_machine] =
			current_solution->machine_compute_time[assigned_machine] + 
			get_etc_value(etc_matrix, assigned_machine, i);
	}	
	
	// Actualiza el makespan de la solución.
	current_solution->makespan = current_solution->machine_compute_time[0];
	for (int i = 1; i < etc_matrix->machines_count; i++) {
		if (current_solution->makespan < current_solution->machine_compute_time[i]) {
			current_solution->makespan = current_solution->machine_compute_time[i];
		}
	}*/
	
	// ===========> DEBUG
	if (DEBUG) {
		validate_solution(etc_matrix, current_solution);
	}
	// <=========== DEBUG
	
	//if (DEBUG) {
		fprintf(stdout, "[DEBUG] Viejo makespan: %f\n", makespan_inicial);
		fprintf(stdout, "[DEBUG] Nuevo makespan: %f\n", current_solution->makespan);
	//}

	// Libero la memoria del dispositivo.
	pals_gpu_prtask_finalize(instance);
	
	// Timming -----------------------------------------------------
	timming_end(">> pals_gpu_prtask_finalize", ts_finalize);
	// Timming -----------------------------------------------------		
}

