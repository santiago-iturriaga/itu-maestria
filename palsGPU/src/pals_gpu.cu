#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include "config.h"
#include "utils.h"
#include "pals_gpu.h"

#define THREADS_PER_BLOCK 128
#define MIN_TASKS_PER_THREAD 4
#define MAX_BLOCKS 1024
//65536

void fake_pals_kernel(int block_id, int thread_id, int task_count, int machine_count, struct matrix etc, struct solution s, 
	struct pals_gpu_instance instance);

__global__ void pals_kernel(int task_count, int block_size, int tasks_per_thread, float *gpu_etc_matrix, 
	int *gpu_task_assignment, int *gpu_best_swaps, float *gpu_best_swaps_delta);

void pals_gpu_init(struct matrix *etc_matrix, struct solution *s, struct pals_gpu_instance *instance) {
	// Cantidad de hilos por bloque.
	instance->block_size = THREADS_PER_BLOCK;
	// Cantidad total de swaps a evaluar.
	instance->total_tasks = (unsigned long)etc_matrix->tasks_count * (unsigned long)etc_matrix->tasks_count;
	// TODO: En realidad la cantidad de tasks esta dada por: (n*n)-((n+1)*(n))/2.
	//       Hay que arreglar esto y arreglar la función de coordenadas.

	int tasks_per_thread = (int)ceil(instance->total_tasks / (unsigned long)(MAX_BLOCKS * THREADS_PER_BLOCK));
	if (tasks_per_thread < MIN_TASKS_PER_THREAD) {
		// Cantidad de swaps evalúa cada hilo.
		instance->tasks_per_thread = MIN_TASKS_PER_THREAD;
		
		// Cantidad de bloques necesarios para evaluar todos los swaps.
		instance->number_of_blocks = (int)ceil(instance->total_tasks / (unsigned long)(THREADS_PER_BLOCK * MIN_TASKS_PER_THREAD));
	} else {
		instance->number_of_blocks = MAX_BLOCKS;
		instance->tasks_per_thread = tasks_per_thread;
	}
	
	if (DEBUG) {
		fprintf(stdout, "[INFO] Block size (block threads)   : %d\n", instance->block_size);
		fprintf(stdout, "[INFO] Tasks per thread             : %d\n", instance->tasks_per_thread);
		fprintf(stdout, "[INFO] Total tasks                  : %ld\n", instance->total_tasks);
		fprintf(stdout, "[INFO] Number of blocks (grid size) : %d\n", instance->number_of_blocks);
	}

	// Pedido de memoria en el dispositivo y copiado de datos.
	timespec ts_2;
	timming_start(ts_2);
	
	// Copio la matriz de ETC.
	int etc_matrix_size = sizeof(float) * etc_matrix->tasks_count * etc_matrix->machines_count;
	hipMalloc((void**)&(instance->gpu_etc_matrix), etc_matrix_size);
	hipMemcpy(instance->gpu_etc_matrix, etc_matrix->data, etc_matrix_size, hipMemcpyHostToDevice);	

	timming_end("gpu_etc_matrix", ts_2);

	timespec ts_3;
	timming_start(ts_3);
		
	// Copio la asignación de tareas a máquinas actuales.
	int task_assignment_size = sizeof(int) * etc_matrix->tasks_count;	
	hipMalloc((void**)&(instance->gpu_task_assignment), task_assignment_size);
	hipMemcpy(instance->gpu_task_assignment, s->task_assignment, task_assignment_size, hipMemcpyHostToDevice);	

	timming_end("gpu_task_assignment", ts_3);

	timespec ts_4;
	timming_start(ts_4);
	
	// Pido memoria para guardar el resultado.
	int best_swaps_size = sizeof(int) * instance->number_of_blocks;	
	hipMalloc((void**)&(instance->gpu_best_swaps), best_swaps_size);
		
	int best_swaps_delta_size = sizeof(float) * instance->number_of_blocks;	
	hipMalloc((void**)&(instance->gpu_best_swaps_delta), best_swaps_delta_size);
	
	timming_end("gpu_best_swaps", ts_4);
}

void pals_gpu_finalize(struct pals_gpu_instance *instance) {
	hipFree(instance->gpu_etc_matrix);
	hipFree(instance->gpu_task_assignment);
	hipFree(instance->gpu_best_swaps);
}

void pals_gpu_wrapper(struct matrix *etc_matrix, struct solution *s, struct pals_gpu_instance *instance, 
	int &best_swaps_count, int best_swaps[], float best_swaps_delta[]) {
	
	dim3 grid(instance->number_of_blocks, 1, 1);
	dim3 threads(instance->block_size, 1, 1);

	/*
	for (int block_id = 0; block_id < instance->number_of_blocks; block_id++) {
		fprintf(stdout, "[DEBUG] Block: %i ===============================================\n", block_id);
		
		for (int thread_id = 0; thread_id < instance->block_size; thread_id++) {
			fprintf(stdout, "[DEBUG] >>> Thread: %i\n", thread_id);
			
			fake_pals_kernel(
				block_id, 
				thread_id,
				etc_matrix->tasks_count, 
				etc_matrix->machines_count, 
				*etc_matrix, *s, *instance);
		}
	}
	*/

	pals_kernel<<< grid, threads >>>(
		etc_matrix->tasks_count, 
		instance->block_size, 
		instance->tasks_per_thread, 
		instance->gpu_etc_matrix, 
		instance->gpu_task_assignment, 
		instance->gpu_best_swaps, 
		instance->gpu_best_swaps_delta);

	// Copio los mejores movimientos desde el dispositivo.
	hipMemcpy(best_swaps, instance->gpu_best_swaps, sizeof(int) * instance->number_of_blocks, hipMemcpyDeviceToHost);
	hipMemcpy(best_swaps_delta, instance->gpu_best_swaps_delta, sizeof(float) * instance->number_of_blocks, hipMemcpyDeviceToHost);
	
	best_swaps_count = instance->number_of_blocks;
}

__global__ void pals_kernel(int task_count, int block_size,
	int tasks_per_thread, float *gpu_etc_matrix, int *gpu_task_assignment, 
	int *gpu_best_swaps, float *gpu_best_swaps_delta)
{
	// Configuración optima (¿?):
	// 128 threads.
	// 16 registros por thread.
	// 2K shared memory por block.

	const unsigned int thread_idx = threadIdx.x;
	const unsigned int block_idx = blockIdx.x;

	const float block_offset_start = block_size * tasks_per_thread * block_idx;

	__shared__ int block_best_swaps[THREADS_PER_BLOCK];
	__shared__ float block_best_swaps_delta[THREADS_PER_BLOCK];

	// Busco el mejor movimiento de cada hilo.
	int i, aux;
	float auxf;

	// Coordenadas del swap.
	auxf = (block_offset_start + thread_idx) / task_count;
	int current_swap_coord_x = (int)auxf;
	int current_swap_coord_y = (int)((auxf - current_swap_coord_x) * task_count);

	// El primer task_per_thread siempre debería tener un swap válido.
	// Calculo el delta de ese primer swap y lo dejo como mejor.		
	aux = gpu_task_assignment[current_swap_coord_x]; // Máquina a.

	float current_swap_delta_xa = gpu_etc_matrix[(aux * task_count) + current_swap_coord_x]; // Resto del ETC de x en a.
	float current_swap_delta_ya = gpu_etc_matrix[(aux * task_count) + current_swap_coord_y]; // Sumo el ETC de y en a.
	
	aux = gpu_task_assignment[current_swap_coord_y]; // Máquina b.
	
	float current_swap_delta_xb = gpu_etc_matrix[(aux * task_count) + current_swap_coord_x]; // Resto el ETC de y en b.
	float current_swap_delta_yb = gpu_etc_matrix[(aux * task_count) + current_swap_coord_y]; // Sumo el ETC de x en b.

	block_best_swaps[thread_idx] = tasks_per_thread * thread_idx;
	block_best_swaps_delta[thread_idx] = current_swap_delta_ya - current_swap_delta_xa + current_swap_delta_xb - current_swap_delta_yb;

	// Para todos los demás task_per_thread.
	// En caso de que task_per_thread = 1, esto nunca se ejecuta y nunca hay divergencia de código.
	for (i = 1; i < tasks_per_thread; i++) {
		auxf = (block_offset_start  + (block_size * i) + thread_idx) / task_count;
		current_swap_coord_x = (int)auxf;
		current_swap_coord_y = (int)((auxf - current_swap_coord_x) * task_count);

		// Si la cantidad de tareas no es divisible entre la cantidad de threads
		// per block, el último bloque puede tener threads sobrantes. En este
		// caso se pierde la coherencia de los threads del último bloque.
		if (current_swap_coord_x < task_count) {
	
			// Prefiero calcular cosas inutiles con tal de mantener la coherencia entre threads.
			//if ((x < y) && (machine_a != machine_b)) {

				// Calculo el delta del swap i.
				aux = gpu_task_assignment[current_swap_coord_x]; // Máquina a.
	
				current_swap_delta_xa = gpu_etc_matrix[(aux * task_count) + current_swap_coord_x]; // Resto del ETC de x en a.
				current_swap_delta_ya = gpu_etc_matrix[(aux * task_count) + current_swap_coord_y]; // Sumo el ETC de y en a.
	
				aux = gpu_task_assignment[current_swap_coord_y]; // Máquina b.
	
				current_swap_delta_xb = gpu_etc_matrix[(aux * task_count) + current_swap_coord_x]; // Resto el ETC de y en b.
				current_swap_delta_yb = gpu_etc_matrix[(aux * task_count) + current_swap_coord_y]; // Sumo el ETC de x en b.

				auxf = current_swap_delta_ya - current_swap_delta_xa + current_swap_delta_xb - current_swap_delta_yb;
	
				if (auxf < block_best_swaps_delta[thread_idx]) {
					// Si es mejor que el mejor delta que tenía hasta el momento, lo guardo.
					
					block_best_swaps[thread_idx] = (tasks_per_thread * thread_idx) + i;
					block_best_swaps_delta[thread_idx] = auxf;
				}
		
			//}
		}
	}
	
	__syncthreads(); // Sincronizo todos los threads para asegurarme que todos los 
					 // mejores swaps esten copiados a la memoria compartida.
	
	// Aplico reduce para quedarme con el mejor delta.
	for (i = 1; i < THREADS_PER_BLOCK; i *= 2) {
		aux = 2 * i * thread_idx;
		
		if (aux < THREADS_PER_BLOCK) {
			if (block_best_swaps_delta[aux] > block_best_swaps_delta[aux + i]) {
				block_best_swaps_delta[aux] = block_best_swaps_delta[aux + i];
				block_best_swaps[aux] = block_best_swaps[aux + i];
			}
		}
		
		__syncthreads();
	}

	if (thread_idx == 0) {
		gpu_best_swaps[block_idx] = block_best_swaps[0]; //best_swap;
		gpu_best_swaps_delta[block_idx] = block_best_swaps_delta[0]; //best_swap_delta;
	}
}

void fake_pals_kernel(int block_id, int thread_id, int task_count, int machine_count, struct matrix etc, 
	struct solution s, struct pals_gpu_instance instance) {
	
	const unsigned int thread_idx = thread_id;
	const unsigned int block_idx = block_id;
	
	const int block_size = instance.block_size;	
	const int tasks_per_thread = instance.tasks_per_thread;
	
	const float block_offset_start = block_size * tasks_per_thread * block_idx;

	for (int i = 0; i < instance.tasks_per_thread; i++) {
		// Coordenadas del swap.
		float auxf = (block_offset_start + (instance.block_size * i) + thread_idx) / task_count;
		int current_swap_coord_x = (int)auxf;
		int current_swap_coord_y = (int)((auxf - current_swap_coord_x) * task_count);
	
		fprintf(stdout, "[%f] %d x %d\n", (block_offset_start + (instance.block_size * i) + thread_idx), current_swap_coord_x, current_swap_coord_y);
	}

	/*	
	int current_swap = (instance.block_size * instance.tasks_per_thread * block_idx) + (instance.block_size * 0) + thread_idx;
	
	current_swap_coord_x = (int)(current_swap / task_count);
	current_swap_coord_y = (int)(current_swap % task_count);
	
	fprintf(stdout, "%d x %d\n", current_swap_coord_x, current_swap_coord_y);
	*/
	/*const int block_size = instance.block_size;	
	const int tasks_per_thread = instance.tasks_per_thread;
	const float *gpu_etc_matrix = etc.data;
	const int *gpu_task_assignment = s.task_assignment;
	
	int block_offset_start = instance.block_size * instance.tasks_per_thread * block_idx;
	int block_offset_end = instance.block_size * instance.tasks_per_thread * (block_idx + 1) - 1;
	
	block_offset_start = block_size * tasks_per_thread * block_idx;

	// Busco el mejor movimiento de cada hilo.
	int current_swap;
	int best_swap;
	float best_swap_delta;

	// Siempre debería haber al menos un task_per_thread.
	current_swap = block_offset_start + thread_idx; // i = 0
	
	// Coordenadas del swap.
	//current_swap_coord_x = (int)floor((float)current_swap / (float)task_count);
	//current_swap_coord_y = (int)fmod((float)current_swap, (float)task_count);

	// El primer task_per_thread siempre debería tener un swap válido.
	// Calculo el delta de ese primer swap y lo dejo como mejor.
	best_swap = current_swap;
	best_swap_delta = 0.0;
		
	fprintf(stdout, "[DEBUG] >>>         [task x: %d]\n", (int)floor((float)current_swap / (float)task_count));
	fprintf(stdout, "[DEBUG] >>>         [machine a: %d]\n", gpu_task_assignment[(int)floor((float)current_swap / (float)task_count)]);
	fprintf(stdout, "[DEBUG] >>>         [task y: %d]\n", (int)fmod((float)current_swap, (float)task_count));
	fprintf(stdout, "[DEBUG] >>>         [machine b: %d]\n", gpu_task_assignment[(int)fmod((float)current_swap, (float)task_count)]);
		
	int machine = gpu_task_assignment[(int)floor((float)current_swap / (float)task_count)]; // Máquina a.
	
	best_swap_delta -= gpu_etc_matrix[machine * ((int)floor((float)current_swap / (float)task_count))]; // Resto del ETC de x en a.
	best_swap_delta += gpu_etc_matrix[machine * ((int)fmod((float)current_swap, (float)task_count))];; // Sumo el ETC de y en a.
	
	machine = gpu_task_assignment[(int)fmod((float)current_swap, (float)task_count)]; // Máquina b.
	
	best_swap_delta -= gpu_etc_matrix[machine * ((int)fmod((float)current_swap, (float)task_count))]; // Resto el ETC de y en b.
	best_swap_delta += gpu_etc_matrix[machine * ((int)floor((float)current_swap / (float)task_count))]; // Sumo el ETC de x en b.

	fprintf(stdout, "[DEBUG] >>>         [rango asignado al bloque: %i-%i]\n", block_offset_start, block_offset_end);
	fprintf(stdout, "[DEBUG] >>>         [swap      : %d]\n", best_swap);
	fprintf(stdout, "[DEBUG] >>>         [swap delta: %f]\n", best_swap_delta);*/
}
