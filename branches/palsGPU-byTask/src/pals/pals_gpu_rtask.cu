#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <limits.h>
#include <assert.h>

#include "../config.h"
#include "../utils.h"
#include "../random/RNG_rand48.h"

#include "pals_gpu_rtask.h"

// ==============================================================================================
// NOTA. Debido al uso del generador de números aleatorios RNG_rand48:
// PALS_GPU_RTASK__BLOCKS * PALS_GPU_RTASK__LOOPS_PER_THREAD = debe ser mútiplo de 6144 (1024*6).
// ==============================================================================================
//#define PALS_GPU_RTASK__BLOCKS 			1024
//#define PALS_GPU_RTASK__THREADS 			256
//#define PALS_GPU_RTASK__LOOPS_PER_THREAD 	24

//#define PALS_GPU_RTASK__BLOCKS 			2048
//#define PALS_GPU_RTASK__THREADS 			128
//#define PALS_GPU_RTASK__LOOPS_PER_THREAD 	24

/*
#define PALS_GPU_RTASK__BLOCKS 				1024
#define PALS_GPU_RTASK__THREADS 			128
#define PALS_GPU_RTASK__LOOPS_PER_THREAD 	48
*/

#define PALS_GPU_RTASK__BLOCKS 				16
#define PALS_GPU_RTASK__THREADS 			16
#define PALS_GPU_RTASK__LOOPS_PER_THREAD 	8

#define INT_HALF_MAX						1073741823

__global__ void pals_rtask_kernel(int machines_count, int tasks_count, int number_of_blocks, 
	int threads_per_block, int tasks_per_thread, float *gpu_etc_matrix, int *gpu_task_assignment, 
	int *gpu_random_numbers, int *gpu_best_swaps, float *gpu_best_swaps_delta
	/*,int *gpu_taskx, int *gpu_tasky, int *gpu_loop, int *gpu_thread*/);

void pals_gpu_rtask_init(struct matrix *etc_matrix, struct solution *s, struct pals_gpu_rtask_instance *instance) {	
	// Asignación del paralelismo del algoritmo.
	instance->number_of_blocks = PALS_GPU_RTASK__BLOCKS;
	instance->threads_per_block = PALS_GPU_RTASK__THREADS;
	instance->tasks_per_thread = PALS_GPU_RTASK__LOOPS_PER_THREAD;
	
	// Cantidad total de movimientos a evaluar.
	instance->total_tasks = PALS_GPU_RTASK__BLOCKS * PALS_GPU_RTASK__THREADS * PALS_GPU_RTASK__LOOPS_PER_THREAD;
	
	if (DEBUG) {
		fprintf(stdout, "[INFO] Number of blocks (grid size)   : %d\n", instance->number_of_blocks);
		fprintf(stdout, "[INFO] Threads per block (block size) : %d\n", instance->threads_per_block);	
		fprintf(stdout, "[INFO] Tasks per thread               : %d\n", instance->tasks_per_thread);
		fprintf(stdout, "[INFO] Total tasks                    : %d\n", instance->total_tasks);
	}

	// Pedido de memoria en el dispositivo y copiado de datos.
	timespec ts_4;
	timming_start(ts_4);
	
	// Pido memoria para guardar el resultado.
	int best_swaps_size = sizeof(int) * instance->number_of_blocks;	
	hipMalloc((void**)&(instance->gpu_best_swaps), best_swaps_size);
		
	int best_swaps_delta_size = sizeof(float) * instance->number_of_blocks;	
	hipMalloc((void**)&(instance->gpu_best_swaps_delta), best_swaps_delta_size);
	
	timming_end("gpu_best_swaps", ts_4);
		
	timespec ts_2;
	timming_start(ts_2);
	
	// Copio la matriz de ETC.
	int etc_matrix_size = sizeof(float) * etc_matrix->tasks_count * etc_matrix->machines_count;
	hipMalloc((void**)&(instance->gpu_etc_matrix), etc_matrix_size);
	hipMemcpy(instance->gpu_etc_matrix, etc_matrix->data, etc_matrix_size, hipMemcpyHostToDevice);	

	timming_end("gpu_etc_matrix", ts_2);

	timespec ts_3;
	timming_start(ts_3);
		
	// Copio la asignación de tareas a máquinas actuales.
	int task_assignment_size = sizeof(int) * etc_matrix->tasks_count;	
	hipMalloc((void**)&(instance->gpu_task_assignment), task_assignment_size);
	hipMemcpy(instance->gpu_task_assignment, s->task_assignment, task_assignment_size, hipMemcpyHostToDevice);	

	timming_end("gpu_task_assignment", ts_3);
}

void pals_gpu_rtask_finalize(struct pals_gpu_rtask_instance *instance) {
	hipFree(instance->gpu_etc_matrix);
	hipFree(instance->gpu_task_assignment);
	hipFree(instance->gpu_best_swaps);
}

void pals_gpu_rtask_wrapper(struct matrix *etc_matrix, struct solution *s, 
	struct pals_gpu_rtask_instance &instance, int seed, 
	struct pals_gpu_rtask_result &result) {

	// ==============================================================================
	// DEBUG: tareas evaluadas.
	/*int *gpu_taskx;
	int taskx_size = sizeof(int) * PALS_GPU_RTASK__BLOCKS;
	hipMalloc((void**)&(gpu_taskx), taskx_size);

	int *gpu_tasky;
	int tasky_size = sizeof(int) * PALS_GPU_RTASK__BLOCKS;
	hipMalloc((void**)&(gpu_tasky), tasky_size);
	
	int *gpu_loop;
	int loop_size = sizeof(int) * PALS_GPU_RTASK__BLOCKS;
	hipMalloc((void**)&(gpu_loop), loop_size);
	
	int *gpu_thread;
	int thread_size = sizeof(int) * PALS_GPU_RTASK__BLOCKS;
	hipMalloc((void**)&(gpu_thread), thread_size);*/
	// ==============================================================================
		
	// ==============================================================================
	// Sorteo de numeros aleatorios.
	// ==============================================================================
	
	// Evals 49.152 rands => 6.291.456 movimientos (1024*24*256)(debe ser múltiplo de 6144).
	//const unsigned int size = 6144;
	const unsigned int size = PALS_GPU_RTASK__BLOCKS * PALS_GPU_RTASK__LOOPS_PER_THREAD * 2;
	
	fprintf(stdout, "[DEBUG] Generando %d números aleatorios...\n", size);
	
	RNG_rand48 r48;
	RNG_rand48_init(r48, seed, size);	
	RNG_rand48_generate(r48);
	
	// ==============================================================================
	// Ejecución del algoritmo.
	// ==============================================================================	
	
	dim3 grid(instance.number_of_blocks, 1, 1);
	dim3 threads(instance.threads_per_block, 1, 1);

	pals_rtask_kernel<<< grid, threads >>>(
		etc_matrix->machines_count,
		etc_matrix->tasks_count,
		instance.number_of_blocks, 
		instance.threads_per_block, 
		instance.tasks_per_thread, 
		instance.gpu_etc_matrix, 
		instance.gpu_task_assignment, 
		r48.res,
		instance.gpu_best_swaps, 
		instance.gpu_best_swaps_delta
		/*,gpu_taskx, gpu_tasky, gpu_loop, gpu_thread*/);

	result.best_swaps = (int*)malloc(sizeof(int) * instance.number_of_blocks);
	result.best_swaps_delta = (float*)malloc(sizeof(float) * instance.number_of_blocks);
	result.rands_nums = (int*)malloc(sizeof(int) * size);

	// Copio los mejores movimientos desde el dispositivo.
	hipMemcpy(result.best_swaps, instance.gpu_best_swaps, sizeof(int) * instance.number_of_blocks, hipMemcpyDeviceToHost);
	hipMemcpy(result.best_swaps_delta, instance.gpu_best_swaps_delta, sizeof(float) * instance.number_of_blocks, hipMemcpyDeviceToHost);
	hipMemcpy(result.rands_nums, r48.res, sizeof(int) * size, hipMemcpyDeviceToHost);

	// ==============================================================================
	// DEBUG: tareas evaluadas.	
	/*result.taskx = (int*)malloc(sizeof(int) * instance.number_of_blocks);
	result.tasky = (int*)malloc(sizeof(int) * instance.number_of_blocks);
	result.loop = (int*)malloc(sizeof(int) * instance.number_of_blocks);
	result.thread = (int*)malloc(sizeof(int) * instance.number_of_blocks);
	
	hipMemcpy(result.taskx, gpu_taskx, sizeof(int) * instance.number_of_blocks, hipMemcpyDeviceToHost);
	hipMemcpy(result.tasky, gpu_tasky, sizeof(int) * instance.number_of_blocks, hipMemcpyDeviceToHost);
	hipMemcpy(result.loop, gpu_loop, sizeof(int) * instance.number_of_blocks, hipMemcpyDeviceToHost);
	hipMemcpy(result.thread, gpu_thread, sizeof(int) * instance.number_of_blocks, hipMemcpyDeviceToHost);
	
	hipFree(gpu_taskx);
	hipFree(gpu_tasky);
	hipFree(gpu_loop);
	hipFree(gpu_thread);*/
	// ==============================================================================
	
	// Libera la memoria del dispositivo con los números aleatorios.
	RNG_rand48_cleanup(r48);
}

__global__ void pals_rtask_kernel(int machines_count, int tasks_count, int number_of_blocks, 
	int threads_per_block, int tasks_per_thread, float *gpu_etc_matrix, int *gpu_task_assignment, 
	int *gpu_random_numbers, int *gpu_best_swaps, float *gpu_best_swaps_delta
	/*,int *gpu_taskx, int *gpu_tasky, int *gpu_loop, int *gpu_thread*/)
{
	const unsigned int thread_idx = threadIdx.x;
	const unsigned int block_idx = blockIdx.x;

	__shared__ int block_best_swap;
	__shared__ float block_best_swap_delta;
	
	__shared__ int block_swaps[PALS_GPU_RTASK__THREADS];
	__shared__ float block_swaps_delta[PALS_GPU_RTASK__THREADS];

	// Offset de los random numbers asignados al block (2 rand x loop).
	const int r_block_offset_start = block_idx * (2 * PALS_GPU_RTASK__LOOPS_PER_THREAD);
		
	for (int loop = 0; loop < PALS_GPU_RTASK__LOOPS_PER_THREAD; loop++) {
		// El primer rand. num. es tiempre task 1.
		int raux1, raux2, aux;
		raux1 = gpu_random_numbers[r_block_offset_start + loop];
	
		// Tipo de movimiento.	
		//if (raux1 % 2 == 0) { //TODO: el módulo es muy ineficiente.
			// Movimiento SWAP.

			raux1 = raux1 % tasks_count;
			//assert(raux1 < tasks_count);
					
			raux2 = gpu_random_numbers[r_block_offset_start + loop + 1];
			raux2 = raux2 % (tasks_count - 1 - PALS_GPU_RTASK__THREADS);
			raux2 = raux2 + thread_idx;
			
			if (raux2 >= raux1) {
				raux2 = raux2 + 1;
				
				if (raux2 == tasks_count) raux2 = 0;
			}
			//assert(raux2 < tasks_count);
			
			// Calculo el delta del swap sorteado.
			float current_swap_delta = 0.0;

			aux = gpu_task_assignment[raux1]; // Máquina a.
			current_swap_delta = current_swap_delta - gpu_etc_matrix[(aux * tasks_count) + raux1]; // Resto del ETC de x en a.
			current_swap_delta = current_swap_delta + gpu_etc_matrix[(aux * tasks_count) + raux2]; // Sumo el ETC de y en a.
	
			aux = gpu_task_assignment[raux2]; // Máquina b.	
			current_swap_delta = current_swap_delta - gpu_etc_matrix[(aux * tasks_count) + raux2]; // Resto el ETC de y en b.
			current_swap_delta = current_swap_delta + gpu_etc_matrix[(aux * tasks_count) + raux1]; // Sumo el ETC de x en b.

			block_swaps[thread_idx] = (loop * PALS_GPU_RTASK__THREADS) + thread_idx;
			block_swaps_delta[thread_idx] = current_swap_delta;
			
			/*if (thread_idx == 0) {
				gpu_taskx[block_idx] = raux1;
				gpu_tasky[block_idx] = raux2;
				gpu_loop[block_idx] = loop;
				gpu_thread[block_idx] = thread_idx;
			}*/
		//} else {
			// Movimiento MOVE.
			// TODO: hacer!!!
		//}
		
		__syncthreads(); // Sincronizo todos los threads para asegurarme que todos los 
					 	 // swaps esten copiados a la memoria compartida.
	
		// Aplico reduce para quedarme con el mejor delta.
		for (int i = 1; i < PALS_GPU_RTASK__THREADS; i *= 2) {
			aux = 2 * i * thread_idx;
		
			if (aux < PALS_GPU_RTASK__THREADS) {
				if (block_swaps_delta[aux] > block_swaps_delta[aux + i]) {
					block_swaps_delta[aux] = block_swaps_delta[aux + i];
					block_swaps[aux] = block_swaps[aux + i];
				}
			}
		
			__syncthreads();
		}
		
		if (thread_idx == 0) {
			if (loop == 0) {
				block_best_swap = block_swaps[0];
				block_best_swap_delta = block_swaps_delta[0];
			} else if (block_best_swap_delta > block_swaps_delta[0]) {
				block_best_swap = block_swaps[0];
				block_best_swap_delta = block_swaps_delta[0];
			}
		}
	}
	
	if (thread_idx == 0) {
		gpu_best_swaps[block_idx] = block_best_swap; //best_swap;
		gpu_best_swaps_delta[block_idx] = block_best_swap_delta; //best_swap_delta;
	}
}

