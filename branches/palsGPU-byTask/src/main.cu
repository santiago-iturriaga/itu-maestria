//============================================================================
// Name        : palsGPU.cu
// Author      : Santiago
// Version     : 1.0
// Copyright   : 
// Description : 
//============================================================================

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <limits.h>

#include "load_params.h"
#include "load_instance.h"
#include "etc_matrix.h"
#include "mct.h"
#include "solution.h"
#include "config.h"
#include "utils.h"

#include "pals/pals_serial.h"
#include "pals/pals_gpu.h"
#include "pals/pals_gpu_rtask.h"

/// Búsqueda serial sobre el todo el dominio del problema.
void pals_serial(struct params &input, struct matrix *etc_matrix, struct solution *current_solution);

/// Búsqueda masivamente paralela sobre todo el dominio del problema.
void pals_gpu(struct params &input, struct matrix *etc_matrix, struct solution *current_solution);

/// Búsqueda masivamente paralela sobre un subdominio del problema. 
/// Se sortea el subdominio por tarea.
void pals_gpu_rtask(struct params &input, struct matrix *etc_matrix, struct solution *current_solution);

/// Búsqueda masivamente paralela sobre un subdominio del problema. 
/// Se sortea el subdominio por máquina y se evalúan todas las tareas de esa máquina.
void pals_gpu_rmachine(struct params &input, struct matrix *etc_matrix, struct solution *current_solution);

int main(int argc, char** argv)
{
	// =============================================================
	// Loading input parameters
	// =============================================================
	struct params input;
	if (load_params(argc, argv, &input) == EXIT_FAILURE) {
		fprintf(stderr, "[ERROR] Ocurrió un error leyendo los parametros de entrada.\n");
		return EXIT_FAILURE;
	}

	// =============================================================
	// Loading problem instance
	// =============================================================
	if (DEBUG) fprintf(stdout, "[DEBUG] Loading problem instance...\n");
	
	// Se pide el espacio de memoria para la matriz de ETC.
	struct matrix *etc_matrix = create_etc_matrix(&input);

	// Se carga la matriz de ETC.
	if (load_instance(&input, etc_matrix) == EXIT_FAILURE) {
		fprintf(stderr, "[ERROR] Ocurrió un error leyendo el archivo de instancia.\n");
		return EXIT_FAILURE;
	}

	//show_etc_matrix(etc_matrix);

	// =============================================================
	// Candidate solution
	// =============================================================
	if (DEBUG) fprintf(stdout, "[DEBUG] Creating initial candiate solution...\n");

	struct solution *current_solution = create_empty_solution(etc_matrix);
	compute_mct(etc_matrix, current_solution);
	
	validate_solution(etc_matrix, current_solution);

	// =============================================================
	// PALS
	// =============================================================
	if (DEBUG) fprintf(stdout, "[DEBUG] Executing PALS...\n");
	
	// Timming -----------------------------------------------------
	timespec ts;
	timming_start(ts);
	// Timming -----------------------------------------------------
	
	if (input.pals_flavour == PALS_Serial) {
		// =============================================================
		// Serial
		// =============================================================
		
		pals_serial(input, etc_matrix, current_solution);
		
	} else if (input.pals_flavour == PALS_GPU) {
		// =============================================================
		// CUDA
		// =============================================================		
		
		pals_gpu(input, etc_matrix, current_solution);
		
	} else if (input.pals_flavour == PALS_GPU_randTask) {
	
		pals_gpu_rtask(input, etc_matrix, current_solution);
		
	} else if (input.pals_flavour == PALS_GPU_randMachine) {
	
		pals_gpu_rmachine(input, etc_matrix, current_solution);
		
	}
	
	// Timming -----------------------------------------------------
	timming_end("Elapsed PALS time", ts);
	// Timming -----------------------------------------------------

	// =============================================================
	// Release memory
	// =============================================================
	free_etc_matrix(etc_matrix);
	free_solution(current_solution);

	return EXIT_SUCCESS;
}

void pals_serial(struct params &input, struct matrix *etc_matrix, struct solution *current_solution) {
	int best_swap_task_a;
	int best_swap_task_b;
	float best_swap_delta;
	
	for (int i = 0; i < PALS_COUNT; i++) {
		pals_serial(etc_matrix, current_solution, best_swap_task_a, best_swap_task_b, best_swap_delta);
	}
	
	fprintf(stdout, "[DEBUG] Best swap: task %d for task %d. Gain %f.\n", best_swap_task_a, best_swap_task_b, best_swap_delta);
}

void pals_gpu(struct params &input, struct matrix *etc_matrix, struct solution *current_solution) {
	struct pals_gpu_instance instance;

	// Timming -----------------------------------------------------
	timespec ts_init;
	timming_start(ts_init);
	// Timming -----------------------------------------------------
			
	// Inicializo la memoria en el dispositivo.
	pals_gpu_init(etc_matrix, current_solution, &instance);

	// Timming -----------------------------------------------------
	timming_end("pals_gpu_init", ts_init);
	// Timming -----------------------------------------------------

	int best_swap_count;
	int best_swaps[instance.number_of_blocks];
	float best_swaps_delta[instance.number_of_blocks];

	// Timming -----------------------------------------------------
	timespec ts_wrapper;
	timming_start(ts_wrapper);
	// Timming -----------------------------------------------------
	
	// Ejecuto GPUPALS.
	// for (int i = 0; i < PALS_COUNT; i++) {
	pals_gpu_wrapper(etc_matrix, current_solution, &instance, best_swap_count, best_swaps, best_swaps_delta);
	// }
	
	// Timming -----------------------------------------------------
	timming_end("pals_gpu_wrapper", ts_wrapper);
	// Timming -----------------------------------------------------

	// Debug ------------------------------------------------------------------------------------------
	if (DEBUG) {
		unsigned long current_swap;
		int task_x, task_y;
		int machine_a, machine_b;

		fprintf(stdout, "[DEBUG] Mejores swaps:\n");
		for (int i = 0; i < instance.number_of_blocks; i++) {
			int block_idx = i;
			int thread_idx = best_swaps[i] / instance.tasks_per_thread;
			int task_idx = best_swaps[i] % instance.tasks_per_thread;
		
			current_swap = ((unsigned long)instance.block_size * (unsigned long)instance.tasks_per_thread * (unsigned long)block_idx) 
				+ ((unsigned long)instance.block_size * (unsigned long)task_idx) + (unsigned long)thread_idx;

			float block_offset_start = instance.block_size * instance.tasks_per_thread * block_idx;											
			float auxf = (block_offset_start  + (instance.block_size * task_idx) + thread_idx) / etc_matrix->tasks_count;
			task_x = (int)auxf;
			task_y = (int)((auxf - task_x) * etc_matrix->tasks_count);
			
			if (task_x >= etc_matrix->tasks_count) task_x = etc_matrix->tasks_count - 1;
			if (task_y >= etc_matrix->tasks_count) task_y = etc_matrix->tasks_count - 1;
			if (task_x < 0) task_x = 0;
			if (task_y < 0) task_y = 0;

			machine_a = current_solution->task_assignment[task_x];
			machine_b = current_solution->task_assignment[task_y];

			float swap_delta = 0.0;
			swap_delta -= get_etc_value(etc_matrix, machine_a, task_x); // Resto del ETC de x en a.
			swap_delta += get_etc_value(etc_matrix, machine_a, task_y); // Sumo el ETC de y en a.
			swap_delta -= get_etc_value(etc_matrix, machine_b, task_y); // Resto el ETC de y en b.
			swap_delta += get_etc_value(etc_matrix, machine_b, task_x); // Sumo el ETC de x en b.

			fprintf(stdout, "   GPU Result %d. Swap ID %ld. Task x %d, Task y %d. Delta %f (%f). Task %d in %d swaps with task %d in %d.\n", 
				best_swaps[i], current_swap, (int)auxf, (int)((auxf - task_x) * etc_matrix->tasks_count), 
				best_swaps_delta[i], swap_delta, task_x, machine_a, task_y, machine_b);
		}
	}
	// Debug ------------------------------------------------------------------------------------------

	// Timming -----------------------------------------------------
	timespec ts_finalize;
	timming_start(ts_finalize);
	// Timming -----------------------------------------------------

	// Libero la memoria del dispositivo.
	pals_gpu_finalize(&instance);
	
	// Timming -----------------------------------------------------
	timming_end("pals_gpu_finalize", ts_finalize);
	// Timming -----------------------------------------------------	
}

void pals_gpu_rtask(struct params &input, struct matrix *etc_matrix, struct solution *current_solution) {	
	// ==============================================================================
	// PALS aleatorio por tarea.
	// ==============================================================================
	
	struct pals_gpu_rtask_instance instance;

	// Timming -----------------------------------------------------
	timespec ts_init;
	timming_start(ts_init);
	// Timming -----------------------------------------------------
			
	// Inicializo la memoria en el dispositivo.
	pals_gpu_rtask_init(etc_matrix, current_solution, &instance);

	// Timming -----------------------------------------------------
	timming_end("pals_gpu_rtask_init", ts_init);
	// Timming -----------------------------------------------------

	// Timming -----------------------------------------------------
	timespec ts_wrapper;
	timming_start(ts_wrapper);
	// Timming -----------------------------------------------------
	
	// Ejecuto GPUPALS.
	int seed = input.seed;
	struct pals_gpu_rtask_result result;
	
	//for (int i = 0; i < PALS_COUNT; i++) {
		pals_gpu_rtask_wrapper(etc_matrix, current_solution, instance, seed, result);
		
		// TODO: Evalúo las soluciones...
		// seed++;
	//}
	
	// Timming -----------------------------------------------------
	timming_end("pals_gpu_rtask_wrapper", ts_wrapper);
	// Timming -----------------------------------------------------

	// Debug ------------------------------------------------------------------------------------------
	if (DEBUG) {
		int task_x, task_y;
		int machine_a, machine_b;

		fprintf(stdout, "[DEBUG] Mejores swaps:\n");
		for (int i = 0; i < instance.number_of_blocks; i++) {
			int block_idx = i;
			int thread_idx = result.best_swaps[i] % instance.tasks_per_thread;
			int loop = result.best_swaps[i] / instance.tasks_per_thread;

			int r_block_offset_start = block_idx * (2 * instance.tasks_per_thread);

			int raux1 = result.rands_nums[r_block_offset_start + loop];
			task_x = raux1 % etc_matrix->tasks_count;
			
			int raux2 = result.rands_nums[r_block_offset_start + loop + 1];
			task_y = raux2 % (etc_matrix->tasks_count - 1 - instance.threads_per_block);
			
			if (task_y >= task_x) {
				task_y = task_y + 1;
				
				if (task_y == etc_matrix->tasks_count) task_y = 0;
			}
		
			machine_a = current_solution->task_assignment[task_x];
			machine_b = current_solution->task_assignment[task_y];

			float swap_delta = 0.0;
			swap_delta -= get_etc_value(etc_matrix, machine_a, task_x); // Resto del ETC de x en a.
			swap_delta += get_etc_value(etc_matrix, machine_a, task_y); // Sumo el ETC de y en a.
			swap_delta -= get_etc_value(etc_matrix, machine_b, task_y); // Resto el ETC de y en b.
			swap_delta += get_etc_value(etc_matrix, machine_b, task_x); // Sumo el ETC de x en b.

			fprintf(stdout, "   GPU Result %d. Delta %f (%f).\n", 
				result.best_swaps[i], result.best_swaps_delta[i], swap_delta);
			fprintf(stdout, "   >> [GPU] Task %d in %d swaps with task %d in %d.\n", 
				result.taskx[i], 0, result.tasky[i], 0);
			fprintf(stdout, "   >> [CPU] Task %d in %d swaps with task %d in %d.\n", 
				task_x, machine_a, task_y, machine_b);
		}
	}
	// Debug ------------------------------------------------------------------------------------------

	// Timming -----------------------------------------------------
	timespec ts_finalize;
	timming_start(ts_finalize);
	// Timming -----------------------------------------------------

	// Libero la memoria del dispositivo.
	pals_gpu_rtask_finalize(&instance);
	
	// Timming -----------------------------------------------------
	timming_end("pals_gpu_randTask_finalize", ts_finalize);
	// Timming -----------------------------------------------------		
}

void pals_gpu_rmachine(struct params &input, struct matrix *etc_matrix, struct solution *current_solution) {
	// No implementado.
}

