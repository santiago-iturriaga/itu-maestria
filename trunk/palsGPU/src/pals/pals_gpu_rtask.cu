#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <limits.h>
#include <assert.h>

#include "../config.h"
#include "../utils.h"

#include "../random/cpu_rand.h"
#include "../random/RNG_rand48.h"

#include "pals_gpu_rtask.h"

#define PALS_RTASK_RANDS                6144*20

#define PALS_GPU_RTASK__THREADS         128

__global__ void pals_rtask_kernel(
    ushort loops_count, ushort machines_count, ushort tasks_count, float current_makespan,
    float *gpu_etc_matrix, ushort *gpu_task_assignment, float *gpu_machine_compute_time,
    int *gpu_random_numbers, int *gpu_best_movements, float *gpu_best_deltas)
{
    const unsigned int thread_idx = threadIdx.x;
    const unsigned int block_idx = blockIdx.x;
    const unsigned int block_dim = blockDim.x; // Cantidad de threads.

    const short mov_type = (short)(block_idx & 0x1);

    const unsigned int random1 = gpu_random_numbers[block_idx];
    const unsigned int random2 = gpu_random_numbers[block_idx + 1];

    __shared__ short block_operations[PALS_GPU_RTASK__THREADS];
    __shared__ ushort block_threads[PALS_GPU_RTASK__THREADS];
    __shared__ ushort block_loops[PALS_GPU_RTASK__THREADS];
    __shared__ float block_deltas[PALS_GPU_RTASK__THREADS];

    for (ushort loop = 0; loop < loops_count; loop++) {
        // Tipo de movimiento.
        if (mov_type == 0) { // Comparación a nivel de bit para saber si es par o impar.
            // Si es impar...
            // Movimiento SWAP.

            ushort task_x, task_y;
            ushort machine_a, machine_b;

            float machine_a_ct_old, machine_b_ct_old;
            float machine_a_ct_new, machine_b_ct_new;

            float delta;
            delta = 0.0;

            // ================= Obtengo las tareas sorteadas.
            task_x = (random1 + loop) % tasks_count;

            task_y = ((random2 >> 1) + (loop * block_dim)  + thread_idx) % (tasks_count - 1);
            if (task_y >= task_x) task_y++;

            // ================= Obtengo las máquinas a las que estan asignadas las tareas.
            machine_a = gpu_task_assignment[task_x]; // Máquina a.
            machine_b = gpu_task_assignment[task_y]; // Máquina b.

            if (machine_a != machine_b) {
                // Calculo el delta del swap sorteado.

                // Máquina 1.
                machine_a_ct_old = gpu_machine_compute_time[machine_a];

                machine_a_ct_new = machine_a_ct_old;
                machine_a_ct_new = machine_a_ct_new - gpu_etc_matrix[(machine_a * tasks_count) + task_x]; // Resto del ETC de x en a.
                machine_a_ct_new = machine_a_ct_new + gpu_etc_matrix[(machine_a * tasks_count) + task_y]; // Sumo el ETC de y en a.

                // Máquina 2.
                machine_b_ct_old = gpu_machine_compute_time[machine_b];

                machine_b_ct_new = machine_b_ct_old;
                machine_b_ct_new = machine_b_ct_new - gpu_etc_matrix[(machine_b * tasks_count) + task_y]; // Resto el ETC de y en b.
                machine_b_ct_new = machine_b_ct_new + gpu_etc_matrix[(machine_b * tasks_count) + task_x]; // Sumo el ETC de x en b.

                if ((machine_a_ct_new > current_makespan) || (machine_b_ct_new > current_makespan)) {
                    // Luego del movimiento aumenta el makespan. Intento desestimularlo lo más posible.
                    if (machine_a_ct_new > current_makespan) delta = delta + (machine_a_ct_new - current_makespan);
                    if (machine_b_ct_new > current_makespan) delta = delta + (machine_b_ct_new - current_makespan);
                } else if ((machine_a_ct_old+1 >= current_makespan) || (machine_b_ct_old+1 >= current_makespan)) {
                    // Antes del movimiento una las de máquinas definía el makespan. Estos son los mejores movimientos.

                    if (machine_a_ct_old+1 >= current_makespan) {
                        delta = delta + (machine_a_ct_new - machine_a_ct_old);
                    } else {
                        delta = delta + 1/(machine_a_ct_new - machine_a_ct_old);
                    }

                    if (machine_b_ct_old+1 >= current_makespan) {
                        delta = delta + (machine_b_ct_new - machine_b_ct_old);
                    } else {
                        delta = delta + 1/(machine_b_ct_new - machine_b_ct_old);
                    }
                } else {
                    // Ninguna de las máquinas intervenía en el makespan. Intento favorecer lo otros movimientos.
                    delta = delta + (machine_a_ct_new - machine_a_ct_old);
                    delta = delta + (machine_b_ct_new - machine_b_ct_old);
                    delta = 1 / delta;
                }
            }

            if ((loop == 0) || (block_deltas[thread_idx] > delta)) {
                block_operations[thread_idx] = PALS_GPU_RTASK_SWAP;
                block_threads[thread_idx] = (short)thread_idx;
                block_loops[thread_idx] = loop;
                block_deltas[thread_idx] = delta;
            }
        } else {
            // Si es par...
            // Movimiento MOVE.

            ushort task_x;
            ushort machine_a, machine_b;

            float machine_a_ct_old, machine_b_ct_old;
            float machine_a_ct_new, machine_b_ct_new;

            float delta;
            delta = 0.0;

            // ================= Obtengo la tarea sorteada, la máquina a la que esta asignada,
            // ================= y el compute time de la máquina.
            task_x = (random1 + loop) % tasks_count;
            machine_a = gpu_task_assignment[task_x]; // Máquina a.
            machine_a_ct_old = gpu_machine_compute_time[machine_a];

            // ================= Obtengo la máquina destino sorteada.
            machine_b = ((random2 >> 1) + (loop * block_dim) + thread_idx) % (machines_count - 1);
            if (machine_b >= machine_a) machine_b++;

            machine_b_ct_old = gpu_machine_compute_time[machine_b];

            // Calculo el delta del swap sorteado.
            machine_a_ct_new = machine_a_ct_old - gpu_etc_matrix[(machine_a * tasks_count) + task_x]; // Resto del ETC de x en a.
            machine_b_ct_new = machine_b_ct_old + gpu_etc_matrix[(machine_b * tasks_count) + task_x]; // Sumo el ETC de x en b.

            if (machine_b_ct_new > current_makespan) {
                // Luego del movimiento aumenta el makespan. Intento desestimularlo lo más posible.
                delta = delta + (machine_b_ct_new - current_makespan);
            } else if (machine_a_ct_old+1 >= current_makespan) {
                // Antes del movimiento una las de máquinas definía el makespan. Estos son los mejores movimientos.
                delta = delta + (machine_a_ct_new - machine_a_ct_old);
                delta = delta + 1/(machine_b_ct_new - machine_b_ct_old);
            } else {
                // Ninguna de las máquinas intervenía en el makespan. Intento favorecer lo otros movimientos.
                delta = delta + (machine_a_ct_new - machine_a_ct_old);
                delta = delta + (machine_b_ct_new - machine_b_ct_old);
                delta = 1 / delta;
            }

            if ((loop == 0) || (block_deltas[thread_idx] > delta)) {
                block_operations[thread_idx] = PALS_GPU_RTASK_MOVE;
                block_threads[thread_idx] = (short)thread_idx;
                block_loops[thread_idx] = loop;
                block_deltas[thread_idx] = delta;
            }
        }
    }

    __syncthreads();

    // Aplico reduce para quedarme con el mejor delta.
    int pos;
    for (int i = 1; i < block_dim; i *= 2) {
        pos = 2 * i * thread_idx;

        if (pos < block_dim) {
            if (block_deltas[pos] > block_deltas[pos + i]) {
                block_operations[pos] = block_operations[pos + i];
                block_loops[pos] = block_loops[pos + i];
                block_threads[pos] = block_threads[pos + i];
                block_deltas[pos] = block_deltas[pos + i];
            }
        }

        __syncthreads();
    }

    if (thread_idx == 0) {
        gpu_best_movements[block_idx * 3] = (int)block_operations[0]; // Best movement operation.
        gpu_best_movements[(block_idx * 3) + 1] = (int)block_threads[0]; // Best movement thread index.
        gpu_best_movements[(block_idx * 3) + 2] = (int)block_loops[0]; // Best movement loop index.
        gpu_best_deltas[block_idx] = block_deltas[0];  // Best movement delta.
    }
}

void pals_gpu_rtask_init(struct matrix *etc_matrix, struct solution *s,
    struct pals_gpu_rtask_instance &instance, struct pals_gpu_rtask_result &result) {

    // Asignación del paralelismo del algoritmo.
    instance.blocks = 32; //128;
    instance.threads = PALS_GPU_RTASK__THREADS;
    instance.loops = 16; //32; //32;

    // Cantidad total de movimientos a evaluar.
    instance.total_tasks = instance.blocks * instance.threads * instance.loops;

    // Cantidad de resultados retornados por iteración.
    instance.result_count = instance.blocks;

    if (DEBUG) {
        fprintf(stdout, "[INFO] Number of blocks (grid size)   : %d\n", instance.blocks);
        fprintf(stdout, "[INFO] Threads per block (block size) : %d\n", instance.threads);
        fprintf(stdout, "[INFO] Loops per thread               : %d\n", instance.loops);
        fprintf(stdout, "[INFO] Total tasks                    : %ld\n", instance.total_tasks);
        fprintf(stdout, "[INFO] Movements per iteration        : %d\n", instance.result_count);
    }

    // =========================================================================

    // Pedido de memoria en el dispositivo y copiado de datos.
    timespec ts_1;
    timming_start(ts_1);

    // Pido memoria para guardar el resultado.
    int best_movements_size = sizeof(int) * instance.blocks * 3;
    if (hipMalloc((void**)&(instance.gpu_best_movements), best_movements_size) != hipSuccess) {
        fprintf(stderr, "[ERROR] Solicitando memoria gpu_best_movements_size (%d bytes).\n", best_movements_size);
        exit(EXIT_FAILURE);
    }

    int best_deltas_size = sizeof(float) * instance.blocks;
    if (hipMalloc((void**)&(instance.gpu_best_deltas), best_deltas_size) != hipSuccess) {
        fprintf(stderr, "[ERROR] Solicitando memoria gpu_best_deltas (%d bytes).\n", best_deltas_size);
        exit(EXIT_FAILURE);
    }

    timming_end(".. gpu_best_movements", ts_1);

    // =========================================================================

    timespec ts_2;
    timming_start(ts_2);

    // Copio la matriz de ETC.
    int etc_matrix_size = sizeof(float) * etc_matrix->tasks_count * etc_matrix->machines_count;
    if (hipMalloc((void**)&(instance.gpu_etc_matrix), etc_matrix_size) != hipSuccess) {
        fprintf(stderr, "[ERROR] Solicitando memoria etc_matrix (%d bytes).\n", etc_matrix_size);
        exit(EXIT_FAILURE);
    }

    if (hipMemcpy(instance.gpu_etc_matrix, etc_matrix->data, etc_matrix_size, hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "[ERROR] Copiando etc_matrix al dispositivo (%d bytes).\n", etc_matrix_size);
        exit(EXIT_FAILURE);
    }

    timming_end(".. gpu_etc_matrix", ts_2);

    // =========================================================================

    timespec ts_3;
    timming_start(ts_3);

    // Copio la asignación de tareas a máquinas actuales.
    int task_assignment_size = sizeof(short) * etc_matrix->tasks_count;
    if (hipMalloc((void**)&(instance.gpu_task_assignment), task_assignment_size) != hipSuccess) {
        fprintf(stderr, "[ERROR] Solicitando memoria task_assignment (%d bytes).\n", task_assignment_size);
        exit(EXIT_FAILURE);
    }

    if (hipMemcpy(instance.gpu_task_assignment, s->task_assignment, task_assignment_size, hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "[ERROR] Copiando task_assignment al dispositivo (%d bytes).\n", task_assignment_size);
        exit(EXIT_FAILURE);
    }

    timming_end(".. gpu_task_assignment", ts_3);

    // =========================================================================

    timespec ts_4;
    timming_start(ts_4);

    // Copio el compute time de las máquinas en la solución actual.
    int machine_compute_time_size = sizeof(float) * etc_matrix->machines_count;
    if (hipMalloc((void**)&(instance.gpu_machine_compute_time), machine_compute_time_size) != hipSuccess) {
        fprintf(stderr, "[ERROR] Solicitando memoria machine_compute_time (%d bytes).\n", machine_compute_time_size);
        exit(EXIT_FAILURE);
    }

    if (hipMemcpy(instance.gpu_machine_compute_time, s->machine_compute_time, machine_compute_time_size, hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "[ERROR] Copiando machine_compute_time al dispositivo (%d bytes).\n", machine_compute_time_size);
        exit(EXIT_FAILURE);
    }

    timming_end(".. gpu_machine_compute_time", ts_4);

    // =========================================================================

    result.move_type = (short*)malloc(sizeof(short) * instance.result_count);
    result.origin = (ushort*)malloc(sizeof(short) * instance.result_count);
    result.destination = (ushort*)malloc(sizeof(short) * instance.result_count);
    result.delta = (float*)malloc(sizeof(float) * instance.result_count);
}

void pals_gpu_rtask_reinit(struct pals_gpu_rtask_instance &instance, struct pals_gpu_rtask_result &result) {
    // Cantidad total de movimientos a evaluar.
    instance.total_tasks = instance.blocks * instance.threads * instance.loops;

    // Cantidad de resultados retornados por iteración.
    instance.result_count = instance.blocks;

    if (DEBUG) {
        fprintf(stdout, "[INFO] Number of blocks (grid size)   : %d\n", instance.blocks);
        fprintf(stdout, "[INFO] Threads per block (block size) : %d\n", instance.threads);
        fprintf(stdout, "[INFO] Loops per thread               : %d\n", instance.loops);
        fprintf(stdout, "[INFO] Total tasks                    : %ld\n", instance.total_tasks);
        fprintf(stdout, "[INFO] Movements per iteration        : %d\n", instance.result_count);
    }

    // =========================================================================

    // Pedido de memoria en el dispositivo y copiado de datos.
    timespec ts_1;
    timming_start(ts_1);

    if (hipFree(instance.gpu_best_deltas) != hipSuccess) {
        fprintf(stderr, "[ERROR] Liberando la memoria solicitada para best_swaps.\n");
        exit(EXIT_FAILURE);
    }

    if (hipFree(instance.gpu_best_movements) != hipSuccess) {
        fprintf(stderr, "[ERROR] Liberando la memoria solicitada para best_movements.\n");
        exit(EXIT_FAILURE);
    }

    // Pido memoria para guardar el resultado.
    int best_movements_size = sizeof(int) * instance.blocks * 3;
    if (hipMalloc((void**)&(instance.gpu_best_movements), best_movements_size) != hipSuccess) {
        fprintf(stderr, "[ERROR] Solicitando memoria gpu_best_movements_size (%d bytes).\n", best_movements_size);
        exit(EXIT_FAILURE);
    }

    int best_deltas_size = sizeof(float) * instance.blocks;
    if (hipMalloc((void**)&(instance.gpu_best_deltas), best_deltas_size) != hipSuccess) {
        fprintf(stderr, "[ERROR] Solicitando memoria gpu_best_deltas (%d bytes).\n", best_deltas_size);
        exit(EXIT_FAILURE);
    }

    timming_end(".. gpu_best_movements", ts_1);

    // =========================================================================

    free(result.move_type);
    free(result.origin);
    free(result.destination);
    free(result.delta);

    result.move_type = (short*)malloc(sizeof(short) * instance.result_count);
    result.origin = (ushort*)malloc(sizeof(short) * instance.result_count);
    result.destination = (ushort*)malloc(sizeof(short) * instance.result_count);
    result.delta = (float*)malloc(sizeof(float) * instance.result_count);
}

void pals_gpu_rtask_finalize(struct pals_gpu_rtask_instance &instance) {
    if (hipFree(instance.gpu_etc_matrix) != hipSuccess) {
        fprintf(stderr, "[ERROR] Liberando la memoria solicitada para etc_matrix.\n");
        exit(EXIT_FAILURE);
    }

    if (hipFree(instance.gpu_task_assignment) != hipSuccess) {
        fprintf(stderr, "[ERROR] Liberando la memoria solicitada para task_assignment.\n");
        exit(EXIT_FAILURE);
    }

    if (hipFree(instance.gpu_machine_compute_time) != hipSuccess) {
        fprintf(stderr, "[ERROR] Liberando la memoria solicitada para machine_compute_time.\n");
        exit(EXIT_FAILURE);
    }

    if (hipFree(instance.gpu_best_deltas) != hipSuccess) {
        fprintf(stderr, "[ERROR] Liberando la memoria solicitada para best_swaps.\n");
        exit(EXIT_FAILURE);
    }

    if (hipFree(instance.gpu_best_movements) != hipSuccess) {
        fprintf(stderr, "[ERROR] Liberando la memoria solicitada para best_movements.\n");
        exit(EXIT_FAILURE);
    }
}

void pals_gpu_rtask_clean_result(struct pals_gpu_rtask_result &result) {
    free(result.move_type);
    free(result.origin);
    free(result.destination);
    free(result.delta);
}

void pals_gpu_rtask_wrapper(struct matrix *etc_matrix, struct solution *s,
    struct pals_gpu_rtask_instance &instance, int *gpu_random_numbers,
    struct pals_gpu_rtask_result &result) {

    // Timming -----------------------------------------------------
    timespec ts_pals_pre;
    timming_start(ts_pals_pre);
    // Timming -----------------------------------------------------

    // Timming -----------------------------------------------------
    timming_end(".. pals_gpu_rtask_pals_pre", ts_pals_pre);
    // Timming -----------------------------------------------------

    // ==============================================================================
    // Ejecución del algoritmo.
    // ==============================================================================

    // Timming -----------------------------------------------------
    timespec ts_pals;
    timming_start(ts_pals);
    // Timming -----------------------------------------------------

    dim3 grid(instance.blocks, 1, 1);
    dim3 threads(instance.threads, 1, 1);

    pals_rtask_kernel<<< grid, threads >>>(
        instance.loops,
        etc_matrix->machines_count,
        etc_matrix->tasks_count,
        s->makespan,
        instance.gpu_etc_matrix,
        instance.gpu_task_assignment,
        instance.gpu_machine_compute_time,
        gpu_random_numbers,
        instance.gpu_best_movements,
        instance.gpu_best_deltas);

    hipError_t e;
    e = hipGetLastError();
    if (e != hipSuccess) {
        fprintf(stderr, "[ERROR] Failure in kernel call.\n%s\n", hipGetErrorString(e));
        exit(EXIT_FAILURE);
    }

    // Pido el espacio de memoria para obtener los resultados desde la gpu.
    int *best_movements = (int*)malloc(sizeof(int) * instance.blocks * 3);
    float *best_deltas = (float*)malloc(sizeof(float) * instance.blocks);
    int *rands_nums = (int*)malloc(sizeof(int) * instance.blocks * 2);

    // Copio los mejores movimientos desde el dispositivo.
    if (hipMemcpyAsync(best_movements, instance.gpu_best_movements, sizeof(int) * instance.blocks * 3,
        hipMemcpyDeviceToHost, 0) != hipSuccess) {

        fprintf(stderr, "[ERROR] Copiando los mejores movimientos al host (best_swaps).\n");
        exit(EXIT_FAILURE);
    }

    if (hipMemcpyAsync(best_deltas, instance.gpu_best_deltas, sizeof(float) * instance.blocks,
        hipMemcpyDeviceToHost, 0) != hipSuccess) {

        fprintf(stderr, "[ERROR] Copiando los mejores movimientos al host (best_swaps_delta).\n");
        exit(EXIT_FAILURE);
    }

    if (hipMemcpyAsync(rands_nums, gpu_random_numbers, sizeof(int) * instance.blocks * 2,
        hipMemcpyDeviceToHost, 0) != hipSuccess) {

        fprintf(stderr, "[ERROR] Copiando al host los números aleatorios sorteados.\n");
        exit(EXIT_FAILURE);
    }

    hipDeviceSynchronize();

    // Timming -----------------------------------------------------
    timming_end(".. pals_gpu_rtask_pals", ts_pals);
    // Timming -----------------------------------------------------

    // =====================================================================
    // Se cargan los resultados a la respuesta.
    // (lo mejor sería usar la GPU para generar el resultado).
    // =====================================================================

    // Timming -----------------------------------------------------
    timespec ts_pals_post;
    timming_start(ts_pals_post);
    // Timming -----------------------------------------------------

    // Busco el block que encontró el mejor movimiento.
    int best_block_idx = 0;

    for (int i = 1; i < instance.blocks; i++) {
        if (best_deltas[i] < best_deltas[best_block_idx]) {
            best_block_idx = i;
        }

        /*
        if (DEBUG) {
            fprintf(stdout, ".. id=%d, eval=%f.\n", i, best_deltas[i]);
        }
        */
    }

    for (int i = 0; i < instance.blocks; i++) {
        // Calculo cuales fueron los elementos modificados en ese mejor movimiento.
        int block_idx = (i + best_block_idx) % instance.blocks;

        int movement_idx = block_idx * 3;
        int move_type = best_movements[movement_idx];
        int thread_idx = best_movements[movement_idx + 1];
        int loop_idx = best_movements[movement_idx + 2];

        float delta = best_deltas[block_idx];

        int random_idx = block_idx;
        int random1 = rands_nums[random_idx];
        int random2 = rands_nums[random_idx + 1];

        if (move_type == PALS_GPU_RTASK_SWAP) { // Movement type: SWAP
            ushort task_x = (ushort)((random1 + loop_idx) % etc_matrix->tasks_count);

            ushort task_y = (ushort)(((random2 >> 1) + (loop_idx * instance.threads) + thread_idx) % (etc_matrix->tasks_count - 1));
            if (task_y >= task_x) task_y++;

            result.move_type[i] = (short)move_type; // SWAP
            result.origin[i] = task_x;
            result.destination[i] = task_y;
            result.delta[i] = delta;

            // =======> DEBUG
            if (DEBUG) {
                ushort machine_a = s->task_assignment[task_x];
                ushort machine_b = s->task_assignment[task_y];

                fprintf(stdout, "[DEBUG] Task %d in %d swaps with task %d in %d. Delta %f.\n",
                    task_x, machine_a, task_y, machine_b, delta);
            }
            // <======= DEBUG
        } else if (move_type == PALS_GPU_RTASK_MOVE) { // Movement type: MOVE
            ushort task_x = (ushort)((random1 + loop_idx) % etc_matrix->tasks_count);
            ushort machine_a = s->task_assignment[task_x];

            ushort machine_b = (ushort)(((random2 >> 1) + (loop_idx * instance.threads) + thread_idx) % (etc_matrix->machines_count - 1));
            if (machine_b >= machine_a) machine_b++;

            result.move_type[i] = (short)move_type; // MOVE
            result.origin[i] = task_x;
            result.destination[i] = machine_b;
            result.delta[i] = delta;

            // =======> DEBUG
            if (DEBUG) {
                fprintf(stdout, "[DEBUG] Task %d in %d is moved to machine %d. Delta %f.\n",
                    task_x, machine_a, machine_b, delta);
            }
            // <======= DEBUG
        }
    }

    // Timming -----------------------------------------------------
    timming_end(".. pals_gpu_rtask_pals_post", ts_pals_post);
    // Timming -----------------------------------------------------
}

void pals_gpu_rtask_move(struct pals_gpu_rtask_instance &instance, ushort task, ushort to_machine) {
    if (hipMemcpy(&(instance.gpu_task_assignment[task]), &to_machine, sizeof(short), hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "[ERROR] Error moviendo la task %d a la máquina %d.\n", task, to_machine);
        exit(EXIT_FAILURE);
    }
}

void pals_gpu_rtask_update_machine(struct pals_gpu_rtask_instance &instance, ushort machine, float compute_time) {
    if (hipMemcpy(&(instance.gpu_machine_compute_time[machine]), &compute_time, sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "[ERROR] Error actualizando el compute time de la máquina %d.\n", machine);
        exit(EXIT_FAILURE);
    }
}

void pals_gpu_rtask(struct params &input, struct matrix *etc_matrix, struct solution *current_solution) {
    // ==============================================================================
    // PALS aleatorio por tarea.
    // ==============================================================================

    // Time stop condition -----------------------------------------
    timespec ts_stop_condition_start, ts_stop_condition_current;
    clock_gettime(CLOCK_REALTIME, &ts_stop_condition_start);

    // Timming -----------------------------------------------------
    timespec ts_init;
    timming_start(ts_init);
    // Timming -----------------------------------------------------

    struct pals_gpu_rtask_instance instance;
    struct pals_gpu_rtask_result result;

    // Inicializo la memoria en el dispositivo.
    pals_gpu_rtask_init(etc_matrix, current_solution, instance, result);

    if (DEBUG) {
        // Validación de la memoria del dispositivo.
        fprintf(stdout, ">> VALIDANDO MEMORIA GPU\n");

        ushort aux_task_assignment[etc_matrix->tasks_count];

        if (hipMemcpy(aux_task_assignment, instance.gpu_task_assignment, etc_matrix->tasks_count * sizeof(short),
            hipMemcpyDeviceToHost) != hipSuccess) {

            fprintf(stderr, "[ERROR] Copiando task_assignment al host (%ld bytes).\n", etc_matrix->tasks_count * sizeof(short));
            exit(EXIT_FAILURE);
        }

        for (ushort i = 0; i < etc_matrix->tasks_count; i++) {
            if (current_solution->task_assignment[i] != aux_task_assignment[i]) {
                fprintf(stdout, "[INFO] task assignment diff => task %d on host: %d, on device: %d\n",
                    i, current_solution->task_assignment[i], aux_task_assignment[i]);
            }
        }

        float aux_machine_compute_time[etc_matrix->machines_count];

        if (hipMemcpy(aux_machine_compute_time, instance.gpu_machine_compute_time, etc_matrix->machines_count * sizeof(float),
            hipMemcpyDeviceToHost) != hipSuccess) {

            fprintf(stderr, "[ERROR] Copiando machine_compute_time al host (%ld bytes).\n", etc_matrix->machines_count * sizeof(float));
            exit(EXIT_FAILURE);
        }

        for (ushort i = 0; i < etc_matrix->machines_count; i++) {
            if (current_solution->machine_compute_time[i] != aux_machine_compute_time[i]) {
                fprintf(stdout, "[INFO] machine CT diff => machine %d on host: %f, on device: %f\n",
                    i, current_solution->machine_compute_time[i], aux_machine_compute_time[i]);
            }
        }
    }

    // Timming -----------------------------------------------------
    timming_end(">> pals_gpu_rtask_init", ts_init);
    // Timming -----------------------------------------------------

    // ===========> DEBUG
    if (DEBUG) {
        validate_solution(etc_matrix, current_solution);
    }
    // <=========== DEBUG

    float makespan_inicial = current_solution->makespan;

    // Ejecuto GPUPALS.
    int seed = input.seed;

    RNG_rand48 r48;
    RNG_rand48_init(r48, PALS_RTASK_RANDS); // Debe ser múltiplo de 6144

    // Cantidad de números aleatorios por invocación.
    unsigned int rand_iter_size = instance.blocks * 2;
    uint current_rand_offset = 0;

    const short cant_iter_generadas = PALS_RTASK_RANDS / rand_iter_size;
    if (DEBUG) fprintf(stdout, "[INFO] Cantidad de iteraciones por generación de numeros aleatorios: %d.\n", cant_iter_generadas);

    char result_task_history[etc_matrix->tasks_count];
    char result_machine_history[etc_matrix->machines_count];

    short increase_depth;
    increase_depth = 0;

    ulong cantidad_swaps = 0;
    ulong cantidad_movs = 0;

    short convergence_flag;
    convergence_flag = 0;

    struct solution *best_solution = create_empty_solution(etc_matrix);
    clone_solution(etc_matrix, best_solution, current_solution);

    int best_solution_iter = -1;

    clock_gettime(CLOCK_REALTIME, &ts_stop_condition_current);

    int iter;
    /*for (iter = 0; (iter < PALS_COUNT) && (convergence_flag == 0)
        && (ts_stop_condition_current.tv_sec - ts_stop_condition_start.tv_sec) <= 5; iter++) {*/
    for (iter = 0; (iter < PALS_COUNT); iter++) {
        
        if (DEBUG) fprintf(stdout, "[INFO] Iteracion %d =====================\n", iter);

        // ==============================================================================
        // Sorteo de numeros aleatorios.
        // ==============================================================================

        timespec ts_rand;
        timming_start(ts_rand);

        if (iter % cant_iter_generadas == 0) {
        //if (current_rand_offset + rand_iter_size > PALS_RTASK_RANDS) {
            if (DEBUG) fprintf(stdout, "[INFO] Generando %d números aleatorios...\n", PALS_RTASK_RANDS);
            RNG_rand48_generate(r48, seed);

            current_rand_offset = 0;
        }

        timming_end(">> RNG_rand48", ts_rand);

        // Timming -----------------------------------------------------
        timespec ts_wrapper;
        timming_start(ts_wrapper);
        // Timming -----------------------------------------------------

        pals_gpu_rtask_wrapper(etc_matrix, current_solution, instance,
            &(r48.res[(iter % cant_iter_generadas) * rand_iter_size]), result);

        //pals_gpu_rtask_wrapper(etc_matrix, current_solution, instance,
        //  &(r48.res[current_rand_offset]), result);

        current_rand_offset += rand_iter_size;

        // Timming -----------------------------------------------------
        timming_end(">> pals_gpu_rtask_wrapper", ts_wrapper);
        // Timming -----------------------------------------------------

        // Timming -----------------------------------------------------
        timespec ts_post;
        timming_start(ts_post);
        // Timming -----------------------------------------------------

        // Aplico el mejor movimiento.
        memset(result_task_history, 0, etc_matrix->tasks_count); //for (int i = 0; i < etc_matrix->tasks_count; i++) result_task_history[i] = 0;
        memset(result_machine_history, 0, etc_matrix->machines_count); //for (int i = 0; i < etc_matrix->machines_count; i++) result_machine_history[i] = 0;

        ulong cantidad_swaps_iter, cantidad_movs_iter;
        cantidad_swaps_iter = 0;
        cantidad_movs_iter = 0;

        for (int result_idx = 0; result_idx < instance.result_count; result_idx++) {
            //if (DEBUG) fprintf(stdout, "[DEBUG] Movement %d, delta = %f.\n", result_idx, result.delta[result_idx]);

            if (result.delta[result_idx] < 0.0) { //|| (increase_depth < 50)) {
                if (result.move_type[result_idx] == PALS_GPU_RTASK_SWAP) {
                    ushort task_x = result.origin[result_idx];
                    ushort task_y = result.destination[result_idx];

                    ushort machine_a = current_solution->task_assignment[result.origin[result_idx]];
                    ushort machine_b = current_solution->task_assignment[result.destination[result_idx]];

                    /*if (DEBUG) fprintf(stdout, "        (swap) Task %d in %d swaps with task %d in %d. Delta %f.\n",
                        result.origin[result_idx], machine_a, result.destination[result_idx], machine_b, result.delta[result_idx]);*/

                    if ((result_task_history[task_x] == 0) && (result_task_history[task_y] == 0) &&
                        (result_machine_history[machine_a] == 0) && (result_machine_history[machine_b] == 0))   {

                        cantidad_swaps_iter++;

                        result_task_history[task_x] = 1;
                        result_task_history[task_y] = 1;
                        result_machine_history[machine_a] = 1;
                        result_machine_history[machine_b] = 1;

                        /*if (DEBUG) {
                            fprintf(stdout, ">> [pre-update]:\n");
                            fprintf(stdout, "   machine_a: %d, old_machine_a_ct: %f.\n", machine_a, current_solution->machine_compute_time[machine_a]);
                            fprintf(stdout, "   machine_b: %d, old_machine_b_ct: %f.\n", machine_b, current_solution->machine_compute_time[machine_b]);
                        }*/

                        // Actualizo la asignación de cada tarea en el host.
                        current_solution->task_assignment[task_x] = machine_b;
                        current_solution->task_assignment[task_y] = machine_a;

                        // Actualizo los compute time de cada máquina luego del move en el host.
                        current_solution->machine_compute_time[machine_a] =
                            current_solution->machine_compute_time[machine_a] +
                            get_etc_value(etc_matrix, machine_a, task_y) -
                            get_etc_value(etc_matrix, machine_a, task_x);

                        current_solution->machine_compute_time[machine_b] =
                            current_solution->machine_compute_time[machine_b] +
                            get_etc_value(etc_matrix, machine_b, task_x) -
                            get_etc_value(etc_matrix, machine_b, task_y);

                        // Actualizo la asignación de cada tarea en el dispositivo.
                        pals_gpu_rtask_move(instance, task_x, machine_b);
                        pals_gpu_rtask_move(instance, task_y, machine_a);
                        pals_gpu_rtask_update_machine(instance, machine_a, current_solution->machine_compute_time[machine_a]);
                        pals_gpu_rtask_update_machine(instance, machine_b, current_solution->machine_compute_time[machine_b]);

                        /*if (DEBUG) {
                            fprintf(stdout, ">> [update]:\n");
                            fprintf(stdout, "   task_x: %d, task_x_machine: %d.\n", task_x, machine_b);
                            fprintf(stdout, "   task_y: %d, task_y_machine: %d.\n", task_y, machine_a);
                            fprintf(stdout, "   machine_a: %d, machine_a_ct: %f.\n", machine_a, current_solution->machine_compute_time[machine_a]);
                            fprintf(stdout, "   machine_b: %d, machine_b_ct: %f.\n", machine_b, current_solution->machine_compute_time[machine_b]);
                            fprintf(stdout, "   old_makespan: %f.\n", current_solution->makespan);
                        }*/
                    } else {
                        //if (DEBUG) fprintf(stdout, "[DEBUG] Lo ignoro porque una tarea o máquina de este movimiento ya fue modificada.\n");
                    }
                } else if (result.move_type[result_idx] == PALS_GPU_RTASK_MOVE) {
                    ushort task_x = result.origin[result_idx];
                    ushort machine_a = current_solution->task_assignment[task_x];
                    ushort machine_b = result.destination[result_idx];

                    /*if (DEBUG) fprintf(stdout, "        (move) Task %d in %d is moved to machine %d. Delta %f.\n",
                        result.origin[result_idx], machine_a, result.destination[result_idx], result.delta[result_idx]);*/

                    if ((result_task_history[task_x] == 0) &&
                        (result_machine_history[machine_a] == 0) &&
                        (result_machine_history[machine_b] == 0))   {

                        cantidad_movs_iter++;

                        result_task_history[task_x] = 1;
                        result_machine_history[machine_a] = 1;
                        result_machine_history[machine_b] = 1;

                        /*if (DEBUG) {
                            fprintf(stdout, ">> [pre-update]:\n");
                            fprintf(stdout, "   machine_a: %d, old_machine_a_ct: %f.\n", machine_a, current_solution->machine_compute_time[machine_a]);
                            fprintf(stdout, "   machine_b: %d, old_machine_b_ct: %f.\n", machine_b, current_solution->machine_compute_time[machine_b]);
                        }*/

                        current_solution->task_assignment[task_x] = machine_b;

                        // Actualizo los compute time de cada máquina luego del move en el host.
                        current_solution->machine_compute_time[machine_a] =
                            current_solution->machine_compute_time[machine_a] -
                            get_etc_value(etc_matrix, machine_a, task_x);

                        current_solution->machine_compute_time[machine_b] =
                            current_solution->machine_compute_time[machine_b] +
                            get_etc_value(etc_matrix, machine_b, task_x);

                        // Actualizo la asignación de cada tarea en el dispositivo.
                        pals_gpu_rtask_move(instance, task_x, machine_b);
                        pals_gpu_rtask_update_machine(instance, machine_a, current_solution->machine_compute_time[machine_a]);
                        pals_gpu_rtask_update_machine(instance, machine_b, current_solution->machine_compute_time[machine_b]);

                        /*if (DEBUG) {
                            fprintf(stdout, ">> [update]:\n");
                            fprintf(stdout, "   task_x: %d, task_x_machine: %d.\n", task_x, machine_b);
                            fprintf(stdout, "   machine_a: %d, machine_a_ct: %f.\n", machine_a, current_solution->machine_compute_time[machine_a]);
                            fprintf(stdout, "   machine_b: %d, machine_b_ct: %f.\n", machine_b, current_solution->machine_compute_time[machine_b]);
                            fprintf(stdout, "   old_makespan: %f.\n", current_solution->makespan);
                        }*/
                    } else {
                        //if (DEBUG) fprintf(stdout, "[DEBUG] Lo ignoro porque una tarea o máquina de este movimiento ya fue modificada.\n");
                    }
                }
            }
        }

        if ((cantidad_movs_iter > 0) || (cantidad_swaps_iter > 0)) {
            // Actualiza el makespan de la solución.
            // Si cambio el makespan, busco el nuevo makespan.
            ushort machine = 0;
            current_solution->makespan = current_solution->machine_compute_time[0];

            for (ushort i = 1; i < etc_matrix->machines_count; i++) {
                if (current_solution->makespan < current_solution->machine_compute_time[i]) {
                    current_solution->makespan = current_solution->machine_compute_time[i];
                    machine = i;
                }
            }

            if (current_solution->makespan < best_solution->makespan) {
                clone_solution(etc_matrix, best_solution, current_solution);
                best_solution_iter = iter;
            }

            if (DEBUG) {
                fprintf(stdout, "   swaps performed  : %ld.\n", cantidad_swaps_iter);
                fprintf(stdout, "   movs performed   : %ld.\n", cantidad_movs_iter);
            }

            cantidad_swaps += cantidad_swaps_iter;
            cantidad_movs += cantidad_movs_iter;
        }

        if (best_solution_iter == iter) {
            increase_depth = 0;

            if (DEBUG) {
                fprintf(stdout, "   makespan improved: %f.\n", current_solution->makespan);
            }
        } else {
            increase_depth++;

            if (DEBUG) {
                fprintf(stdout, "   makespan unchanged: %f (%d).\n", current_solution->makespan, increase_depth);
            }
        }

        //if (increase_depth >= 500) {
		if (increase_depth >= (etc_matrix->machines_count * 1000)) {
            /*if (DEBUG) fprintf(stdout, "[DEBUG] Increase depth on iteration %d.\n", iter);

            instance.blocks += 8;

            if ((instance.blocks == 96) && (instance.loops == 32)) {
                instance.blocks = 32;
                instance.loops = 64;
            }

            fprintf(stdout, "[DEBUG] REINIT! Blocks = %d, Loops = %d.\n", instance.blocks, instance.loops);

            if ((instance.blocks == 96) && (instance.loops = 64)) {
                                convergence_flag = 1;
                                if (DEBUG) fprintf(stdout, "[DEBUG] Convergence detected! Iteration: %d.\n", iter);
            } else {
                pals_gpu_rtask_reinit(instance, result);
            }*/

            convergence_flag = 1;
            increase_depth = 0;
        }

        // Timming -----------------------------------------------------
        timming_end(">> pals_gpu_rtask_post", ts_post);
        // Timming -----------------------------------------------------

        // Nuevo seed.
        seed++;

        clock_gettime(CLOCK_REALTIME, &ts_stop_condition_current);
    }

    // Timming -----------------------------------------------------
    timespec ts_finalize;
    timming_start(ts_finalize);
    // Timming -----------------------------------------------------

    clone_solution(etc_matrix, current_solution, best_solution);

    if (DEBUG) {
        fprintf(stdout, "[DEBUG] Total iterations       : %d.\n", iter);
        fprintf(stdout, "[DEBUG] Iter. best sol. found  : %d.\n", best_solution_iter);

        fprintf(stdout, "[DEBUG] Total swaps performed  : %ld.\n", cantidad_swaps);
        fprintf(stdout, "[DEBUG] Total movs performed   : %ld.\n", cantidad_movs);

        fprintf(stdout, "[DEBUG] Current blocks count   : %d.\n", instance.blocks);
        fprintf(stdout, "[DEBUG] Current loops count    : %d.\n", instance.loops);
    }

    if (DEBUG) {
        // Validación de la memoria del dispositivo.
        fprintf(stdout, ">> VALIDANDO MEMORIA GPU\n");

        ushort aux_task_assignment[etc_matrix->tasks_count];

        if (hipMemcpy(aux_task_assignment, instance.gpu_task_assignment, etc_matrix->tasks_count * sizeof(short),
            hipMemcpyDeviceToHost) != hipSuccess) {

            fprintf(stderr, "[ERROR] Copiando task_assignment al host (%ld bytes).\n", etc_matrix->tasks_count * sizeof(short));
            exit(EXIT_FAILURE);
        }

        for (ushort i = 0; i < etc_matrix->tasks_count; i++) {
            if (current_solution->task_assignment[i] != aux_task_assignment[i]) {
                fprintf(stdout, "[INFO] task assignment diff => task %d on host: %d, on device: %d\n",
                    i, current_solution->task_assignment[i], aux_task_assignment[i]);
            }
        }

        float aux_machine_compute_time[etc_matrix->machines_count];

        if (hipMemcpy(aux_machine_compute_time, instance.gpu_machine_compute_time, etc_matrix->machines_count * sizeof(float),
            hipMemcpyDeviceToHost) != hipSuccess) {

            fprintf(stderr, "[ERROR] Copiando machine_compute_time al host (%ld bytes).\n", etc_matrix->machines_count * sizeof(float));
            exit(EXIT_FAILURE);
        }

        for (ushort i = 0; i < etc_matrix->machines_count; i++) {
            if (current_solution->machine_compute_time[i] != aux_machine_compute_time[i]) {
                fprintf(stdout, "[INFO] machine CT diff => machine %d on host: %f, on device: %f\n",
                    i, current_solution->machine_compute_time[i], aux_machine_compute_time[i]);
            }
        }
    }

    // Limpio el objeto resultado.
    pals_gpu_rtask_clean_result(result);

    // Libera la memoria del dispositivo con los números aleatorios.
    RNG_rand48_cleanup(r48);
    //hipFree(gpu_randoms);

    // Reconstruye el compute time de cada máquina.
    // NOTA: tengo que hacer esto cada tanto por errores acumulados en el redondeo.
    for (ushort i = 0; i < etc_matrix->machines_count; i++) {
        current_solution->machine_compute_time[i] = 0.0;
    }

    for (ushort i = 0; i < etc_matrix->tasks_count; i++) {
        ushort assigned_machine = current_solution->task_assignment[i];

        current_solution->machine_compute_time[assigned_machine] =
            current_solution->machine_compute_time[assigned_machine] +
            get_etc_value(etc_matrix, assigned_machine, i);
    }

    // Actualiza el makespan de la solución.
    current_solution->makespan = current_solution->machine_compute_time[0];
    for (ushort i = 1; i < etc_matrix->machines_count; i++) {
        if (current_solution->makespan < current_solution->machine_compute_time[i]) {
            current_solution->makespan = current_solution->machine_compute_time[i];
        }
    }

    // ===========> DEBUG
    if (DEBUG) {
        validate_solution(etc_matrix, current_solution);
    }
    // <=========== DEBUG

    if (DEBUG) {
        fprintf(stdout, "[DEBUG] Viejo makespan: %f\n", makespan_inicial);
        fprintf(stdout, "[DEBUG] Nuevo makespan: %f\n", current_solution->makespan);
    } else {
        if (!OUTPUT_SOLUTION) fprintf(stdout, "%f\n", current_solution->makespan);
        fprintf(stderr, "CANT_ITERACIONES|%d\n", iter);
        fprintf(stderr, "BEST_FOUND|%d\n", best_solution_iter);
                fprintf(stderr, "TOTAL_SWAPS|%ld\n", cantidad_swaps);
                fprintf(stderr, "TOTAL_MOVES|%ld\n", cantidad_movs);
    }

    // Libero la memoria del dispositivo.
    pals_gpu_rtask_finalize(instance);

    // Timming -----------------------------------------------------
    timming_end(">> pals_gpu_rtask_finalize", ts_finalize);
    // Timming -----------------------------------------------------
}
