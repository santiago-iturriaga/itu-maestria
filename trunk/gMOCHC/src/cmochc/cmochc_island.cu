#include "hip/hip_runtime.h"
#include <pthread.h>
#include <math.h>

#include "cmochc_island.h"

#include "../config.h"
#include "../solution.h"
#include "../load_params.h"
#include "../scenario.h"
#include "../etc_matrix.h"
#include "../energy_matrix.h"
#include "../utils.h"
#include "../basic/mct.h"
#include "../random/random.h"
#include "../archivers/aga.h"

//#define LOCAL_ITERATION_COUNT   1
#define LOCAL_ITERATION_COUNT   50
//#define LOCAL_ITERATION_COUNT   100
//#define LOCAL_ITERATION_COUNT   2000

#define BEST_SOLS_KEPT  3

#define CROSS_MAX_THRESHOLD_DIVISOR     4
/* Aprox. one cataclysm every 5 local iterations without change */
#define CROSS_THRESHOLD_STEP_DIVISOR    10

#define MAX_GLOBAL_POP  50

/* Only hux_custom */
#define CROSSOVER_FLIP_PROB     0.25

struct cmochc {
    struct params *input;
    struct scenario *current_scenario;
    struct etc_matrix *etc;
    struct energy_matrix *energy;

    /* Coleccion de esclavos */
    pthread_t *threads;

    /* Poblacion de cada esclavo */
    struct solution **population;
    int **sorted_population;
    
    /* Poblacion elite global mantenida por el master */
    struct solution *global_elite_pop;
    struct aga_state archiver;

    float **weights;
    int stopping_condition;

    /* Random generator de cada esclavo */
    RAND_STATE *rand_state;

    /* Sync */
    pthread_barrier_t sync_barrier;

    /* Statistics */
    int *generations_no_children_born;
    int *generations_no_children_inserted;
    int *generations_at_least_one_children_inserted;
    int *generations_improved_sols;
    int *generations_cataclysm_count;
};

struct cmochc_thread {
    /* Id del esclavo */
    int thread_id;

    struct cmochc *data;
};

/* Inicializa los hilos y las estructuras de datos */
void init(struct cmochc &instance, struct cmochc_thread **threads_data,
    struct params &input, struct scenario &current_scenario,
    struct etc_matrix &etc, struct energy_matrix &energy);

/* Obtiene los mejores elementos de cada población */
void gather(struct cmochc &instance);

/* Muestra el resultado de la ejecución */
void display_results(struct cmochc &instance);

/* Libera los recursos pedidos y finaliza la ejecución */
void finalize(struct cmochc &instance, struct cmochc_thread *threads);

/* Logica de los esclavos */
void* slave_thread(void *data);

void compute_cmochc_island(struct params &input, struct scenario &current_scenario,
    struct etc_matrix &etc, struct energy_matrix &energy) {

    // ==============================================================================
    // CPU CHC (islands)
    // ==============================================================================
    #if defined(DEBUG_0)
        fprintf(stderr, "[DEBUG] starting CPU CHC (islands)...\n");
    #endif

    // Timming -----------------------------------------------------
    TIMMING_START(ts_init);
    // Timming -----------------------------------------------------

    // Inicializo la memoria e inicializo los hilos de ejecucion.
    #if defined(DEBUG_1)
        fprintf(stderr, "[DEBUG] CPU CHC (islands): init\n");
    #endif

    struct cmochc instance;
    struct cmochc_thread *threads;
    init(instance, &threads, input, current_scenario, etc, energy);

    #if defined(DEBUG_1)
        fprintf(stderr, " [OK]\n");
    #endif

    // Timming -----------------------------------------------------
    TIMMING_END(">> cmochc_init", ts_init);
    // Timming -----------------------------------------------------

    int rc;
    for (int iteracion = 0; iteracion < input.max_iterations; iteracion++) {
        /* ************************************************** */
        /* Espero a que los esclavos terminen de evolucionar. */
        /* ************************************************** */
        rc = pthread_barrier_wait(&instance.sync_barrier);
        if(rc != 0 && rc != PTHREAD_BARRIER_SERIAL_THREAD)
        {
            printf("Could not wait on barrier\n");
            exit(EXIT_FAILURE);
        }

        /* Los esclavos copian sus mejores soluciones y comienzan a migrar */
       
        if (iteracion + 1 >= input.max_iterations) {
            /* Si esta es la úlitma iteracion, les aviso a los esclavos */
            instance.stopping_condition = 1;
        }
       
        /* ************************************************ */
        /* Espero que los esclavos terminen el intercambio. */
        /* ************************************************ */
        rc = pthread_barrier_wait(&instance.sync_barrier);
        if(rc != 0 && rc != PTHREAD_BARRIER_SERIAL_THREAD)
        {
            printf("Could not wait on barrier\n");
            exit(EXIT_FAILURE);
        }

        /* Incorporo las mejores soluciones al repositorio de soluciones */
        TIMMING_START(ts_gather);
        #if defined(DEBUG_3)
            fprintf(stderr, "[DEBUG] CPU CHC (islands): gather\n");
        #endif
        
        gather(instance);
        
        TIMMING_END(">> cmochc_gather", ts_gather);
    }
   
    /* Bloqueo la ejecucion hasta que terminen todos los hilos. */
    for(int i = 0; i < instance.input->thread_count; i++)
    {
        if(pthread_join(instance.threads[i], NULL))
        {
            fprintf(stderr, "Could not join thread %d\n", i);
            exit(EXIT_FAILURE);
        }
        else
        {
            #if defined(DEBUG_1)
                fprintf(stderr, "[DEBUG] thread %d <OK>\n", i);
            #endif
        }
    }

    /* Libero la memoria. */
    #if defined(DEBUG_1)
        fprintf(stderr, "[DEBUG] CPU CHC (islands): finalize\n");
    #endif

    display_results(instance);
    finalize(instance, threads);
}

void display_results(struct cmochc &instance) {
    /* Show solutions */
    
    #if defined(OUTPUT_SOLUTION)        
        fprintf(stdout, "%d\n", instance.archiver.population_count);
        for (int i = 0; i < MAX_GLOBAL_POP; i++) {
            if (instance.archiver.population[i].initialized == 1) {
                for (int task_id = 0; task_id < instance.etc->tasks_count; task_id++) {
                    fprintf(stdout, "%d\n", instance.archiver.population[i].task_assignment[task_id]);
                }
            }
        }
    #endif

    fprintf(stderr, "[DEBUG] Elite archive solutions [makespan energy]\n");

    for (int i = 0; i < MAX_GLOBAL_POP; i++) {
        if (instance.archiver.population[i].initialized == 1) {
            fprintf(stderr, "%f %f\n",
                instance.archiver.population[i].makespan, 
                instance.archiver.population[i].energy_consumption);
        }
    }
    fprintf(stderr, "> total solutions: %d\n", instance.archiver.population_count);

    #ifdef DEBUG_1
        fprintf(stderr, "[INFO] == Statistics ==========================================\n");
        for (int t = 0; t < instance.input->thread_count; t++)
            fprintf(stderr, "       [thread %d] NO CHILDREN BORN COUNT     : %d\n", t, instance.generations_no_children_born[t]);
        for (int t = 0; t < instance.input->thread_count; t++)
            fprintf(stderr, "       [thread %d] NO CHILDREN INSERTED COUNT : %d\n", t, instance.generations_no_children_inserted[t]);
        for (int t = 0; t < instance.input->thread_count; t++)
            fprintf(stderr, "       [thread %d] AT LEAST ONE CHILDREN INS. : %d\n", t, instance.generations_at_least_one_children_inserted[t]);
        for (int t = 0; t < instance.input->thread_count; t++)
            fprintf(stderr, "       [thread %d] IMPROVED SOLUTIONS COUNT   : %d\n", t, instance.generations_improved_sols[t]);
        for (int t = 0; t < instance.input->thread_count; t++)
            fprintf(stderr, "       [thread %d] CATACLYSM COUNT            : %d\n", t, instance.generations_cataclysm_count[t]);
        fprintf(stderr, "[INFO] ========================================================\n");
    #endif
}

/* Inicializa los hilos y las estructuras de datos */
void init(struct cmochc &instance, struct cmochc_thread **threads_data,
    struct params &input, struct scenario &current_scenario,
    struct etc_matrix &etc, struct energy_matrix &energy) {

    fprintf(stderr, "[INFO] == Global configuration constants ======================\n");
    fprintf(stderr, "       LOCAL ITERATION_COUNT        : %d\n", LOCAL_ITERATION_COUNT);
    fprintf(stderr, "       CROSS_MAX_THRESHOLD_DIVISOR  : %d\n", CROSS_MAX_THRESHOLD_DIVISOR);
    fprintf(stderr, "       CROSS_THRESHOLD_STEP_DIVISOR : %d\n", CROSS_THRESHOLD_STEP_DIVISOR);
    fprintf(stderr, "[INFO] ========================================================\n");

    /* Estado relacionado con el problema. */
    instance.input = &input;
    instance.current_scenario = &current_scenario;
    instance.etc = &etc;
    instance.energy = &energy;
    instance.stopping_condition = 0;

    /* Estado del generador aleatorio. */
    instance.rand_state = (RAND_STATE*)(malloc(sizeof(RAND_STATE) * input.thread_count));

    /* Weights */
    instance.weights = (float**)(malloc(sizeof(float*) * input.thread_count));

    /* Statistics */
    instance.generations_no_children_born = (int*)(malloc(sizeof(int) * input.thread_count));
    instance.generations_no_children_inserted = (int*)(malloc(sizeof(int) * input.thread_count));
    instance.generations_at_least_one_children_inserted = (int*)(malloc(sizeof(int) * input.thread_count));
    instance.generations_improved_sols = (int*)(malloc(sizeof(int) * input.thread_count));
    instance.generations_cataclysm_count = (int*)(malloc(sizeof(int) * input.thread_count));

    /* Estado de la población de cada hilo. */
    instance.population = (struct solution**)(malloc(sizeof(struct solution*) * input.thread_count));
    instance.sorted_population = (int**)(malloc(sizeof(int*) * input.thread_count));

    /* Sync */
    if (pthread_barrier_init(&(instance.sync_barrier), NULL, input.thread_count + 1))
    {
        fprintf(stderr, "[ERROR] could not create a sync barrier.\n");
        exit(EXIT_FAILURE);
    }

    /* Inicializo los hilos */
    instance.threads = (pthread_t*)malloc(sizeof(pthread_t) * input.thread_count);
    *threads_data = (struct cmochc_thread*)malloc(sizeof(struct cmochc_thread) * input.thread_count);

    for (int i = 0; i < input.thread_count; i++)
    {
        pthread_t *t;
        t = &(instance.threads[i]);

        struct cmochc_thread *t_data;
        t_data = &((*threads_data)[i]);
        t_data->thread_id = i;
        t_data->data = &instance;

        if (pthread_create(t, NULL, slave_thread, (void*) t_data))
        {
            fprintf(stderr, "[ERROR] could not create slave thread %d\n", i);
            exit(EXIT_FAILURE);
        }
    }

    /* Estado de la población elite global */
    instance.global_elite_pop = (struct solution*)(malloc(sizeof(struct solution) * (input.thread_count * BEST_SOLS_KEPT)));
    
    for (int i = 0; i < (input.thread_count * BEST_SOLS_KEPT); i++) {
        create_empty_solution(&instance.global_elite_pop[i], &current_scenario, &etc, &energy);
    }
    
    /* Inicializo el archivador */
    archivers_aga_init(&instance.archiver, MAX_GLOBAL_POP, instance.global_elite_pop, (input.thread_count * BEST_SOLS_KEPT));
}

/* Obtiene los mejores elementos de cada población */
void gather(struct cmochc &instance) {
    #ifdef DEBUG_3
        fprintf(stderr, "[DEBUG] Gathering...\n");
        fprintf(stderr, "[DEBUG] Current iteration elite solutions:\n");
        
        int cantidad = 0;
        for (int i = 0; i < instance.archiver.new_solutions_size; i++) {
            if (instance.archiver.new_solutions[i].initialized == 1) cantidad++;
            
            fprintf(stderr, "> %d state=%d makespan=%f energy=%f\n", 
                i, instance.archiver.new_solutions[i].initialized,
                instance.archiver.new_solutions[i].makespan,
                instance.archiver.new_solutions[i].energy_consumption);
        }
        
        ASSERT(cantidad > 0);
    #endif
    
    int new_solutions;
    new_solutions = archivers_aga(&instance.archiver);

    #ifdef DEBUG_3
        fprintf(stderr, "[DEBUG] Total solutions gathered      = %d\n", new_solutions);
        fprintf(stderr, "[DEBUG] Current solutions in archiver = %d\n", instance.archiver.population_count);
    #endif
}

/* Libera los recursos pedidos y finaliza la ejecución */
void finalize(struct cmochc &instance, struct cmochc_thread *threads) {
    archivers_aga_free(&instance.archiver);
    pthread_barrier_destroy(&(instance.sync_barrier));
    
    free(instance.generations_cataclysm_count);
    free(instance.generations_improved_sols);
    free(instance.generations_no_children_born);
    free(instance.generations_no_children_inserted);
    free(instance.generations_at_least_one_children_inserted);
    free(instance.weights);
    free(instance.population);
    free(instance.sorted_population);
    free(instance.global_elite_pop);
    free(instance.rand_state);
    free(instance.threads);

    free(threads);
}

inline int distance(struct solution *s1, struct solution *s2) {
    int distance = 0;

    for (int i = 0; i < s1->etc->tasks_count; i++) {
        if (s1->task_assignment[i] != s2->task_assignment[i]) distance++;
    }

    ASSERT(distance >= 0)
    ASSERT(distance < s1->etc->tasks_count)

    return distance;
}

inline void hux_custom(RAND_STATE &rand_state,
    struct solution *p1, struct solution *p2,
    struct solution *c1, struct solution *c2) {

    double random;
    for (int task_index = 0; task_index < p1->etc->tasks_count; task_index++) {
        random = RAND_GENERATE(rand_state);

        if (random <= CROSSOVER_FLIP_PROB) {
            /* Si la máscara vale 1 copio las asignaciones cruzadas de la tarea */
            c1->task_assignment[task_index] = p2->task_assignment[task_index];
            c2->task_assignment[task_index] = p1->task_assignment[task_index];
        } else {
            /* Si la máscara vale 0 copio las asignaciones derecho de la tarea */
            c1->task_assignment[task_index] = p1->task_assignment[task_index];
            c2->task_assignment[task_index] = p2->task_assignment[task_index];
        }
    }

    refresh_solution(c1);
    refresh_solution(c2);
}

inline void hux(RAND_STATE &rand_state,
    struct solution *p1, struct solution *p2,
    struct solution *c1, struct solution *c2) {

    double random;
    int current_task_index = 0;

    while (current_task_index < p1->etc->tasks_count) {
        random = RAND_GENERATE(rand_state);

        int mask = 0x0;
        int mask_size = 256; // 8-bit mask
        float base_step = 1.0/(double)mask_size;
        float base = base_step;

        while (random > base) {
            base += base_step;
            mask += 0x1;
        }

        int mask_index = 0x1;
        while ((mask_index < mask_size) && (current_task_index < p1->etc->tasks_count)) {
            if ((mask & 0x1) == 1) {
                random = RAND_GENERATE(rand_state);
                
                if (random < 0.1) {
                    // Si la máscara vale 1 copio las asignaciones cruzadas de la tarea
                    c1->task_assignment[current_task_index] = p2->task_assignment[current_task_index];
                    c2->task_assignment[current_task_index] = p1->task_assignment[current_task_index];
                } else {
                    // Si la máscara vale 0 copio las asignaciones derecho de la tarea
                    c1->task_assignment[current_task_index] = p1->task_assignment[current_task_index];
                    c2->task_assignment[current_task_index] = p2->task_assignment[current_task_index];
                }
            } else {
                // Si la máscara vale 0 copio las asignaciones derecho de la tarea
                c1->task_assignment[current_task_index] = p1->task_assignment[current_task_index];
                c2->task_assignment[current_task_index] = p2->task_assignment[current_task_index];
            }

            // Desplazo la máscara hacia la derecha
            mask = mask >> 1;
            mask_index = mask_index << 1;
            current_task_index++;
        }
    }

    c1->initialized = 1;
    c2->initialized = 1;

    refresh_solution(c1);
    refresh_solution(c2);
}

inline void mutate(RAND_STATE &rand_state, struct solution *seed, struct solution *mutation) {
    int current_task_index = 0;
    int tasks_count = seed->etc->tasks_count;
    int machines_count = seed->etc->machines_count;

    while (current_task_index < tasks_count) {
        double random;
        random = RAND_GENERATE(rand_state);

        int mask = 0x0;
        int mask_size = 256; // 8-bit mask
        float base_step = 1.0/(double)mask_size;
        float base = base_step;

        while (random > base) {
            base += base_step;
            mask += 0x1;
        }

        int destination_machine;
        int mask_index = 0x1;
        while ((mask_index < mask_size) && (current_task_index < tasks_count)) {
            if ((mask & 0x1) == 1) {
                random = RAND_GENERATE(rand_state);
                
                if (random < 0.1) {
                    random = RAND_GENERATE(rand_state);
                    destination_machine = (int)(floor(machines_count * random));
                    
                    ASSERT(destination_machine >= 0)
                    ASSERT(destination_machine < machines_count)
                    
                    // Si la máscara vale 1 copio reubico aleariamente la tarea
                    mutation->task_assignment[current_task_index] = destination_machine;
                    
                    //#ifdef DEBUG_3
                    //    fprintf(stderr, "task=%d>>machine=%d, ", current_task_index, destination_machine);
                    //#endif
                }
            } else {
                // Si la máscara vale 0 copio las asignaciones derecho de la tarea
                mutation->task_assignment[current_task_index] = seed->task_assignment[current_task_index];
            }

            // Desplazo la máscara hacia la derecha
            mask = mask >> 1;
            mask_index = mask_index << 1;
            current_task_index++;
        }
    }

    refresh_solution(mutation);
}

inline float fitness(struct solution *population, float *fitness_population, float *weights, 
    float makespan_norm_value, float energy_norm_value, int index) {
        
    if (isnan(fitness_population[index])) {
        fitness_population[index] = ((population[index].makespan/makespan_norm_value) * weights[0]) +
            ((population[index].energy_consumption/energy_norm_value) * weights[1]);
    }

    return fitness_population[index];
}

inline void merge_sort(struct solution *population, float *weights, 
    float makespan_norm_value, float energy_norm_value, int *sorted_population, 
    float *fitness_population, int population_size);

/* Logica de los esclavos */
void* slave_thread(void *data) {
    struct cmochc_thread *t_data = (struct cmochc_thread*)data;
    struct cmochc *instance = t_data->data;

    int thread_id = t_data->thread_id;

    struct params *input = instance->input;
    struct scenario *current_scenario = instance->current_scenario;
    struct etc_matrix *etc = instance->etc;
    struct energy_matrix *energy = instance->energy;

    RAND_STATE *rand_state = instance->rand_state;

    int *generations_no_children_born = instance->generations_no_children_born;
    int *generations_no_children_inserted = instance->generations_no_children_inserted;
    int *generations_at_least_one_children_inserted = instance->generations_at_least_one_children_inserted;
    int *generations_improved_sols = instance->generations_improved_sols;
    int *generations_cataclysm_count = instance->generations_cataclysm_count;

    float makespan_norm_value, energy_norm_value;

    // ================================================================
    // Inicializo el thread.
    // ================================================================

    generations_no_children_born[thread_id] = 0;
    generations_no_children_inserted[thread_id] = 0;
    generations_at_least_one_children_inserted[thread_id] = 0;
    generations_improved_sols[thread_id] = 0;
    generations_cataclysm_count[thread_id] = 0;

    /* Inicialización del estado del generador aleatorio */
    RAND_INIT(thread_id,rand_state[thread_id]);
    double random;

    /* Inicializo el peso asignado a este thread */
    instance->weights[thread_id] = (float*)(malloc(sizeof(float) * 2));
    float *weights = instance->weights[thread_id];

    float thread_weight_step = 0.0;
    if (input->thread_count > 1) {
        thread_weight_step = 1.0 / (float)(input->thread_count-1);
        
        weights[0] = (float)thread_id * thread_weight_step;
        weights[1] = 1 - weights[0];
    } else {
        weights[0] = 0.5;
        weights[1] = 0.5;
    }

    #ifdef DEBUG_1
        fprintf(stderr, "[DEBUG] Thread %d, weight (%f,%f)\n", thread_id, weights[0], weights[1]);
    #endif

    ASSERT(weights[0] >= 0)
    ASSERT(weights[0] <= 1)
    ASSERT(weights[1] >= 0)
    ASSERT(weights[1] <= 1)
    
    /* Busco los threads mas cercanos */
    int *n_closest_threads = (int*)malloc(sizeof(int) * input->thread_count);
    for (int n = 0; n < input->thread_count; n++) n_closest_threads[n] = -1;
    
    float current_distance = thread_weight_step;
    float upper_bound, lower_bound;
    int upper_neigh, lower_neigh;
    int next_neigh = 0;
    
    if (current_distance > 0) {
        while ((current_distance <= 1.0) && (next_neigh < input->thread_count)) {
            upper_bound = weights[0] + current_distance;
            if ((upper_bound >= 0.0)&&(upper_bound <= 1.0)) {
                upper_neigh = upper_bound / thread_weight_step;
                n_closest_threads[next_neigh] = upper_neigh;
                next_neigh++;
            }

            lower_bound = weights[0] - current_distance;
            if ((lower_bound >= 0.0)&&(lower_bound <= 1.0)&&(next_neigh < input->thread_count)) {
                lower_neigh = lower_bound / thread_weight_step;
                n_closest_threads[next_neigh] = lower_neigh;
                next_neigh++;
            }
            
            current_distance += thread_weight_step;
        }
    }
    
    #ifdef DEBUG_1
        //if (thread_id == 1) {
            fprintf(stderr, "[DEBUG] Thread %d, closest neighbours:\n", thread_id);
            float w0,w1;
            for (int n = 0; n < input->thread_count; n++) {
                w0 = (float)n_closest_threads[n] * thread_weight_step;
                w1 = 1 - w0;
                
                fprintf(stderr, "<%d> %d (%f,%f)\n", thread_id, n_closest_threads[n], w0, w1);
            }
            fprintf(stderr, "\n");
        //}
    #endif

    /* Inicializo la población de padres y limpio la de hijos */
    int max_pop_sols = 2 * input->population_size;

    /* Poblacion de cada esclavo */
    instance->population[thread_id] = (struct solution*)(malloc(sizeof(struct solution) * max_pop_sols));
    struct solution *population = instance->population[thread_id];

    instance->sorted_population[thread_id] = (int*)(malloc(sizeof(int) * max_pop_sols));
    int *sorted_population = instance->sorted_population[thread_id];

    float *fitness_population;
    fitness_population = (float*)(malloc(sizeof(float) * max_pop_sols));

    for (int i = 0; i < max_pop_sols; i++) {
        // Random init.
        create_empty_solution(&(population[i]),current_scenario,etc,energy);

        if (i > 0) {
            random = RAND_GENERATE(rand_state[thread_id]);
            int starting_pos;
            starting_pos = (int)(floor(etc->tasks_count * random));

            #ifdef DEBUG_3
                fprintf(stderr, "[DEBUG] Thread %d, inicializando solution %d, starting %d, direction %d...\n",
                    thread_id, i, starting_pos, i & 0x1);
            #endif

            compute_mct_random(&(population[i]), starting_pos, i & 0x1);
        } else {
            compute_mct(&(population[i]));
            
            makespan_norm_value = population[i].makespan;
            energy_norm_value = population[i].energy_consumption;
            
            #ifdef DEBUG_3
                fprintf(stderr, "[DEBUG] Thread %d, makespan_norm_value=%f energy_norm_value=%f\n",
                    thread_id, makespan_norm_value, energy_norm_value);
            #endif
        }
        
        sorted_population[i] = i;
        fitness_population[i] = NAN;
        fitness(population, fitness_population, weights, 
            makespan_norm_value, energy_norm_value, i);

        #ifdef DEBUG_3
            fprintf(stderr, "[DEBUG] Thread %d, inicializado solution %d, fitness %f\n",
                thread_id, i, fitness(population, fitness_population, weights, 
                    makespan_norm_value, energy_norm_value, i));
        #endif
    }

    // ================================================================
    // Main iteration
    // ================================================================
    int next_avail_children;
    int max_children = input->population_size / 2;

    int max_distance = etc->tasks_count;

    int threshold_max = max_distance / CROSS_MAX_THRESHOLD_DIVISOR;
    int threshold_step = threshold_max / CROSS_THRESHOLD_STEP_DIVISOR;
    if (threshold_step == 0) threshold_step = 1;
    int threshold = threshold_max;

    #ifdef DEBUG_1
        fprintf(stderr, "[DEBUG] Threshold Max %d.\n", threshold_max);
        fprintf(stderr, "[DEBUG] Threshold Step %d.\n", threshold_step);
    #endif

    int rc;

    while (instance->stopping_condition == 0) {
        for (int iteracion = 0; iteracion < LOCAL_ITERATION_COUNT; iteracion++) {
            #ifdef DEBUG_3
                fprintf(stderr, "[DEBUG] Iteration %d.\n", iteracion);
            #endif

            // =======================================================
            // Mating
            // =======================================================
            next_avail_children = input->population_size;

            float d;
            int p1_idx, p2_idx;
            int p1_rand, p2_rand;
            int c1_idx, c2_idx;
            for (int child = 0; child < max_children; child++) {
                if (next_avail_children + 1 < max_pop_sols) {
                    // Padre aleatorio 1
                    random = RAND_GENERATE(rand_state[thread_id]);
                    p1_rand = (int)(floor(input->population_size * random));
                    p1_idx = sorted_population[p1_rand];

                    // Padre aleatorio 2
                    random = RAND_GENERATE(rand_state[thread_id]);
                    p2_rand = (int)(floor((input->population_size - 1) * random));
                    if (p2_rand >= p1_rand) p2_rand++;
                    p2_idx = sorted_population[p2_rand];

                    /*
                    #ifdef DEBUG_3
                        fprintf(stderr, "[DEBUG] Selected parents %d(%d) and %d(%d) [%d]\n", p1_idx, p1_rand, p2_idx, p2_rand, input->population_size);
                    #endif
                    * */

                    // Chequeo la distancia entre padres
                    d = distance(&population[p1_idx],&population[p2_idx]);

                    if (d > threshold) {
                        // Aplico HUX y creo dos hijos
                        c1_idx = sorted_population[next_avail_children];
                        c2_idx = sorted_population[next_avail_children+1];

                        hux(rand_state[thread_id],
                            &population[p1_idx],&population[p2_idx],
                            &population[c1_idx],&population[c2_idx]);

                        fitness_population[c1_idx] = NAN;
                        fitness_population[c2_idx] = NAN;
            
                        fitness(population, fitness_population, weights, 
                            makespan_norm_value, energy_norm_value, c1_idx);
                        fitness(population, fitness_population, weights, 
                            makespan_norm_value, energy_norm_value, c2_idx);

                        next_avail_children += 2;
                    }
                }
            }

            if (next_avail_children > input->population_size) {
                #ifdef DEBUG_3
                    fprintf(stderr, "[DEBUG] %d children born.\n", next_avail_children - input->population_size);
                #endif

                // =======================================================
                // Sort parent+children population
                // =======================================================

                float best_parent;
                best_parent = fitness(population, fitness_population, weights, 
                    makespan_norm_value, energy_norm_value, sorted_population[0]);

                float worst_parent;
                worst_parent = fitness(population, fitness_population, weights, 
                    makespan_norm_value, energy_norm_value, sorted_population[input->population_size-1]);

                merge_sort(population, weights, makespan_norm_value, energy_norm_value, 
                    sorted_population, fitness_population, max_pop_sols);

                #ifdef DEBUG_3
                    fprintf(stderr, "[DEBUG] Post-sorted population\n");
                    fprintf(stderr, "parents> ");
                    for (int i = 0; i < input->population_size; i++) {
                        fprintf(stderr, "%d(%f)<%d>  ", sorted_population[i], fitness_population[sorted_population[i]], population[sorted_population[i]].initialized);
                    }
                    fprintf(stderr, "\n");
                    fprintf(stderr, "childs > ");
                    for (int i = input->population_size; i < max_pop_sols; i++) {
                        fprintf(stderr, "%d(%f)<%d>  ", sorted_population[i], fitness_population[sorted_population[i]], population[sorted_population[i]].initialized);
                    }
                    fprintf(stderr, "\n");
                #endif

                if (worst_parent > fitness(population, fitness_population, weights, 
                    makespan_norm_value, energy_norm_value, sorted_population[input->population_size-1])) {
                        
                    #ifdef DEBUG_1
                        generations_at_least_one_children_inserted[thread_id]++;
                    #endif

                    #ifdef DEBUG_3
                        fprintf(stderr, "[DEBUG] At least one children inserted.\n");
                    #endif
                } else {
                    #ifdef DEBUG_1
                        generations_no_children_inserted[thread_id]++;
                    #endif

                    threshold -= threshold_step;

                    #ifdef DEBUG_3
                        fprintf(stderr, "[DEBUG] No children inserted into the population.\n");
                    #endif
                }
                
                if (best_parent > fitness(population, fitness_population, weights, 
                    makespan_norm_value, energy_norm_value, sorted_population[0])) {
                        
                    #ifdef DEBUG_1
                        generations_improved_sols[thread_id]++;
                    #endif

                    #ifdef DEBUG_3
                        fprintf(stderr, "[DEBUG] Current best solution was improved (from %f to %f)!\n",
                            best_parent, fitness(population, fitness_population, weights, 
                                makespan_norm_value, energy_norm_value, sorted_population[0]));
                    #endif
                }
            } else {
                #ifdef DEBUG_1
                    generations_no_children_born[thread_id]++;
                #endif

                threshold -= threshold_step;

                #ifdef DEBUG_3
                    fprintf(stderr, "[DEBUG] No children born.\n");
                #endif
            }

            #ifdef DEBUG_3
                fprintf(stderr, "[DEBUG] Threshold %d\n", threshold);
            #endif

            if (threshold < 0) {
                threshold = threshold_max;

                // =======================================================
                // Cataclysm
                // =======================================================
                
                #ifdef DEBUG_1
                    generations_cataclysm_count[thread_id]++;
                #endif

                #ifdef DEBUG_3
                    fprintf(stderr, "[DEBUG] Cataclysm %d!\n", generations_cataclysm_count[thread_id]);
                #endif
               
                int solution_seed;
                for (int i = BEST_SOLS_KEPT; i < max_pop_sols; i++) { /* No muto la mejor solución */
                    if (population[sorted_population[i]].initialized == 1) {
                        solution_seed = RAND_GENERATE(rand_state[thread_id]) * BEST_SOLS_KEPT;
                        mutate(rand_state[thread_id], &population[sorted_population[solution_seed]], &population[sorted_population[i]]);
                        
                        fitness_population[sorted_population[i]] = NAN;
                        fitness(population, fitness_population, weights, makespan_norm_value, energy_norm_value, sorted_population[i]);
                    }
                }
                #ifdef DEBUG_3
                    fprintf(stderr, "\n");
                #endif
                
                /* Re-sort de population */
                merge_sort(population, weights, makespan_norm_value, energy_norm_value, 
                    sorted_population, fitness_population, max_pop_sols);

                #ifdef DEBUG_3
                    fprintf(stderr, "[DEBUG] Post-mutate population\n");
                    fprintf(stderr, "parents> ");
                    for (int i = 0; i < input->population_size; i++) {
                        fprintf(stderr, "%d(%f)<%d>  ", sorted_population[i], fitness_population[sorted_population[i]], population[sorted_population[i]].initialized);
                    }
                    fprintf(stderr, "\n");
                    fprintf(stderr, "childs > ");
                    for (int i = input->population_size; i < max_pop_sols; i++) {
                        fprintf(stderr, "%d(%f)<%d>  ", sorted_population[i], fitness_population[sorted_population[i]], population[sorted_population[i]].initialized);
                    }
                    fprintf(stderr, "\n");
                #endif
            }
        }
                
        /* ***************************************** */
        /* Espero a que los demas esclavos terminen. */
        /* ***************************************** */
        rc = pthread_barrier_wait(&instance->sync_barrier);
        if(rc != 0 && rc != PTHREAD_BARRIER_SERIAL_THREAD)
        {
            printf("Could not wait on barrier\n");
            exit(EXIT_FAILURE);
        }
        
        /* Copio mi mejor solución a la población temp principal */
        for (int i = 0; i < BEST_SOLS_KEPT; i++) {
            #ifdef DEBUG_3
                fprintf(stderr, "[DEBUG] Thread %d, copiando mejor sol. %d en pos %d a para gather\n", 
                    thread_id, i, sorted_population[i]);
            #endif
            clone_solution(&instance->global_elite_pop[thread_id * BEST_SOLS_KEPT + i], &population[sorted_population[i]]);
        }
        
        /* Migro soluciones desde poblaciones elite vecinas */
        int neigh_pop, migrated;
        for (int n = BEST_SOLS_KEPT; n < max_pop_sols; n++) {
            migrated = 0;
            neigh_pop = n-BEST_SOLS_KEPT;
            
            /*if (neigh_pop < input->thread_count) {
                if (n_closest_threads[neigh_pop] != -1) {
                    #ifdef DEBUG_3
                        fprintf(stderr, "[DEBUG] Thread %d, migrando solución %d desde %d (to pos %d)\n", 
                            thread_id, instance->sorted_population[neigh_pop][1], neigh_pop, sorted_population[n]);
                    #endif
                    
                    clone_solution(&population[sorted_population[n]], 
                        &(instance->population[neigh_pop][instance->sorted_population[neigh_pop][1]]));
                        
                    migrated = 1;
                }
            } */
            
            if (migrated == 0) {
                #ifdef DEBUG_3
                    fprintf(stderr, "[DEBUG] Thread %d, muto la sol %d usando la sol %d\n", 
                        thread_id, sorted_population[n], sorted_population[0]);
                #endif
                
                mutate(rand_state[thread_id], &population[sorted_population[0]], &population[sorted_population[n]]);
            }
            
            fitness_population[sorted_population[n]] = NAN;
            fitness(population, fitness_population, weights, makespan_norm_value, energy_norm_value, sorted_population[n]);
        }
        
        /* ***************************************** */
        /* Espero a que los demas esclavos terminen. */
        /* ***************************************** */
        rc = pthread_barrier_wait(&instance->sync_barrier);
        if(rc != 0 && rc != PTHREAD_BARRIER_SERIAL_THREAD)
        {
            printf("Could not wait on barrier\n");
            exit(EXIT_FAILURE);
        }

        // Muto la mejor solución que tenía.
        //for (int i = 0; i < BEST_SOLS_KEPT; i++) {
        for (int i = 0; i < 1; i++) {
            mutate(rand_state[thread_id], &population[sorted_population[i]], &population[sorted_population[i]]);
            fitness_population[sorted_population[i]] = NAN;
            fitness(population, fitness_population, weights, makespan_norm_value, energy_norm_value, sorted_population[i]);
        }
                
        /* Re-sort de population */
        merge_sort(population, weights, makespan_norm_value, energy_norm_value, 
            sorted_population, fitness_population, max_pop_sols);

        #ifdef DEBUG_3
            fprintf(stderr, "[DEBUG] Post-migration population\n");
            fprintf(stderr, "parents> ");
            for (int i = 0; i < input->population_size; i++) {
                fprintf(stderr, "%d(%f)<%d>  ", sorted_population[i], fitness_population[sorted_population[i]], population[sorted_population[i]].initialized);
            }
            fprintf(stderr, "\n");
            fprintf(stderr, "childs > ");
            for (int i = input->population_size; i < max_pop_sols; i++) {
                fprintf(stderr, "%d(%f)<%d>  ", sorted_population[i], fitness_population[sorted_population[i]], population[sorted_population[i]].initialized);
            }
            fprintf(stderr, "\n");
        #endif
    }

    // ================================================================
    // Finalizo el thread.
    // ================================================================
    for (int i = 0; i < max_pop_sols; i++) {
        free_solution(&(population[i]));
    }

    free(population);
    free(sorted_population);
    free(fitness_population);

    return 0;
}

inline void merge_sort(struct solution *population, float *weights, 
    float makespan_norm_value, float energy_norm_value, int *sorted_population, 
    float *fitness_population, int population_size) {
        
    int increment, l, l_max, r, r_max, current, i;
    int *tmp;

    increment = 1;
    tmp = (int*)malloc(sizeof(int) * population_size);

    float fitness_r, fitness_l;

    while (increment < population_size) {
        l = 0;
        r = increment;
        l_max = r - 1;
        r_max = (l_max + increment < population_size) ? l_max + increment : population_size - 1;

        current = 0;

        while (current < population_size) {
            while (l <= l_max && r <= r_max) {
                fitness_r = fitness(population, fitness_population, weights, 
                    makespan_norm_value, energy_norm_value, sorted_population[r]);
                fitness_l = fitness(population, fitness_population, weights, 
                    makespan_norm_value, energy_norm_value, sorted_population[l]);
                
                /*fitness_r = fitness_population[sorted_population[r]];
                fitness_l = fitness_population[sorted_population[l]];*/

                if (!isnan(fitness_r) && !isnan(fitness_l)) {
                    if (fitness_r < fitness_l) {
                        tmp[current] = sorted_population[r++];
                    } else {
                        tmp[current] = sorted_population[l++];
                    }
                } else if (!isnan(fitness_r) && isnan(fitness_l)) {
                    tmp[current] = sorted_population[r++];
                } else if (isnan(fitness_r) && !isnan(fitness_l)) {
                    tmp[current] = sorted_population[l++];
                } else {
                    /* Ambos son NAN, no importa */
                    tmp[current] = sorted_population[l++];
                }

                current++;
            }

            while (r <= r_max) tmp[current++] = sorted_population[r++];
            while (l <= l_max) tmp[current++] = sorted_population[l++];

            l = r;
            r += increment;
            l_max = r - 1;
            r_max = (l_max + increment < population_size) ? l_max + increment : population_size - 1;
        }

        increment *= 2;

        for (i = 0; i < population_size; i++) {
            sorted_population[i] = tmp[i];
        }
    }

    free(tmp);
}
