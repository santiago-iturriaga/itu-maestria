#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include "config.h"
#include "util.h"
#include "cuda-util.h"
#include "mtgp-1.1/mtgp32-cuda.h"
#include "billionga.h"

//#define SHOW_PROB_VECTOR_BITS   16
#define SHOW_PROB_VECTOR_BITS   128
#define SHOW_SAMPLE_BITS        128

#define SAMPLE_PROB_VECTOR_BLOCKS    128
#define SAMPLE_PROB_VECTOR_THREADS   256
#define SAMPLE_PROB_VECTOR_SHMEM     (SAMPLE_PROB_VECTOR_THREADS >> 5)

#define UPDATE_PROB_VECTOR_BLOCKS    128
#define UPDATE_PROB_VECTOR_THREADS   256
#define UPDATE_PROB_VECTOR_SHMEM     (UPDATE_PROB_VECTOR_THREADS >> 5)

// Paso 1 del algoritmo.
void bga_initialization(struct bga_state *state, long number_of_bits, int number_of_prob_vectors, int number_of_samples) {
    state->number_of_bits = number_of_bits;
    state->number_of_samples = number_of_samples;
    state->number_of_prob_vectors = number_of_prob_vectors;

    state->population_size = POPULATION_SIZE; //sqrt(3.1416) * sqrt(number_of_bits) * log10(number_of_bits) / 2;
    state->update_value = 1; // / state->population_size;
    state->max_prob_sum = (number_of_bits * POPULATION_SIZE);

    //#if defined(INFO) || defined(DEBUG)
        fprintf(stdout, "[INFO] === Initializing Billion GA ====================\n");
        fprintf(stdout, "[INFO] Problem size   : %ld\n", number_of_bits);
        fprintf(stdout, "[INFO] Population size: %d\n", state->population_size);
        fprintf(stdout, "[INFO] Update value   : %d\n", state->update_value);
        fprintf(stdout, "[INFO] Num. of vectors: %d\n", state->number_of_prob_vectors);
    //#endif

    // === Pido la memoria =============================================================
    #if defined(INFO) || defined(DEBUG)
        fprintf(stdout, "[INFO] === Solicitando memoria =======================\n");
    #endif

    // === Pido la memoria para el vector de probabilidades ==================================

    state->prob_vector_bit_count = state->number_of_bits / number_of_prob_vectors;

    int bits_left = state->number_of_bits % number_of_prob_vectors;
    if (bits_left == 0) {
        state->last_prob_vector_bit_count = state->prob_vector_bit_count;
    } else {
        state->last_prob_vector_bit_count = bits_left;
    }

    #ifdef INFO
        fprintf(stdout, "[INFO] Requesting a size %d prob_vector_size CPU memory\n", state->number_of_prob_vectors);
    #endif

    size_t prob_vectors_acc_prob_array_size = sizeof(float) * state->number_of_prob_vectors;
    state->prob_vectors_acc_prob = (float*)malloc(prob_vectors_acc_prob_array_size);
    if (!state->prob_vectors_acc_prob) {
        fprintf(stderr, "[ERROR] Requesting CPU memory for the prob_vectors_acc_prob\n");
        exit(EXIT_FAILURE);
    }

    size_t prob_vector_array_size = sizeof(int*) * state->number_of_prob_vectors;
    state->gpu_prob_vectors = (int**)malloc(prob_vector_array_size);
    if (!state->gpu_prob_vectors) {
        fprintf(stderr, "[ERROR] Requesting CPU memory for the prob_vector\n");
        exit(EXIT_FAILURE);
    }

    // === Pido la memoria para los samples ==================================================

    #ifdef INFO
        fprintf(stdout, "[INFO] Requesting a size %d samples CPU memory\n", state->number_of_samples);
    #endif

    size_t samples_vector_fitness_array_size = sizeof(int*) * state->number_of_samples;
    state->samples_vector_fitness = (int**)malloc(samples_vector_fitness_array_size);
    if (!state->samples_vector_fitness) {
        fprintf(stderr, "[ERROR] Requesting samples_vector_fitness CPU memory\n");
        exit(EXIT_FAILURE);
    }

    size_t samples_array_size = sizeof(int*) * state->number_of_samples;
    state->gpu_samples = (int***)malloc(samples_array_size);
    if (!state->gpu_samples) {
        fprintf(stderr, "[ERROR] Requesting samples_fitness CPU memory\n");
        exit(EXIT_FAILURE);
    }

    for (int sample_number = 0; sample_number < state->number_of_samples; sample_number++) {
        #ifdef INFO
            fprintf(stdout, "[INFO] > Requesting CPU memory for sample %d vectors array\n", sample_number);
        #endif

        size_t samples_vector_array_size = sizeof(int*) * state->number_of_prob_vectors;
        state->gpu_samples[sample_number] = (int**)malloc(samples_vector_array_size);
        if (!state->gpu_samples) {
            fprintf(stderr, "[ERROR] > Requesting CPU memory for sample_vector_array[%d]\n", sample_number);
            exit(EXIT_FAILURE);
        }

        size_t samples_vector_fitness_size = sizeof(int) * state->number_of_prob_vectors;
        state->samples_vector_fitness[sample_number] = (int*)malloc(samples_vector_fitness_size);
        if (!state->samples_vector_fitness[sample_number]) {
            fprintf(stderr, "[ERROR] Requesting samples_fitness CPU memory\n");
            exit(EXIT_FAILURE);
        }
    }

    size_t samples_fitness_size = sizeof(int*) * state->number_of_samples;

    #ifdef INFO
        fprintf(stdout, "[INFO] Requesting samples_fitness CPU memory (size: %i)\n", state->number_of_samples);
    #endif

    state->samples_fitness = (int*)malloc(samples_fitness_size);
    if (!state->samples_fitness) {
        fprintf(stderr, "[ERROR] > Requesting CPU memory for samples_fitness_size\n");
        exit(EXIT_FAILURE);
    }

    // === Memoria auxiliar ==================================================================
    size_t gpu_int32_vector_sum_size = sizeof(long*) * state->number_of_prob_vectors;
    state->gpu_int32_vector_sum = (long**)malloc(gpu_int32_vector_sum_size);

    size_t cpu_int32_vector_sum_size = sizeof(long*) * state->number_of_prob_vectors;
    state->cpu_int32_vector_sum = (long**)malloc(cpu_int32_vector_sum_size);

    size_t gpu_bit_vector_sum_size = sizeof(int*) * state->number_of_prob_vectors;
    state->gpu_bit_vector_sum = (int**)malloc(gpu_bit_vector_sum_size);

    size_t cpu_bit_vector_sum_size = sizeof(int*) * state->number_of_prob_vectors;
    state->cpu_bit_vector_sum = (int**)malloc(cpu_bit_vector_sum_size);
}

void bga_initialize_thread(struct bga_state *state, int prob_vector_number) {
    hipError_t error;

    #if defined(TIMMING)
        float gputime;
        hipEvent_t start;
        hipEvent_t end;

        ccudaEventCreate(&start);
        ccudaEventCreate(&end);

        ccudaEventRecord(start, 0);
    #endif

    #if defined(INFO) || defined(DEBUG)
        fprintf(stdout, "[INFO] === Solicitando memoria para el thread %d =====\n", prob_vector_number);
    #endif

    // === Pido la memoria para el vector de probabilidades ==================================
    {
        int current_prob_vector_number_of_bits = state->prob_vector_bit_count;
        if (prob_vector_number + 1 == state->number_of_prob_vectors) {
            current_prob_vector_number_of_bits = state->last_prob_vector_bit_count;
        }

        size_t prob_vector_size = sizeof(int) * current_prob_vector_number_of_bits;
        #ifdef INFO
            fprintf(stdout, "[INFO] > Requesting %d bits GPU memory for prob_vector %d (size: %i / %lu Mb)\n",
                current_prob_vector_number_of_bits, prob_vector_number, current_prob_vector_number_of_bits,
                prob_vector_size >> 20);
        #endif
        error = hipMalloc((void**)&(state->gpu_prob_vectors[prob_vector_number]), prob_vector_size);

        if (error != hipSuccess) {
            fprintf(stderr, "[ERROR] Requesting GPU memory for prob_vector_number[%d]\n", prob_vector_number);
            exit(EXIT_FAILURE);
        }
    }
    
    #if defined(TIMMING)
        ccudaEventRecord(end, 0);
        ccudaEventSynchronize(end);
        ccudaEventElapsedTime(&gputime, start, end);
        fprintf(stdout, "TIME] Processing time: %f (ms)\n", gputime);
        
        ccudaEventRecord(start, 0);
    #endif

    // === Pido la memoria para los samples ==================================================
    #ifdef INFO
        fprintf(stdout, "[INFO] Requesting a size %d samples CPU memory\n", state->number_of_samples);
    #endif

    {
        for (int sample_number = 0; sample_number < state->number_of_samples; sample_number++) {
            int current_prob_vector_number_of_bits = state->prob_vector_bit_count;
            if (prob_vector_number + 1 == state->number_of_prob_vectors) {
                current_prob_vector_number_of_bits = state->last_prob_vector_bit_count;
            }
            size_t sample_vector_size = sizeof(int) * (current_prob_vector_number_of_bits >> 5);

            int right_size = current_prob_vector_number_of_bits & ((1<<5)-1);
            assert(right_size == 0);

            #ifdef INFO
                fprintf(stdout, "[INFO] > Requesting sample %d GPU memory for vector %d (size: %i / %lu Mb)\n",
                    sample_number, prob_vector_number, current_prob_vector_number_of_bits >> 5, sample_vector_size >> 20);
            #endif

            error = hipMalloc((void**)&(state->gpu_samples[sample_number][prob_vector_number]), sample_vector_size);
            if (error != hipSuccess) {
                fprintf(stderr, "[ERROR] > Requesting GPU memory for sample_number[%d]\n", sample_number);
                exit(EXIT_FAILURE);
            }
        }
    }

    #if defined(TIMMING)
        ccudaEventRecord(end, 0);
        ccudaEventSynchronize(end);
        ccudaEventElapsedTime(&gputime, start, end);
        fprintf(stdout, "TIME] Processing time: %f (ms)\n", gputime);
        
        ccudaEventRecord(start, 0);
    #endif

    // === Inicializo el vector de probabilidades ============================================
    #if defined(INFO) || defined(DEBUG)
        fprintf(stdout, "[INFO] === Inicializando memoria =======================\n");
    #endif

    #if defined(TIMMING)
        ccudaEventCreate(&start);
        ccudaEventCreate(&end);

        ccudaEventRecord(start, 0);
    #endif

    {
        int current_prob_vector_number_of_bits = state->prob_vector_bit_count;
        if (prob_vector_number + 1 == state->number_of_prob_vectors) {
            current_prob_vector_number_of_bits = state->last_prob_vector_bit_count;
        }

        #ifdef INFO
            fprintf(stdout, "[INFO] Inicializando GPU memory of prob_vector %d (%d bits)\n",
                prob_vector_number, current_prob_vector_number_of_bits);
        #endif

        vector_set_int(state->gpu_prob_vectors[prob_vector_number],
            current_prob_vector_number_of_bits, INIT_PROB_VECTOR_VALUE);
    }

    #if defined(TIMMING)
    ccudaEventRecord(end, 0);
    ccudaEventSynchronize(end);
    ccudaEventElapsedTime(&gputime, start, end);
    fprintf(stdout, "[TIME] Processing time: %f (ms)\n", gputime);

    ccudaEventDestroy(start);
    ccudaEventDestroy(end);
    #endif

    // === Memoria auxiliar ==================================================================
    vector_sum_int_alloc(&(state->gpu_int32_vector_sum[prob_vector_number]),
        &(state->cpu_int32_vector_sum[prob_vector_number]));

    vector_sum_bit_alloc(&(state->gpu_bit_vector_sum[prob_vector_number]),
        &(state->cpu_bit_vector_sum[prob_vector_number]));
}

float bga_get_part_accumulated_prob(struct bga_state *state, int prob_vector_number) {
    vector_sum_int_init(state->gpu_int32_vector_sum[prob_vector_number]);

    int current_prob_vector_number_of_bits = state->prob_vector_bit_count;
    if (prob_vector_number + 1 == state->number_of_prob_vectors) {
        current_prob_vector_number_of_bits = state->last_prob_vector_bit_count;
    }

    vector_sum_int(state->gpu_prob_vectors[prob_vector_number],
        state->gpu_int32_vector_sum[prob_vector_number],
        current_prob_vector_number_of_bits);

    state->prob_vectors_acc_prob[prob_vector_number] = (vector_sum_int_get(
        state->gpu_int32_vector_sum[prob_vector_number],
        state->cpu_int32_vector_sum[prob_vector_number]) / state->population_size);

    return state->prob_vectors_acc_prob[prob_vector_number];
}

float bga_get_full_accumulated_prob(struct bga_state *state) {
    float result = 0.0;

    for (int prob_vector_number = 0; prob_vector_number < state->number_of_prob_vectors; prob_vector_number++) {
        result += state->prob_vectors_acc_prob[prob_vector_number];
    }

    return result;
}

void bga_show_prob_vector_state(struct bga_state *state) {
    #if defined(TIMMING)
    float gputime;
    hipEvent_t start;
    hipEvent_t end;

    ccudaEventCreate(&start);
    ccudaEventCreate(&end);

    ccudaEventRecord(start, 0);
    #endif

    fprintf(stdout, "[INFO] === Probability vector status =======================\n");

    vector_sum_int_init(state->gpu_int32_vector_sum[0]);
    
    vector_sum_int_show(state->gpu_int32_vector_sum[0], state->cpu_int32_vector_sum[0]);

    fprintf(stdout, "[INFO] Prob. vector sample:");

    for (int prob_vector_number = 0; prob_vector_number < state->number_of_prob_vectors; prob_vector_number++) {
        int current_prob_vector_number_of_bits = state->prob_vector_bit_count;
        if (prob_vector_number + 1 == state->number_of_prob_vectors) {
            current_prob_vector_number_of_bits = state->last_prob_vector_bit_count;
        }

        if (prob_vector_number == 0) {
            int probs_to_show_count = SHOW_PROB_VECTOR_BITS;
            if (current_prob_vector_number_of_bits < SHOW_PROB_VECTOR_BITS)
                probs_to_show_count = state->prob_vector_bit_count;

            int *probs_to_show = (int*)malloc(sizeof(int) * probs_to_show_count);
            
            ccudaMemcpy(probs_to_show, state->gpu_prob_vectors[prob_vector_number],
                sizeof(uint32_t) * probs_to_show_count, hipMemcpyDeviceToHost);

            long sum = 0;

            for (int i = 0; i < probs_to_show_count; i++) {               
                fprintf(stdout, " %d (%.4f)", probs_to_show[i], (float)probs_to_show[i] / (float)state->population_size);
                sum += probs_to_show[i];
            }
                      
            fprintf(stdout, "... Total [%d]: %ld ( %f )\n", probs_to_show_count, sum, (float)sum / (float)(probs_to_show_count * state->population_size));
            
            free(probs_to_show);
        }

        vector_sum_int(state->gpu_prob_vectors[prob_vector_number],
            state->gpu_int32_vector_sum[0], current_prob_vector_number_of_bits);
    }

    long accumulated_probability = 0;
    accumulated_probability = vector_sum_int_get(
        state->gpu_int32_vector_sum[0], 
        state->cpu_int32_vector_sum[0]);
    fprintf(stdout, "[INFO] Prob. vector accumulated probability (%ld / %ld): %f\n", 
        accumulated_probability, state->max_prob_sum, 
        (float)accumulated_probability / (float)state->max_prob_sum);

    #if defined(TIMMING)
    ccudaEventRecord(end, 0);
    ccudaEventSynchronize(end);
    ccudaEventElapsedTime(&gputime, start, end);
    fprintf(stdout, "[TIME] Processing time: %f (ms)\n", gputime);

    ccudaEventDestroy(start);
    ccudaEventDestroy(end);
    #endif
}

void bga_compute_sample_part_fitness(struct bga_state *state, int prob_vector_number) {
    #if defined(TIMMING)
        float gputime;
        hipEvent_t start;
        hipEvent_t end;

        ccudaEventCreate(&start);
        ccudaEventCreate(&end);

        ccudaEventRecord(start, 0);
    #endif
    
    #if defined(INFO)
        fprintf(stdout, "[INFO] === Sample vectors fitness =============================\n");
    #endif

    for (int sample_number = 0; sample_number < state->number_of_samples; sample_number++) {
        vector_sum_bit_init(state->gpu_bit_vector_sum[prob_vector_number]);

        #if defined(DEBUG)
            fprintf(stdout, "[INFO] Computing sample vector %d fitness: ", sample_number);
        #endif

        int current_prob_vector_number_of_bits = state->prob_vector_bit_count;
        if (prob_vector_number + 1 == state->number_of_prob_vectors) {
            current_prob_vector_number_of_bits = state->last_prob_vector_bit_count;
        }

        vector_sum_bit(state->gpu_samples[sample_number][prob_vector_number],
            state->gpu_bit_vector_sum[prob_vector_number], current_prob_vector_number_of_bits);

        state->samples_vector_fitness[sample_number][prob_vector_number] = vector_sum_bit_get(
            state->gpu_bit_vector_sum[prob_vector_number], 
            state->cpu_bit_vector_sum[prob_vector_number]);

        #if defined(DEBUG)
            fprintf(stdout, "%d\n", state->samples_vector_fitness[sample_number][prob_vector_number]);
        #endif
    }

    #if defined(TIMMING)
        ccudaEventRecord(end, 0);
        ccudaEventSynchronize(end);
        ccudaEventElapsedTime(&gputime, start, end);
        fprintf(stdout, "[TIME] Processing time: %f (ms)\n", gputime);

        ccudaEventDestroy(start);
        ccudaEventDestroy(end);
    #endif
}

void bga_compute_sample_full_fitness(struct bga_state *state) {
    int result;

    for (int sample_number = 0; sample_number < state->number_of_samples; sample_number++) {
        result = 0;

        for (int prob_vector_number = 0; prob_vector_number < state->number_of_prob_vectors; prob_vector_number++) {
            result += state->samples_vector_fitness[sample_number][prob_vector_number];
        }

        state->samples_fitness[sample_number] = result;
    }
}

void bga_show_samples(struct bga_state *state) {
    #if defined(TIMMING)
        float gputime;
        hipEvent_t start;
        hipEvent_t end;

        ccudaEventCreate(&start);
        ccudaEventCreate(&end);

        ccudaEventRecord(start, 0);
    #endif

    fprintf(stdout, "[INFO] === Sample vectors =====================================\n");

    for (int sample_number = 0; sample_number < state->number_of_samples; sample_number++) {
        fprintf(stdout, "[INFO] Sample vector sample (%d):", sample_number);

        for (int prob_vector_number = 0; prob_vector_number < state->number_of_prob_vectors; prob_vector_number++) {
            int current_prob_vector_number_of_bits = state->prob_vector_bit_count;
            if (prob_vector_number + 1 == state->number_of_prob_vectors) {
                current_prob_vector_number_of_bits = state->last_prob_vector_bit_count;
            }

            if (prob_vector_number == 0) {
                int bits_to_show_count = SHOW_SAMPLE_BITS;
                if (current_prob_vector_number_of_bits < SHOW_SAMPLE_BITS)
                    bits_to_show_count = state->prob_vector_bit_count;

                int bytes_to_show_count = bits_to_show_count >> 5;
                int *bytes_to_show = (int*)malloc(sizeof(int) * bytes_to_show_count);

                ccudaMemcpy(bytes_to_show, state->gpu_samples[sample_number][prob_vector_number],
                    sizeof(uint32_t) * bytes_to_show_count, hipMemcpyDeviceToHost);

                for (int i = 0; i < bytes_to_show_count; i++) {
                    fprintf(stdout, " %s", int_to_binary(bytes_to_show[i]));
                }

                free(bytes_to_show);

                fprintf(stdout, "...\n");
            }
        }

        fprintf(stdout, "[INFO] Sample %d fitness: %d\n", sample_number, state->samples_fitness[sample_number]);
    }

    #if defined(TIMMING)
        ccudaEventRecord(end, 0);
        ccudaEventSynchronize(end);
        ccudaEventElapsedTime(&gputime, start, end);
        fprintf(stdout, "[TIME] Processing time: %f (ms)\n", gputime);

        ccudaEventDestroy(start);
        ccudaEventDestroy(end);
    #endif
}

__global__ void kern_sample_prob_vector(int *gpu_prob_vector, int prob_vector_size,
    int prob_vector_starting_pos, float *prng_vector, int prng_vector_size, int *gpu_sample,
    int population_size) {

    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    const int samples_per_loop = gridDim.x * blockDim.x;

    int max_samples_doable = prob_vector_size - prob_vector_starting_pos;
    if (max_samples_doable > prng_vector_size) max_samples_doable = prng_vector_size;

    int loops_count = max_samples_doable / samples_per_loop;
    if (max_samples_doable % samples_per_loop > 0) loops_count++;

    __shared__ int current_block_sample[SAMPLE_PROB_VECTOR_THREADS];

    int prob_vector_position;
    int prng_position;
    int block_starting_pos;

    for (int loop = 0; loop < loops_count; loop++) {
        // 0 por defecto.
        current_block_sample[tid] = 0;

        // Cada loop genera blockDim.x bits y los guarda en el array de __shared__ memory.
        block_starting_pos = (samples_per_loop * loop) + (bid * blockDim.x);
        prng_position = block_starting_pos + tid;
        prob_vector_position = prob_vector_starting_pos + prng_position;

        if (prng_position < max_samples_doable) {
            if ((gpu_prob_vector[prob_vector_position] + population_size) >= (prng_vector[prng_position] * population_size)) {
                // 1
                current_block_sample[tid] = 1 << (tid & ((1 << 5)-1));
            }
        }

        __syncthreads();

        if ((tid << 5) < SAMPLE_PROB_VECTOR_THREADS) {
            int aux = current_block_sample[tid << 5];

            #pragma unroll
            for (int i = 1; i < 32; i++) {
                aux = aux | current_block_sample[(tid << 5)+i];
            }

            int sample_pos = prob_vector_starting_pos + block_starting_pos;

            if ((sample_pos + (tid << 5)) < prob_vector_size) {
                gpu_sample[(sample_pos >> 5) + tid] = aux;
            }
        }

        __syncthreads();
    }
}

// Paso 2 del algoritmo.
void bga_model_sampling_mt(struct bga_state *state, mtgp32_status *mt_status, int prob_vector_number) {
    #if defined(DEBUG)
        fprintf(stdout, "[INFO] === Sampling the model =======================\n");
    #endif
    
    #if defined(TIMMING)
        float gputime;
        hipEvent_t start;
        hipEvent_t end;

        hipEvent_t start_inner;
        hipEvent_t end_inner;
    #endif

    for (int sample_number = 0; sample_number < state->number_of_samples; sample_number++) {
        #if defined(DEBUG)
            fprintf(stdout, "[INFO] > Sample %d ", sample_number);
        #endif
        #if defined(TIMMING)
            ccudaEventCreate(&start);
            ccudaEventCreate(&end);
            ccudaEventRecord(start, 0);

            ccudaEventCreate(&start_inner);
            ccudaEventCreate(&end_inner);
        #endif

        int current_prob_vector_number_of_bits = state->prob_vector_bit_count;
        if (prob_vector_number + 1 == state->number_of_prob_vectors) {
            current_prob_vector_number_of_bits = state->last_prob_vector_bit_count;
        }

        int total_loops;
        total_loops = current_prob_vector_number_of_bits / mt_status->numbers_per_gen;
        if (current_prob_vector_number_of_bits % mt_status->numbers_per_gen > 0) total_loops++;

        int prob_vector_starting_pos;

        for (int loop = 0; loop < total_loops; loop++) {
            prob_vector_starting_pos = mt_status->numbers_per_gen * loop;

            // Genero números aleatorios.
            #if defined(TIMMING)
                fprintf(stdout, "[TIME] Generate mtgp32_generate_float\n", gputime);
                ccudaEventRecord(start_inner, 0);
            #endif

            mtgp32_generate_float(mt_status);

            #if defined(TIMMING)
                ccudaEventRecord(end_inner, 0);
                ccudaEventSynchronize(end_inner);
                ccudaEventElapsedTime(&gputime, start_inner, end_inner);
                fprintf(stdout, "[TIME] Processing time: %f (ms)\n", gputime);
            #endif

            #if defined(DEBUG)
                fprintf(stdout, ".");
            #endif

            #if defined(TIMMING)
                fprintf(stdout, "[TIME] Generate kern_sample_prob_vector\n", gputime);
                ccudaEventRecord(start_inner, 0);
            #endif

            // Sampleo el vector de prob. con los números aleatorios generados.
            kern_sample_prob_vector<<< SAMPLE_PROB_VECTOR_BLOCKS, SAMPLE_PROB_VECTOR_THREADS>>>(
                state->gpu_prob_vectors[prob_vector_number], current_prob_vector_number_of_bits,
                prob_vector_starting_pos, (float*)mt_status->d_data, mt_status->numbers_per_gen,
                state->gpu_samples[sample_number][prob_vector_number], state->population_size);

            #if defined(TIMMING)
                ccudaEventRecord(end_inner, 0);
                ccudaEventSynchronize(end_inner);
                ccudaEventElapsedTime(&gputime, start_inner, end_inner);
                fprintf(stdout, "[TIME] Processing time: %f (ms)\n", gputime);
            #endif
        }

        #if defined(DEBUG)
            fprintf(stdout, "(%d)\n", total_loops);
        #endif
        #if defined(TIMMING)
            ccudaEventRecord(end, 0);
            ccudaEventSynchronize(end);
            ccudaEventElapsedTime(&gputime, start, end);
            fprintf(stdout, "[TIME] Total processing time: %f (ms)\n", gputime);
        #endif
    }

    bga_show_samples(state);

    #if defined(TIMMING)
        ccudaEventDestroy(start_inner);
        ccudaEventDestroy(end_inner);

        ccudaEventDestroy(start);
        ccudaEventDestroy(end);
    #endif
}

void cpu_model_update(int *gpu_prob_vector, int prob_vector_size,
    int *gpu_best_sample, int *gpu_worst_sample, float update_value) {

    int *prob_vector = (int*)malloc(sizeof(int) * prob_vector_size);

    ccudaMemcpy(prob_vector, gpu_prob_vector, sizeof(uint32_t) * prob_vector_size, 
        hipMemcpyDeviceToHost);

    long current_acc_prob = 0, new_acc_prob = 0;
    for (int i = 0; i < prob_vector_size; i++) {
        current_acc_prob += prob_vector[i];
    }

    int sample_size = prob_vector_size >> 5;
    int *best_sample = (int*)malloc(sizeof(int) * sample_size);
    int *worst_sample = (int*)malloc(sizeof(int) * sample_size);

    ccudaMemcpy(best_sample, gpu_best_sample,
        sizeof(uint32_t) * sample_size, hipMemcpyDeviceToHost);
        
    ccudaMemcpy(worst_sample, gpu_worst_sample,
        sizeof(uint32_t) * sample_size, hipMemcpyDeviceToHost);

    int best_sample_current_bit_value;
    int worst_sample_current_bit_value;
    int delta;

    for (int i = 0; i < prob_vector_size; i++) {
        int bit_pos = i & ((1 << 5)-1);
        int int_pos = i >> 5;
        
        best_sample_current_bit_value = (best_sample[int_pos] & (1 << bit_pos)) >> bit_pos;
        worst_sample_current_bit_value = (worst_sample[int_pos] & (1 << bit_pos)) >> bit_pos;

        delta = best_sample_current_bit_value - worst_sample_current_bit_value;
        prob_vector[i] += delta;
    }
    
    for (int i = 0; i < prob_vector_size; i++) {
        new_acc_prob += prob_vector[i];
    }
    
    fprintf(stdout, "[DEBUG][CPU] Acc. prob. => Current %ld , New %ld (delta: %ld )\n", 
        current_acc_prob, new_acc_prob, new_acc_prob - current_acc_prob);
        
    free(prob_vector);
    free(best_sample);
    free(worst_sample);
}

__global__ void kern_model_update(int *gpu_prob_vector, int prob_vector_size,
    int *best_sample, int *worst_sample, float update_value) {

    const int tid = threadIdx.x;
    const int bid = blockIdx.x;

    __shared__ int best_sample_part[UPDATE_PROB_VECTOR_SHMEM];
    __shared__ int worst_sample_part[UPDATE_PROB_VECTOR_SHMEM];

    int loop_size = gridDim.x * blockDim.x;
    int loop_count = prob_vector_size / loop_size;
    if (loop_count % loop_size > 0) loop_count++;

    int prob_vector_position;
    int block_starting_pos;

    const int tid_int = tid >> 5;
    const int tid_bit = tid & ((1 << 5)-1);

    int best_sample_current_bit_value;
    int worst_sample_current_bit_value;
    int delta;

    for (int loop = 0; loop < loop_count; loop++) {
        block_starting_pos = (loop_size * loop) + (bid * blockDim.x);

        if (tid < UPDATE_PROB_VECTOR_SHMEM) {
            if ((block_starting_pos + (tid << 5)) < prob_vector_size) {
                best_sample_part[tid] = best_sample[(block_starting_pos >> 5) + tid];
                worst_sample_part[tid] = worst_sample[(block_starting_pos >> 5) + tid];
            }
        }
        __syncthreads();

        prob_vector_position = block_starting_pos + tid;

        if (prob_vector_position < prob_vector_size) {
            best_sample_current_bit_value = (best_sample_part[tid_int] & (1 << tid_bit)) >> tid_bit;
            worst_sample_current_bit_value = (worst_sample_part[tid_int] & (1 << tid_bit)) >> tid_bit;

            delta = best_sample_current_bit_value - worst_sample_current_bit_value;
            
            //atomicAdd(&(gpu_prob_vector[prob_vector_position]), delta);
            
            float aux = gpu_prob_vector[prob_vector_position];
            gpu_prob_vector[prob_vector_position] = aux + (delta * update_value);
        }
    }
}

// Paso 4 y 5 del algoritmo.
void bga_model_update(struct bga_state *state, int prob_vector_number) {
    #if defined(DEBUG)
        fprintf(stdout, "[INFO] === Updating the model =======================\n");
    #endif
    #if defined(TIMMING)
        float gputime;
        hipEvent_t start;
        hipEvent_t end;

        ccudaEventCreate(&start);
        ccudaEventCreate(&end);
        ccudaEventRecord(start, 0);
    #endif

    assert(state->number_of_samples == 2);

    int best_sample_index, worst_sample_index;

    int fitness_sample_a, fitness_sample_b;
    #if defined(FULL_FITNESS_UPDATE)
        fitness_sample_a = state->samples_fitness[0];
        fitness_sample_b = state->samples_fitness[1];
    #endif
    #if defined(PARTIAL_FITNESS_UPDATE)
        fitness_sample_a = state->samples_vector_fitness[0][prob_vector_number];
        fitness_sample_b = state->samples_vector_fitness[1][prob_vector_number];
    #endif

    if (fitness_sample_a >= fitness_sample_b) {
        best_sample_index = 0;
        worst_sample_index = 1;
    }
    else {
        best_sample_index = 1;
        worst_sample_index = 0;
    }

    int *best_sample;
    int *worst_sample;

    int current_prob_vector_number_of_bits = state->prob_vector_bit_count;
    if (prob_vector_number + 1 == state->number_of_prob_vectors) {
        current_prob_vector_number_of_bits = state->last_prob_vector_bit_count;
    }

    best_sample = state->gpu_samples[best_sample_index][prob_vector_number];
    worst_sample = state->gpu_samples[worst_sample_index][prob_vector_number];

    cpu_model_update(state->gpu_prob_vectors[prob_vector_number], current_prob_vector_number_of_bits,
        best_sample, worst_sample, state->update_value);

    kern_model_update <<< UPDATE_PROB_VECTOR_BLOCKS, UPDATE_PROB_VECTOR_THREADS >>>(
        state->gpu_prob_vectors[prob_vector_number], current_prob_vector_number_of_bits,
        best_sample, worst_sample, state->update_value);

    #if defined(TIMMING)
        ccudaEventRecord(end, 0);
        ccudaEventSynchronize(end);
        ccudaEventElapsedTime(&gputime, start, end);
        fprintf(stdout, "[TIME] Processing time: %f (ms)\n", gputime);

        ccudaEventDestroy(start);
        ccudaEventDestroy(end);
    #endif
}

// Libera la memoria pedida para de estado.
void bga_free(struct bga_state *state) {
    #ifdef INFO
    fprintf(stdout, "[INFO] Freeing memory\n");
    #endif

    for (int vector_number = 0; vector_number < state->number_of_prob_vectors; vector_number++) {
        fprintf(stderr, "[INFO] Freeing gpu_prob_vectors[%d]\n", vector_number);
        hipFree(state->gpu_prob_vectors[vector_number]);
    }

    free(state->gpu_prob_vectors);

    for (int sample_number = 0; sample_number < state->number_of_samples; sample_number++) {
        for (int vector_number = 0; vector_number < state->number_of_prob_vectors; vector_number++) {
            fprintf(stderr, "[INFO] Freeing gpu_samples[%d][%d]\n", sample_number, vector_number);
            hipFree(state->gpu_samples[sample_number][vector_number]);
        }
        free(state->gpu_samples[sample_number]);
    }
       
    free(state->gpu_samples);
    free(state->samples_fitness);

    for (int vector_number = 0; vector_number < state->number_of_prob_vectors; vector_number++) {
        fprintf(stderr, "[INFO] Freeing vector_sum_float_free[%d]\n", vector_number);
        vector_sum_int_free(
            state->gpu_int32_vector_sum[vector_number], 
            state->cpu_int32_vector_sum[vector_number]);
            
        fprintf(stderr, "[INFO] Freeing vector_sum_bit_free[%d]\n", vector_number);
        vector_sum_bit_free(
            state->gpu_bit_vector_sum[vector_number], 
            state->cpu_bit_vector_sum[vector_number]);
    }
    
    free(state->gpu_int32_vector_sum);
    free(state->cpu_int32_vector_sum);
    free(state->gpu_bit_vector_sum);
    free(state->cpu_bit_vector_sum);
}
