#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "config.h"
#include "billionga.h"

// Paso 1 del algoritmo.
void bga_initialization(struct bga_state *state, long number_of_bits, int number_of_samples) {
    state->number_of_bits = number_of_bits;
    state->number_of_samples = number_of_samples;
    
    hipError_t error;
    
    #ifdef INFO
    fprintf(stdout, "[INFO] Requesting prob_vector_size memory\n");
    #endif
    
    if (state->number_of_bits > MAX_PROB_VECTOR_BITS) {
        state->number_of_prob_vectors = state->number_of_bits / MAX_PROB_VECTOR_BITS;
        state->last_prob_vector_bit_count = state->number_of_bits % MAX_PROB_VECTOR_BITS;
        
        if (state->last_prob_vector_bit_count == 0) {
            state->last_prob_vector_bit_count = MAX_PROB_VECTOR_BITS;
        } else {
            state->number_of_prob_vectors++;
        }
    } else {
        state->number_of_prob_vectors = 1;
        state->last_prob_vector_bit_count = state->number_of_bits;
    }
      
    size_t prob_vector_array_size = sizeof(float*) * state->number_of_prob_vectors;
    state->gpu_prob_vectors = (float**)malloc(prob_vector_array_size);
    if (!state->gpu_prob_vectors) {
        fprintf(stderr, "[ERROR] Requesting memory for the prob_vector\n");
        exit(EXIT_FAILURE);
    }

    for (int prob_vector_number = 0; prob_vector_number < state->number_of_prob_vectors; prob_vector_number++) {
        int current_prob_vector_number_of_bits = MAX_PROB_VECTOR_BITS;
        if (prob_vector_number + 1 == state->number_of_prob_vectors) {
            current_prob_vector_number_of_bits = state->last_prob_vector_bit_count;
        }

        #ifdef INFO
        fprintf(stdout, "[INFO] > Requesting %d bits memory for prob_vector %d\n", current_prob_vector_number_of_bits, prob_vector_number);
        #endif

        size_t prob_vector_size = sizeof(float) * current_prob_vector_number_of_bits;
        error = hipMalloc((void**)&(state->gpu_prob_vectors[prob_vector_number]), prob_vector_size);
        
        if (error != hipSuccess) {
            fprintf(stderr, "[ERROR] > Requesting memory for prob_vector_number[%d]\n", prob_vector_number);
            exit(EXIT_FAILURE);
        }
    }
       
    #ifdef INFO
    fprintf(stdout, "[INFO] Requesting samples memory\n");
    #endif
    
    size_t samples_array_size = sizeof(float*) * state->number_of_samples;
    state->gpu_samples = (float**)malloc(samples_array_size);
    if (!state->gpu_samples) {
        fprintf(stderr, "[ERROR] Requesting samples_fitness memory\n");
        exit(EXIT_FAILURE);
    }
   
    size_t sample_size = sizeof(char) * (state->number_of_bits / 8);
    
    for (int sample_number = 0; sample_number < state->number_of_samples; sample_number++) {
        #ifdef INFO
        fprintf(stdout, "[INFO] Requesting memory for sample %d\n", sample_number);
        #endif

        error = hipMalloc((void**)&(state->gpu_samples[sample_number]), sample_size);
        
        if (error != hipSuccess) {
            fprintf(stderr, "[ERROR] > Requesting memory for sample_number[%d]\n", sample_number);
            exit(EXIT_FAILURE);
        }
    }
    
    #ifdef INFO
    fprintf(stdout, "[INFO] Requesting samples_fitness memory\n");
    #endif
    
    size_t samples_fitness_size = sizeof(long) * state->number_of_samples;
    error = hipMalloc((void**)&(state->gpu_samples_fitness), samples_fitness_size);
    if (error != hipSuccess) {
        fprintf(stderr, "[ERROR] Requesting memory for samples_fitness\n");
        exit(EXIT_FAILURE);
    }
}

// Paso 2 del algoritmo.
void bga_model_sampling(struct bga_state *state) {
}

// Paso 3 del algoritmo.
void bga_evaluation(struct bga_state *state) {
}

// Paso 4 y 5 del algoritmo.
void bga_model_update(struct bga_state *state) {
}

// Libera la memoria pedida para de estado.
void bga_free(struct bga_state *state) {
}
