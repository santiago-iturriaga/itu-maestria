#include "hip/hip_runtime.h"
/*
 * Sample Program for CUDA 2.2
 * written by M.Saito (saito@math.sci.hiroshima-u.ac.jp)
 *
 * This sample uses texture reference.
 * The generation speed of PRNG using texture is faster than using
 * constant tabel on Geforce GTX 260.
 *
 * MTGP64-44497
 * This program generates 64-bit unsigned integers.
 * The period of generated integers is 2<sup>44497</sup>-1.
 * This also generates double precision floating point numbers.
 */
#define __STDC_FORMAT_MACROS 1
#define __STDC_CONSTANT_MACROS 1
#include <stdio.h>
#include <cutil.h>
#include <stdint.h>
#include <inttypes.h>
#include <errno.h>
#include <stdlib.h>
extern "C" {
#include "mtgp64-fast.h"
}
#define MEXP 44497
#define N 696
#define THREAD_NUM 512
#define LARGE_SIZE (THREAD_NUM * 3)
#define BLOCK_NUM 32
#define TBL_SIZE 16

/**
 * kernel I/O
 * This structure must be initialized before first use.
 */
struct mtgp64_kernel_status_t {
    uint64_t status[N];
};

/*
 * Texture References.
 */
texture<uint32_t, 1, hipReadModeElementType> tex_param_ref;
texture<uint32_t, 1, hipReadModeElementType> tex_temper_ref;
texture<uint32_t, 1, hipReadModeElementType> tex_double_ref;

/*
 * Generator Parameters.
 */
__constant__ uint32_t pos_tbl[BLOCK_NUM];
__constant__ uint32_t sh1_tbl[BLOCK_NUM];
__constant__ uint32_t sh2_tbl[BLOCK_NUM];
/* high_mask and low_mask should be set by make_constant(), but
 * did not work.
 */
__constant__ uint32_t high_mask = 0xffff8000;
__constant__ uint32_t low_mask =  0x00000000;

/**
 * Shared memory
 * The generator's internal status vector.
 */
__shared__ uint32_t status[2][LARGE_SIZE]; /* 512 * 3 elements, 12288 bytes. */

/**
 * The function of the recursion formula calculation.
 *
 * @param RH 32-bit MSBs of output
 * @param RL 32-bit LSBs of output
 * @param X1H MSBs of the farthest part of state array.
 * @param X1L LSBs of the farthest part of state array.
 * @param X2H MSBs of the second farthest part of state array.
 * @param X2L LSBs of the second farthest part of state array.
 * @param YH MSBs of a part of state array.
 * @param YL LSBs of a part of state array.
 * @param bid block id.
 */
__device__ void para_rec(uint32_t *RH,
			 uint32_t *RL,
			 uint32_t X1H,
			 uint32_t X1L,
			 uint32_t X2H,
			 uint32_t X2L,
			 uint32_t YH,
			 uint32_t YL,
			 int bid) {
    uint32_t XH = (X1H & high_mask) ^ X2H;
    uint32_t XL = (X1L & low_mask) ^ X2L;
    uint32_t MAT;

    XH ^= XH << sh1_tbl[bid];
    XL ^= XL << sh1_tbl[bid];
    YH = XL ^ (YH >> sh2_tbl[bid]);
    YL = XH ^ (YL >> sh2_tbl[bid]);
    MAT = tex1Dfetch(tex_param_ref, bid * 16 + (YL & 0x0f));
    *RH = YH ^ MAT;
    *RL = YL;
}

/**
 * The tempering function.
 *
 * @param VH MSBs of the output value should be tempered.
 * @param VL LSBs of the output value should be tempered.
 * @param TL LSBs of the tempering helper value.
 * @param bid block id.
 * @return the tempered value.
 */
__device__ uint64_t temper(uint32_t VH,
			   uint32_t VL,
			   uint32_t TL,
			   int bid) {
    uint32_t MAT;
    uint64_t r;
    TL ^= TL >> 16;
    TL ^= TL >> 8;
    MAT = tex1Dfetch(tex_temper_ref, bid * 16 + (TL & 0x0f));
    VH ^= MAT;
    r = ((uint64_t)VH << 32) | VL;
    return r;
}

/**
 * The tempering and converting function.
 * By using the presetted table, converting to IEEE format
 * and tempering are done simultaneously.
 *
 * @param VH MSBs of the output value should be tempered.
 * @param VL LSBs of the output value should be tempered.
 * @param TL LSBs of the tempering helper value.
 * @param bid block id.
 * @return the tempered and converted value.
 */
__device__ uint64_t temper_double(uint32_t VH,
				  uint32_t VL,
				  uint32_t TL,
				  int bid) {
    uint32_t MAT;
    uint64_t r;
    TL ^= TL >> 16;
    TL ^= TL >> 8;
    MAT = tex1Dfetch(tex_double_ref, bid * 16 + (TL & 0x0f));
    r = ((uint64_t)VH << 32) | VL;
    r = (r >> 12) ^ ((uint64_t)MAT << 32);
    return r;
}

/**
 * Read the internal state vector from kernel I/O data, and
 * put them into shared memory.
 *
 * @param status shared memory.
 * @param d_status kernel I/O data
 * @param bid block id
 * @param tid thread id
 */
__device__ void status_read(uint32_t status[2][LARGE_SIZE],
			    const mtgp64_kernel_status_t *d_status,
			    int bid,
			    int tid) {
    uint64_t x;

    x = d_status[bid].status[tid];
    status[0][LARGE_SIZE - N + tid] = x >> 32;
    status[1][LARGE_SIZE - N + tid] = x & 0xffffffff;
    if (tid < N - THREAD_NUM) {
	x = d_status[bid].status[THREAD_NUM + tid];
	status[0][LARGE_SIZE - N + THREAD_NUM + tid] = x >> 32;
	status[1][LARGE_SIZE - N + THREAD_NUM + tid] = x & 0xffffffff;
    }
    __syncthreads();
}

/**
 * Read the internal state vector from shared memory, and
 * write them into kernel I/O data.
 *
 * @param status shared memory.
 * @param d_status kernel I/O data
 * @param bid block id
 * @param tid thread id
 */
__device__ void status_write(mtgp64_kernel_status_t *d_status,
			     const uint32_t status[2][LARGE_SIZE],
			     int bid,
			     int tid) {
    uint64_t x;

    x = (uint64_t)status[0][LARGE_SIZE - N + tid] << 32;
    x = x | status[1][LARGE_SIZE - N + tid];
    d_status[bid].status[tid] = x;
    if (tid < N - THREAD_NUM) {
	x = (uint64_t)status[0][4 * THREAD_NUM - N + tid] << 32;
	x = x | status[1][4 * THREAD_NUM - N + tid];
	d_status[bid].status[THREAD_NUM + tid] = x;
    }
    __syncthreads();
}

/**
 * kernel function.
 * This function generates 64-bit unsigned integers in d_data
 *
 * @params d_status kernel I/O data
 * @params d_data output
 * @params size number of output data requested.
 */
__global__ void mtgp64_uint64_kernel(mtgp64_kernel_status_t* d_status,
				     uint64_t* d_data, int size) {
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    int pos = pos_tbl[bid];
    uint32_t YH;
    uint32_t YL;
    uint64_t o;

    // copy status data from global memory to shared memory.
    status_read(status, d_status, bid, tid);

    // main loop
    for (int i = 0; i < size; i += LARGE_SIZE) {

#if defined(DEBUG) && defined(__DEVICE_EMULATION__)
	if ((i == 0) && (bid == 0) && (tid <= 1)) {
	    printf("status[0][LARGE_SIZE - N + tid]:%08x\n",
		   status[0][LARGE_SIZE - N + tid]);
	    printf("status[1][LARGE_SIZE - N + tid]:%08x\n",
		   status[1][LARGE_SIZE - N + tid]);
	    printf("status[0][LARGE_SIZE - N + tid + 1]:%08x\n",
		   status[0][LARGE_SIZE - N + tid + 1]);
	    printf("status[1][LARGE_SIZE - N + tid + 1]:%08x\n",
		   status[1][LARGE_SIZE - N + tid + 1]);
	    printf("status[0][LARGE_SIZE - N + tid + pos]:%08x\n",
		   status[0][LARGE_SIZE - N + tid + pos]);
	    printf("status[1][LARGE_SIZE - N + tid + pos]:%08x\n",
		   status[1][LARGE_SIZE - N + tid + pos]);
	    printf("sh1:%d\n", sh1_tbl[bid]);
	    printf("sh2:%d\n", sh2_tbl[bid]);
	    printf("high_mask:%08x\n", high_mask);
	    printf("low_mask:%08x\n", low_mask);
	    for (int j = 0; j < 16; j++) {
		printf("tbl[%d]:%08x\n", j, param_tbl[0][j]);
	    }
	}
#endif
	para_rec(&YH,
		 &YL,
		 status[0][LARGE_SIZE - N + tid],
		 status[1][LARGE_SIZE - N + tid],
		 status[0][LARGE_SIZE - N + tid + 1],
		 status[1][LARGE_SIZE - N + tid + 1],
		 status[0][LARGE_SIZE - N + tid + pos],
		 status[1][LARGE_SIZE - N + tid + pos],
		 bid);
	status[0][tid] = YH;
	status[1][tid] = YL;
#if defined(DEBUG) && defined(__DEVICE_EMULATION__)
	if ((i == 0) && (bid == 0) && (tid <= 1)) {
	    printf("status[0][tid]:%08x\n",	status[0][tid]);
	    printf("status[1][tid]:%08x\n",	status[1][tid]);
	}
#endif
	o = temper(YH,
		   YL,
		   status[1][LARGE_SIZE - N + tid + pos - 1],
		   bid);
#if defined(DEBUG) && defined(__DEVICE_EMULATION__)
	if ((i == 0) && (bid == 0) && (tid <= 1)) {
	    printf("o:%016" PRIx64 "\n", o);
	}
#endif
	d_data[size * bid + i + tid] = o;
	__syncthreads();
	para_rec(&YH,
		 &YL,
		 status[0][(4 * THREAD_NUM - N + tid) % LARGE_SIZE],
		 status[1][(4 * THREAD_NUM - N + tid) % LARGE_SIZE],
		 status[0][(4 * THREAD_NUM - N + tid + 1) % LARGE_SIZE],
		 status[1][(4 * THREAD_NUM - N + tid + 1) % LARGE_SIZE],
		 status[0][(4 * THREAD_NUM - N + tid + pos) % LARGE_SIZE],
		 status[1][(4 * THREAD_NUM - N + tid + pos) % LARGE_SIZE],
		 bid);
	status[0][tid + THREAD_NUM] = YH;
	status[1][tid + THREAD_NUM] = YL;
	o = temper(YH,
		   YL,
		   status[1][(4 * THREAD_NUM - N + tid + pos - 1) % LARGE_SIZE],
		   bid);
	d_data[size * bid + THREAD_NUM + i + tid] = o;
	__syncthreads();
	para_rec(&YH,
		 &YL,
		 status[0][2 * THREAD_NUM - N + tid],
		 status[1][2 * THREAD_NUM - N + tid],
		 status[0][2 * THREAD_NUM - N + tid + 1],
		 status[1][2 * THREAD_NUM - N + tid + 1],
		 status[0][2 * THREAD_NUM - N + tid + pos],
		 status[1][2 * THREAD_NUM - N + tid + pos],
		 bid);
	status[0][tid + 2 * THREAD_NUM] = YH;
	status[1][tid + 2 * THREAD_NUM] = YL;
	o = temper(YH,
		   YL,
		   status[1][tid + pos - 1 + 2 * THREAD_NUM - N],
		   bid);
	d_data[size * bid + 2 * THREAD_NUM + i + tid] = o;
	__syncthreads();
    }
    // write back status for next call
    status_write(d_status, status, bid, tid);
}

/**
 * kernel function.
 * This function generates double precision floating point numbers in d_data.
 *
 * @params d_status kernel I/O data
 * @params d_data output. IEEE double precision format.
 * @params size number of output data requested.
 */
__global__ void mtgp64_double_kernel(mtgp64_kernel_status_t* d_status,
				     uint64_t* d_data, int size)
{

    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    int pos = pos_tbl[bid];
    uint32_t YH;
    uint32_t YL;
    uint64_t o;

    // copy status data from global memory to shared memory.
    status_read(status, d_status, bid, tid);

    // main loop
    for (int i = 0; i < size; i += LARGE_SIZE) {
	para_rec(&YH,
		 &YL,
		 status[0][LARGE_SIZE - N + tid],
		 status[1][LARGE_SIZE - N + tid],
		 status[0][LARGE_SIZE - N + tid + 1],
		 status[1][LARGE_SIZE - N + tid + 1],
		 status[0][LARGE_SIZE - N + tid + pos],
		 status[1][LARGE_SIZE - N + tid + pos],
		 bid);
	status[0][tid] = YH;
	status[1][tid] = YL;
	o = temper_double(YH,
			  YL,
			  status[1][LARGE_SIZE - N + tid + pos - 1],
			  bid);
	d_data[size * bid + i + tid] = o;
	__syncthreads();
	para_rec(&YH,
		 &YL,
		 status[0][(4 * THREAD_NUM - N + tid) % LARGE_SIZE],
		 status[1][(4 * THREAD_NUM - N + tid) % LARGE_SIZE],
		 status[0][(4 * THREAD_NUM - N + tid + 1) % LARGE_SIZE],
		 status[1][(4 * THREAD_NUM - N + tid + 1) % LARGE_SIZE],
		 status[0][(4 * THREAD_NUM - N + tid + pos) % LARGE_SIZE],
		 status[1][(4 * THREAD_NUM - N + tid + pos) % LARGE_SIZE],
		 bid);
	status[0][tid + THREAD_NUM] = YH;
	status[1][tid + THREAD_NUM] = YL;
	o = temper_double(
	    YH,
	    YL,
	    status[1][(4 * THREAD_NUM - N + tid + pos - 1) % LARGE_SIZE],
	    bid);
	d_data[size * bid + THREAD_NUM + i + tid] = o;
	__syncthreads();
	para_rec(&YH,
		 &YL,
		 status[0][2 * THREAD_NUM - N + tid],
		 status[1][2 * THREAD_NUM - N + tid],
		 status[0][2 * THREAD_NUM - N + tid + 1],
		 status[1][2 * THREAD_NUM - N + tid + 1],
		 status[0][2 * THREAD_NUM - N + tid + pos],
		 status[1][2 * THREAD_NUM - N + tid + pos],
		 bid);
	status[0][tid + 2 * THREAD_NUM] = YH;
	status[1][tid + 2 * THREAD_NUM] = YL;
	o = temper_double(YH,
			  YL,
			  status[1][tid + pos - 1 + 2 * THREAD_NUM - N],
			  bid);
	d_data[size * bid + 2 * THREAD_NUM + i + tid] = o;
	__syncthreads();
    }
    // write back status for next call
    status_write(d_status, status, bid, tid);
}

/**
 * This function sets constants in device memory.
 * @param params input, MTGP64 parameters.
 */
void make_constant(const mtgp64_params_fast_t params[]) {
    const int size1 = sizeof(uint32_t) * BLOCK_NUM;
    uint32_t *h_pos_tbl;
    uint32_t *h_sh1_tbl;
    uint32_t *h_sh2_tbl;
#if 0
    uint32_t *h_high_mask;
    uint32_t *h_low_mask;
#endif
    h_pos_tbl = (uint32_t *)malloc(size1);
    h_sh1_tbl = (uint32_t *)malloc(size1);
    h_sh2_tbl = (uint32_t *)malloc(size1);
#if 0
    h_high_mask = (uint32_t *)malloc(sizeof(uint32_t));
    h_low_mask = (uint32_t *)malloc(sizeof(uint32_t));
#endif
    if (h_pos_tbl == NULL
	|| h_sh1_tbl == NULL
	|| h_sh2_tbl == NULL
#if 0
	|| h_high_mask == NULL
	|| h_low_mask == NULL
#endif
	) {
	printf("failure in allocating host memory for constant table.\n");
	exit(1);
    }
#if 0
    *h_high_mask = params[0].mask >> 32;
    *h_low_mask = params[0].mask & 0xffffffffU;
#endif
    for (int i = 0; i < BLOCK_NUM; i++) {
	h_pos_tbl[i] = params[i].pos;
	h_sh1_tbl[i] = params[i].sh1;
	h_sh2_tbl[i] = params[i].sh2;
    }
    // copy from malloc area only
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(pos_tbl), h_pos_tbl, size1));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(sh1_tbl), h_sh1_tbl, size1));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(sh2_tbl), h_sh2_tbl, size1));
#if 0
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(&high_mask),
				      &h_high_mask, sizeof(uint32_t)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(&low_mask),
				      &h_low_mask, sizeof(uint32_t)));
#endif
    free(h_pos_tbl);
    free(h_sh1_tbl);
    free(h_sh2_tbl);
#if 0
    free(h_high_mask);
    free(h_low_mask);
#endif
}

/**
 * This function sets constants in device memory.
 * @param params input, MTGP64 parameters.
 */
void make_texture(const mtgp64_params_fast_t params[],
		  uint32_t *d_texture_tbl) {
    const int count = BLOCK_NUM * TBL_SIZE;
    const int size = sizeof(uint32_t) * count;
    uint32_t *h_texture_tbl;
    h_texture_tbl = (uint32_t *)malloc(size * 3);
    if (h_texture_tbl == NULL) {
	printf("failure in allocating host memory for texture table.\n");
	exit(1);
    }
    for (int i = 0; i < BLOCK_NUM; i++) {
	for (int j = 0; j < TBL_SIZE; j++) {
	    h_texture_tbl[i * TBL_SIZE + j] = params[i].tbl[j] >> 32;
	    h_texture_tbl[count + i * TBL_SIZE + j]
		= params[i].tmp_tbl[j] >> 32;
	    h_texture_tbl[2 * count + i * TBL_SIZE + j]
		= params[i].dbl_tmp_tbl[j] >> 32;
	}
    }
    CUDA_SAFE_CALL(hipMemcpy(d_texture_tbl, h_texture_tbl, size * 3,
			      hipMemcpyHostToDevice));
    tex_param_ref.filterMode = hipFilterModePoint;
    tex_temper_ref.filterMode = hipFilterModePoint;
    tex_double_ref.filterMode = hipFilterModePoint;
    CUDA_SAFE_CALL(hipBindTexture(0, tex_param_ref, d_texture_tbl, size));
    CUDA_SAFE_CALL(hipBindTexture(0, tex_temper_ref,
				   d_texture_tbl + count, size));
    CUDA_SAFE_CALL(hipBindTexture(0, tex_double_ref,
				   d_texture_tbl + count * 2, size));
    free(h_texture_tbl);
}

/**
 * This function initializes kernel I/O data.
 * @param d_status output kernel I/O data.
 * @param params MTGP64 parameters. needed for the initialization.
 */
void make_kernel_data(mtgp64_kernel_status_t *d_status,
		     mtgp64_params_fast_t params[]) {
    mtgp64_kernel_status_t* h_status = (mtgp64_kernel_status_t *) malloc(
	sizeof(mtgp64_kernel_status_t) * BLOCK_NUM);

    if (h_status == NULL) {
	printf("failure in allocating host memory for kernel I/O data.\n");
	exit(8);
    }
    for (int i = 0; i < BLOCK_NUM; i++) {
	mtgp64_init_state(&(h_status[i].status[0]), &params[i], i + 1);
    }
#if defined(DEBUG)
    printf("h_status[0].status[0]:%016"PRIx64"\n", h_status[0].status[0]);
    printf("h_status[0].status[0]:%016"PRIx64"\n", h_status[0].status[1]);
    printf("h_status[0].status[0]:%016"PRIx64"\n", h_status[0].status[2]);
    printf("h_status[0].status[0]:%016"PRIx64"\n", h_status[0].status[3]);
#endif
    CUDA_SAFE_CALL(hipMemcpy(d_status,
			      h_status,
			      sizeof(mtgp64_kernel_status_t) * BLOCK_NUM,
			      hipMemcpyHostToDevice));
    free(h_status);
}

/**
 * This function is used to compare the outputs with C program's.
 * @param array data to be printed.
 * @param size size of array.
 * @param block number of blocks.
 */
void print_double_array(const double array[], int size, int block) {
    int b = size / block;

    for (int j = 0; j < 3; j += 3) {
	printf("%.18f %.18f %.18f\n",
	       array[j], array[j + 1], array[j + 2]);
    }
    for (int i = 1; i < block; i++) {
	for (int j = -3; j < 4; j += 3) {
	    printf("%.18f %.18f %.18f\n",
		   array[b * i + j],
		   array[b * i + j + 1],
		   array[b * i + j + 2]);
	}
    }
    for (int j = -3; j < 0; j += 3) {
	printf("%.18f %.18f %.18f\n",
	       array[size + j],
	       array[size + j + 1],
	       array[size + j + 2]);
    }
}

/**
 * This function is used to compare the outputs with C program's.
 * @param array data to be printed.
 * @param size size of array.
 * @param block number of blocks.
 */
void print_uint64_array(uint64_t array[], int size, int block) {
    int b = size / block;

    for (int j = 0; j < 3; j += 3) {
	printf("%20" PRIu64 " %20" PRIu64 " %20" PRIu64 "\n",
	       array[j], array[j + 1], array[j + 2]);
    }
    for (int i = 1; i < block; i++) {
	for (int j = -3; j < 3; j += 3) {
	    printf("%20" PRIu64 " %20" PRIu64 " %20" PRIu64 "\n",
		   array[b * i + j],
		   array[b * i + j + 1],
		   array[b * i + j + 2]);
	}
    }
    for (int j = -3; j < 0; j += 3) {
	printf("%20" PRIu64 " %20" PRIu64 " %20" PRIu64 "\n",
	       array[size + j],
	       array[size + j + 1],
	       array[size + j + 2]);
    }
}

/**
 * host function.
 * This function calls corresponding kernel function.
 *
 * @param d_status kernel I/O data.
 * @param num_data number of data to be generated.
 */
void make_uint64_random(mtgp64_kernel_status_t* d_status, int num_data) {
    uint64_t* d_data;
    unsigned int timer = 0;
    uint64_t* h_data;
    hipError_t e;
    float gputime;

    printf("generating 64-bit unsigned random numbers.\n");
    CUDA_SAFE_CALL(hipMalloc((void**)&d_data, sizeof(uint64_t) * num_data));
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    h_data = (uint64_t *) malloc(sizeof(uint64_t) * num_data);
    if (h_data == NULL) {
	printf("failure in allocating host memory for output data.\n");
	exit(1);
    }
    CUT_SAFE_CALL(cutStartTimer(timer));
    if (hipGetLastError() != hipSuccess) {
	printf("error has been occured before kernel call.\n");
	exit(1);
    }

    /* kernel call */
    mtgp64_uint64_kernel<<< BLOCK_NUM, THREAD_NUM>>>(
	d_status, d_data, num_data / BLOCK_NUM);
    hipDeviceSynchronize();

    e = hipGetLastError();
    if (e != hipSuccess) {
	printf("failure in kernel call.\n%s\n", hipGetErrorString(e));
	exit(1);
    }
    CUT_SAFE_CALL(cutStopTimer(timer));
    CUDA_SAFE_CALL(
	hipMemcpy(h_data,
		   d_data,
		   sizeof(uint64_t) * num_data,
		   hipMemcpyDeviceToHost));
    gputime = cutGetTimerValue(timer);
    print_uint64_array(h_data, num_data, BLOCK_NUM);
    printf("generated numbers: %d\n", num_data);
    printf("Processing time: %f (ms)\n", gputime);
    printf("Samples per second: %E \n", num_data / (gputime * 0.001));
    CUT_SAFE_CALL(cutDeleteTimer(timer));
    //free memories
    free(h_data);
    CUDA_SAFE_CALL(hipFree(d_data));
}

/**
 * host function.
 * This function calls corresponding kernel function.
 *
 * @param d_status kernel I/O data.
 * @param num_data number of data to be generated.
 */
void make_double_random(mtgp64_kernel_status_t* d_status, int num_data) {
    uint64_t* d_data;
    unsigned int timer = 0;
    double* h_data;
    hipError_t e;
    float gputime;

    printf("generating double precision floating point random numbers.\n");
    CUDA_SAFE_CALL(hipMalloc((void**)&d_data, sizeof(uint64_t) * num_data));
    CUT_SAFE_CALL(cutCreateTimer(&timer));
    h_data = (double *) malloc(sizeof(double) * num_data);
    if (h_data == NULL) {
	printf("failure in allocating host memory for output data.\n");
	exit(1);
    }
    CUT_SAFE_CALL(cutStartTimer(timer));
    if (hipGetLastError() != hipSuccess) {
	printf("error has been occured before kernel call.\n");
	exit(1);
    }

    /* kernel call */
    mtgp64_double_kernel<<< BLOCK_NUM, THREAD_NUM >>>(
	d_status, d_data, num_data / BLOCK_NUM);
    hipDeviceSynchronize();

    e = hipGetLastError();
    if (e != hipSuccess) {
	printf("failure in kernel call.\n%s\n", hipGetErrorString(e));
	exit(1);
    }
    CUT_SAFE_CALL(cutStopTimer(timer));
    CUDA_SAFE_CALL(
	hipMemcpy(h_data,
		   d_data,
		   sizeof(uint64_t) * num_data,
		   hipMemcpyDeviceToHost));
    gputime = cutGetTimerValue(timer);
    print_double_array(h_data, num_data, BLOCK_NUM);
    printf("generated numbers: %d\n", num_data);
    printf("Processing time: %f (ms)\n", gputime);
    printf("Samples per second: %E \n", num_data / (gputime * 0.001));
    CUT_SAFE_CALL(cutDeleteTimer(timer));
    //free memories
    free(h_data);
    CUDA_SAFE_CALL(hipFree(d_data));
}

int main(int argc, char** argv)
{
    // LARGE_SIZE is a multiple of 16
    int num_data = 10000000;
    int num_unit = LARGE_SIZE * BLOCK_NUM;
    int r;
    mtgp64_kernel_status_t* d_status;
    uint32_t *d_texture;

    CUT_DEVICE_INIT(argc, argv);
    CUDA_SAFE_CALL(hipMalloc((void**)&d_status,
			      sizeof(mtgp64_kernel_status_t) * BLOCK_NUM));
    CUDA_SAFE_CALL(hipMalloc((void**)&d_texture,
			      sizeof(uint32_t) * BLOCK_NUM * TBL_SIZE * 3));
    if (argc >= 2) {
	errno = 0;
	num_data = strtol(argv[1], NULL, 10);
	if (errno) {
	    printf("%s number_of_output\n", argv[0]);
	    return 1;
	}
    } else {
	printf("%s number_of_output\n", argv[0]);
	return 1;
    }
    r = num_data % num_unit;
    if (r != 0) {
	num_data = num_data + num_unit - r;
    }
    make_constant(mtgp64_params_fast_44497);
    make_texture(mtgp64_params_fast_44497, d_texture);
    make_kernel_data(d_status, mtgp64_params_fast_44497);
    make_uint64_random(d_status, num_data);
    make_double_random(d_status, num_data);

    //finalize
    CUDA_SAFE_CALL(hipFree(d_status));
    CUDA_SAFE_CALL(hipFree(d_texture));
    CUT_EXIT(argc, argv);
}
