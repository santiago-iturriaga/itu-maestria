#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include "config.h"
#include "cuda-util.h"
#include "billionga.h"

#define INIT_PROB_VECTOR_VALUE      0.5
#define INIT_PROB_VECTOR_BLOCKS     128
#define INIT_PROB_VECTOR_THREADS    256

#define SHOW_PROB_VECTOR_STARTING_BITS      10
#define SHOW_PROB_VECTOR_ENDING_BITS        10

#define SUM_PROB_VECTOR_BLOCKS      128
#define SUM_PROB_VECTOR_THREADS     512
#define SUM_PROB_VECTOR_SHARED_MEM  512

/*
 * Establece el valor de todos los elementos de un vector a "value"
 */
__global__ void kern_vector_set(float *gpu_prob_vector, int max_size, float value) {
    int bits_per_loop = gridDim.x * blockDim.x;
    
    int loop_count = max_size / bits_per_loop;
    if (max_size % bits_per_loop > 0) loop_count++;
        
    for (int i = 0; i < loop_count; i++) {
        int current_position = (i * bits_per_loop) + (blockIdx.x * blockDim.x + threadIdx.x);
        
        if (current_position < max_size) {
            gpu_prob_vector[current_position] = INIT_PROB_VECTOR_VALUE;
        }
        
        __syncthreads();
    }
}

/*__global__ void kern_sum_prob_vector(float *gpu_partial_sum, float *gpu_prob_vectors, 
    int max_size, int starting_position) {
    
    __shared__ float local_memory[SUM_PROB_VECTOR_SHARED_MEM];
    
    int current_position = starting_position + (blockIdx.x * blockDim.x + threadIdx.x);
    local_memory[threadIdx.x] = gpu_prob_vectors[current_position];
    local_memory[threadIdx.x] += gpu_prob_vectors[current_position + blockDim.x];
    
    unsigned int t = threadIdx.x;
    for (unsigned int stride = blockDim.x; stride > 1; stride >> 1)
    {
        __syncthreads();
        if (t < stride) gpu_partial_sum[t] += gpu_partial_sum[t+stride];
    }
    
    __syncthreads();
    if (t == 0) gpu_prob_vectors[starting_position] = gpu_partial_sum[0];
}*/

__global__ void kern_sum_prob_vector(float *g_idata, float *g_odata, unsigned int max_size)
{
    __shared__ float sdata[SUM_PROB_VECTOR_SHARED_MEM];

    unsigned int tid = threadIdx.x;
    
    unsigned int adds_per_loop = gridDim.x * blockDim.x * 2;
    unsigned int loops_count = max_size / adds_per_loop;
    if (max_size % adds_per_loop > 0) loops_count++;

    unsigned int starting_position;
    
    for (unsigned int loop = 0; loop < loops_count; loop++) {
        // Perform first level of reduction, reading from global memory, writing to shared memory
        starting_position = adds_per_loop * loop;
        
        unsigned int i = starting_position + (blockIdx.x * (blockDim.x * 2) + threadIdx.x);

        float mySum = (i < max_size) ? g_idata[i] : 0;
        if (i + blockDim.x < max_size) 
            mySum += g_idata[i + blockDim.x];  

        sdata[tid] = mySum;
        __syncthreads();

        // do reduction in shared mem
        for(unsigned int s = blockDim.x/2; s > 0; s >>= 1) 
        {
            if (tid < s) 
            {
                sdata[tid] = mySum = mySum + sdata[tid + s];
            }
            __syncthreads();
        }

        // write result for this block to global mem 
        if (tid == 0) g_odata[blockIdx.x] += sdata[0];
    
        __syncthreads();
    }
}

// Paso 1 del algoritmo.
void bga_initialization(struct bga_state *state, long number_of_bits, int number_of_samples) {
    state->number_of_bits = number_of_bits;
    state->number_of_samples = number_of_samples;
    
    // === Pido la memoria =============================================================
    #if defined(INFO) || defined(DEBUG)
    fprintf(stdout, "[INFO] === Solicitando memoria =======================\n");
    #endif

    hipError_t error;

    #if defined(DEBUG)
    float gputime;
    hipEvent_t start;
    hipEvent_t end;
    
    ccudaEventCreate(&start);
    ccudaEventCreate(&end);

    ccudaEventRecord(start, 0);
    #endif
    
    if (state->number_of_bits > MAX_PROB_VECTOR_BITS) {
        state->number_of_prob_vectors = state->number_of_bits / MAX_PROB_VECTOR_BITS;
        state->last_prob_vector_bit_count = state->number_of_bits % MAX_PROB_VECTOR_BITS;
        
        if (state->last_prob_vector_bit_count == 0) {
            state->last_prob_vector_bit_count = MAX_PROB_VECTOR_BITS;
        } else {
            state->number_of_prob_vectors++;
        }
    } else {
        state->number_of_prob_vectors = 1;
        state->last_prob_vector_bit_count = state->number_of_bits;
    }
      
    #ifdef INFO
    fprintf(stdout, "[INFO] Requesting a size %d prob_vector_size CPU memory\n", state->number_of_prob_vectors);
    #endif
      
    size_t prob_vector_array_size = sizeof(float*) * state->number_of_prob_vectors;
    state->gpu_prob_vectors = (float**)malloc(prob_vector_array_size);
    if (!state->gpu_prob_vectors) {
        fprintf(stderr, "[ERROR] Requesting CPU memory for the prob_vector\n");
        exit(EXIT_FAILURE);
    }

    for (int prob_vector_number = 0; prob_vector_number < state->number_of_prob_vectors; prob_vector_number++) {
        int current_prob_vector_number_of_bits = MAX_PROB_VECTOR_BITS;
        if (prob_vector_number + 1 == state->number_of_prob_vectors) {
            current_prob_vector_number_of_bits = state->last_prob_vector_bit_count;
        }

        #ifdef INFO
        fprintf(stdout, "[INFO] > Requesting %d bits GPU memory for prob_vector %d\n", current_prob_vector_number_of_bits, prob_vector_number);
        #endif

        size_t prob_vector_size = sizeof(float) * current_prob_vector_number_of_bits;
        error = hipMalloc((void**)&(state->gpu_prob_vectors[prob_vector_number]), prob_vector_size);
        
        if (error != hipSuccess) {
            fprintf(stderr, "[ERROR] Requesting GPU memory for prob_vector_number[%d]\n", prob_vector_number);
            exit(EXIT_FAILURE);
        }
    }
       
    #ifdef INFO
    fprintf(stdout, "[INFO] Requesting a size %d samples CPU memory\n", state->number_of_samples);
    #endif
    
    size_t samples_array_size = sizeof(float*) * state->number_of_samples;
    state->gpu_samples = (float***)malloc(samples_array_size);
    if (!state->gpu_samples) {
        fprintf(stderr, "[ERROR] Requesting samples_fitness CPU memory\n");
        exit(EXIT_FAILURE);
    }
   
    for (int sample_number = 0; sample_number < state->number_of_samples; sample_number++) {
        #ifdef INFO
        fprintf(stdout, "[INFO] > Requesting CPU memory for sample %d vectors array\n", sample_number);
        #endif

        size_t samples_vector_array_size = sizeof(float*) * state->number_of_prob_vectors;
        state->gpu_samples[sample_number] = (float**)malloc(samples_vector_array_size);
        if (!state->gpu_samples) {
            fprintf(stderr, "[ERROR] > Requesting CPU memory for sample_vector_array[%d]\n", sample_number);
            exit(EXIT_FAILURE);
        }
    
        for (int prob_vector_number = 0; prob_vector_number < state->number_of_prob_vectors; prob_vector_number++) {
            int current_prob_vector_number_of_bits = MAX_PROB_VECTOR_BITS;
            if (prob_vector_number + 1 == state->number_of_prob_vectors) {
                current_prob_vector_number_of_bits = state->last_prob_vector_bit_count;
            }
            size_t sample_vector_size = sizeof(char) * (current_prob_vector_number_of_bits / 8);
            assert(current_prob_vector_number_of_bits % 8 == 0);

            #ifdef INFO
            fprintf(stdout, "[INFO] > Requesting sample %d GPU memory for vector %d\n", sample_number, prob_vector_number);
            #endif

            error = hipMalloc((void**)&(state->gpu_samples[sample_number][prob_vector_number]), sample_vector_size);
            if (error != hipSuccess) {
                fprintf(stderr, "[ERROR] > Requesting GPU memory for sample_number[%d]\n", sample_number);
                exit(EXIT_FAILURE);
            }
        }
    }
    
    #ifdef INFO
    fprintf(stdout, "[INFO] Requesting samples_fitness CPU memory\n");
    #endif
    
    size_t samples_fitness_size = sizeof(long*) * state->number_of_samples;
    state->gpu_samples_fitness = (long**)malloc(samples_fitness_size);
    if (!state->gpu_samples_fitness) {
        fprintf(stderr, "[ERROR] > Requesting CPU memory for samples_fitness_size\n");
        exit(EXIT_FAILURE);
    }
        
    for (int sample_number = 0; sample_number < state->number_of_samples; sample_number++) {
        #ifdef INFO
        fprintf(stdout, "[INFO] > Requesting GPU memory for sample %d fitness vector array\n", sample_number);
        #endif

        size_t samples_fitness_vector_size = sizeof(long) * state->number_of_prob_vectors;
        error = hipMalloc((void**)&(state->gpu_samples_fitness[sample_number]), samples_fitness_vector_size);
        if (error != hipSuccess) {
            fprintf(stderr, "[ERROR] Requesting memory for samples_fitness_vector_size[%d]\n", sample_number);
            exit(EXIT_FAILURE);
        }
    }
    
    #if defined(DEBUG)
    ccudaEventRecord(end, 0);
    ccudaEventSynchronize(end);
    ccudaEventElapsedTime(&gputime, start, end);
    fprintf(stdout, "TIME] Processing time: %f (ms)\n", gputime);
    #endif
    
    // === Inicializo el vector de probabilidades ============================================
    #if defined(INFO) || defined(DEBUG)
    fprintf(stdout, "[INFO] === Inicializando memoria =======================\n");
    #endif

    #if defined(DEBUG)   
    ccudaEventCreate(&start);
    ccudaEventCreate(&end);

    ccudaEventRecord(start, 0);
    #endif
    
    for (int prob_vector_number = 0; prob_vector_number < state->number_of_prob_vectors; prob_vector_number++) {
        int current_prob_vector_number_of_bits = MAX_PROB_VECTOR_BITS;
        if (prob_vector_number + 1 == state->number_of_prob_vectors) {
            current_prob_vector_number_of_bits = state->last_prob_vector_bit_count;
        }

        #ifdef INFO
        fprintf(stdout, "[INFO] Inicializando GPU memory of prob_vector %d (%d bits)\n", 
            prob_vector_number, current_prob_vector_number_of_bits);
        #endif

        kern_vector_set<<< INIT_PROB_VECTOR_BLOCKS, INIT_PROB_VECTOR_THREADS >>>(
            state->gpu_prob_vectors[prob_vector_number], 
            current_prob_vector_number_of_bits, INIT_PROB_VECTOR_VALUE);
    }
    
    #if defined(DEBUG)
    ccudaEventRecord(end, 0);
    ccudaEventSynchronize(end);
    ccudaEventElapsedTime(&gputime, start, end);
    fprintf(stdout, "[TIME] Processing time: %f (ms)\n", gputime);
        
    ccudaEventDestroy(start);
    ccudaEventDestroy(end);
    #endif
}

void bga_show_prob_vector_state(struct bga_state *state) {
    double accumulated_probability = 0.0;

    #if defined(DEBUG)
    float gputime;
    hipEvent_t start;
    hipEvent_t end;
    
    ccudaEventCreate(&start);
    ccudaEventCreate(&end);

    ccudaEventRecord(start, 0);
    #endif

    fprintf(stdout, "[INFO] === Probability vector status =======================\n");
    const int max_partial_mem = SUM_PROB_VECTOR_BLOCKS * SUM_PROB_VECTOR_THREADS;
       
    float *partial_sum;
    ccudaMalloc((void**)&(partial_sum), sizeof(float) * max_partial_mem);

    kern_vector_set<<< INIT_PROB_VECTOR_BLOCKS, INIT_PROB_VECTOR_THREADS >>>(
        partial_sum, max_partial_mem, 0.0);

    fprintf(stdout, "[INFO] Prob. vector sample:");
    
    for (int prob_vector_number = 0; prob_vector_number < state->number_of_prob_vectors; prob_vector_number++) {
        int current_prob_vector_number_of_bits = MAX_PROB_VECTOR_BITS;
        
        if (prob_vector_number + 1 == state->number_of_prob_vectors) {
            if (prob_vector_number != 0) {
                fprintf(stdout, "...");
            }
            
            current_prob_vector_number_of_bits = state->last_prob_vector_bit_count;
            
            int probs_to_show_count = SHOW_PROB_VECTOR_STARTING_BITS;
            if (current_prob_vector_number_of_bits < SHOW_PROB_VECTOR_STARTING_BITS) 
                probs_to_show_count = current_prob_vector_number_of_bits;
            
            float *probs_to_show = (float*)malloc(sizeof(float) * probs_to_show_count);
            ccudaMemcpy(probs_to_show, state->gpu_prob_vectors[prob_vector_number], 
                sizeof(uint32_t) * probs_to_show_count, hipMemcpyDeviceToHost);
            
            for (int i = 0; i < probs_to_show_count; i++) {
                fprintf(stdout, " %.4f", probs_to_show[i]);
            }
        } if (prob_vector_number == 0) {
            int probs_to_show_count = SHOW_PROB_VECTOR_STARTING_BITS;
            if (MAX_PROB_VECTOR_BITS < SHOW_PROB_VECTOR_STARTING_BITS) 
                probs_to_show_count = MAX_PROB_VECTOR_BITS;
            
            float *probs_to_show = (float*)malloc(sizeof(float) * probs_to_show_count);
            ccudaMemcpy(probs_to_show, state->gpu_prob_vectors[prob_vector_number], 
                sizeof(uint32_t) * probs_to_show_count, hipMemcpyDeviceToHost);
            
            for (int i = 0; i < probs_to_show_count; i++) {
                fprintf(stdout, " %.4f", probs_to_show[i]);
            }
        }
                   
        kern_sum_prob_vector<<< SUM_PROB_VECTOR_BLOCKS, SUM_PROB_VECTOR_THREADS >>>( 
            state->gpu_prob_vectors[prob_vector_number], partial_sum,
            current_prob_vector_number_of_bits);
    }
    
    kern_sum_prob_vector<<< SUM_PROB_VECTOR_BLOCKS, SUM_PROB_VECTOR_THREADS >>>( 
        partial_sum, partial_sum, max_partial_mem);
    
    ccudaMemcpy(&accumulated_probability, partial_sum, sizeof(float), hipMemcpyDeviceToHost);
    
    fprintf(stdout, "\n[INFO] Prob. vector accumulated probability: %.4f\n", accumulated_probability);
    
    #if defined(DEBUG)
    ccudaEventRecord(end, 0);
    ccudaEventSynchronize(end);
    ccudaEventElapsedTime(&gputime, start, end);
    fprintf(stdout, "[TIME] Processing time: %f (ms)\n", gputime);
        
    ccudaEventDestroy(start);
    ccudaEventDestroy(end);
    #endif
}

// Paso 2 del algoritmo.
void bga_model_sampling_mt(struct bga_state *state) {
    //mtgp32_generate_float(&mt_status);
    //mtgp32_print_generated_floats(&mt_status);

    //mtgp32_generate_uint32(&mt_status);
    //mtgp32_print_generated_uint32(&mt_status);
}

// Paso 3 del algoritmo.
void bga_evaluation(struct bga_state *state) {
}

// Paso 4 y 5 del algoritmo.
void bga_model_update(struct bga_state *state) {
}

// Libera la memoria pedida para de estado.
void bga_free(struct bga_state *state) {
    #ifdef INFO
    fprintf(stdout, "[INFO] Freeing memory\n");
    #endif

    for (int vector_number = 0; vector_number < state->number_of_prob_vectors; vector_number++) {
        hipFree(state->gpu_prob_vectors[vector_number]);
    }
    
    free(state->gpu_prob_vectors);
    
    for (int vector_number = 0; vector_number < state->number_of_prob_vectors; vector_number++) {
        hipFree(state->gpu_prob_vectors[vector_number]);
    }

    for (int sample_number = 0; sample_number < state->number_of_samples; sample_number++) {
        for (int vector_number = 0; vector_number < state->number_of_prob_vectors; vector_number++) {
            hipFree(state->gpu_samples[sample_number][vector_number]);
        }
        free(state->gpu_samples[sample_number]);
    }
    free(state->gpu_samples);

    for (int sample_number = 0; sample_number < state->number_of_samples; sample_number++) {
        hipFree(state->gpu_samples_fitness[sample_number]);
    }
    free(state->gpu_samples_fitness);   
}
