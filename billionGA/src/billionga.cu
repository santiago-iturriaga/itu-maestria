#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include "config.h"
#include "cuda-util.h"
#include "billionga.h"

#define VECTOR_SET_BLOCKS       128
#define VECTOR_SET_THREADS      256

#define VECTOR_SUM_BLOCKS       128
#define VECTOR_SUM_THREADS      512
#define VECTOR_SUM_SHARED_MEM   512

#define INIT_PROB_VECTOR_VALUE              0.5
#define SHOW_PROB_VECTOR_STARTING_BITS      10
#define SHOW_PROB_VECTOR_ENDING_BITS        10

/*
 * Establece el valor de todos los elementos de un vector a "value".
 */
__global__ void kern_vector_set(float *gpu_vector, int size, float value) {
    int bits_per_loop = gridDim.x * blockDim.x;
    
    int loop_count = size / bits_per_loop;
    if (size % bits_per_loop > 0) loop_count++;
        
    for (int i = 0; i < loop_count; i++) {
        int current_position = (i * bits_per_loop) + (blockIdx.x * blockDim.x + threadIdx.x);
        
        if (current_position < size) {
            gpu_vector[current_position] = value;
        }
        
        __syncthreads();
    }
}

void vector_sum_init(float **partial_sum) {      
    ccudaMalloc((void**)partial_sum, sizeof(float) * VECTOR_SUM_BLOCKS);

    kern_vector_set<<< 1, VECTOR_SUM_BLOCKS >>>(
        *partial_sum, VECTOR_SUM_BLOCKS, 0.0);
}

float vector_sum_free(float *partial_sum) {
    float accumulated_probability = 0.0;
    
    float *cpu_partial_sum;
    cpu_partial_sum = (float*)malloc(sizeof(float) * VECTOR_SUM_BLOCKS);
    
    ccudaMemcpy(cpu_partial_sum, partial_sum, sizeof(float) * VECTOR_SUM_BLOCKS, hipMemcpyDeviceToHost);
    for (int i = 0; i < VECTOR_SUM_BLOCKS; i++) {
        //fprintf(stdout, "%f ", cpu_partial_sum[i]);
        accumulated_probability += cpu_partial_sum[i];
    }

    ccudaFree(partial_sum);
    return accumulated_probability;
}

/*
 * Reduce un array sumando cada uno de sus elementos.
 * gpu_output_data debe tener un elemento por bloque del kernel.
 */
__global__ void kern_vector_sum(float *gpu_input_data, float *gpu_output_data, unsigned int size)
{
    __shared__ float sdata[VECTOR_SUM_SHARED_MEM];

    unsigned int tid = threadIdx.x;
    
    unsigned int adds_per_loop = gridDim.x * blockDim.x * 2;
    unsigned int loops_count = size / adds_per_loop;
    if (size % adds_per_loop > 0) loops_count++;

    unsigned int starting_position;
    
    for (unsigned int loop = 0; loop < loops_count; loop++) {
        // Perform first level of reduction, reading from global memory, writing to shared memory
        starting_position = adds_per_loop * loop;
        
        unsigned int i = starting_position + (blockIdx.x * (blockDim.x * 2) + threadIdx.x);

        float mySum;
        if (i < size) {
            mySum = gpu_input_data[i];
            
            if (i + blockDim.x < size) {
                mySum += gpu_input_data[i + blockDim.x];  
            }
        } else {
            mySum = 0;
        }

        sdata[tid] = mySum;
        __syncthreads();

        // do reduction in shared mem
        for(unsigned int s = blockDim.x/2; s > 0; s >>= 1) 
        {
            if (tid < s) 
            {
                sdata[tid] = mySum = mySum + sdata[tid + s];
            }
            __syncthreads();
        }

        // write result for this block to global mem 
        if (tid == 0) gpu_output_data[blockIdx.x] += sdata[0];
    
        __syncthreads();
    }
}

// Paso 1 del algoritmo.
void bga_initialization(struct bga_state *state, long number_of_bits, int number_of_samples) {
    state->number_of_bits = number_of_bits;
    state->number_of_samples = number_of_samples;
    
    // === Pido la memoria =============================================================
    #if defined(INFO) || defined(DEBUG)
    fprintf(stdout, "[INFO] === Solicitando memoria =======================\n");
    #endif

    hipError_t error;

    #if defined(DEBUG)
    float gputime;
    hipEvent_t start;
    hipEvent_t end;
    
    ccudaEventCreate(&start);
    ccudaEventCreate(&end);

    ccudaEventRecord(start, 0);
    #endif
    
    if (state->number_of_bits > MAX_PROB_VECTOR_BITS) {
        state->number_of_prob_vectors = state->number_of_bits / MAX_PROB_VECTOR_BITS;
        state->last_prob_vector_bit_count = state->number_of_bits % MAX_PROB_VECTOR_BITS;
        
        if (state->last_prob_vector_bit_count == 0) {
            state->last_prob_vector_bit_count = MAX_PROB_VECTOR_BITS;
        } else {
            state->number_of_prob_vectors++;
        }
    } else {
        state->number_of_prob_vectors = 1;
        state->last_prob_vector_bit_count = state->number_of_bits;
    }
      
    #ifdef INFO
    fprintf(stdout, "[INFO] Requesting a size %d prob_vector_size CPU memory\n", state->number_of_prob_vectors);
    #endif
      
    size_t prob_vector_array_size = sizeof(float*) * state->number_of_prob_vectors;
    state->gpu_prob_vectors = (float**)malloc(prob_vector_array_size);
    if (!state->gpu_prob_vectors) {
        fprintf(stderr, "[ERROR] Requesting CPU memory for the prob_vector\n");
        exit(EXIT_FAILURE);
    }

    for (int prob_vector_number = 0; prob_vector_number < state->number_of_prob_vectors; prob_vector_number++) {
        int current_prob_vector_number_of_bits = MAX_PROB_VECTOR_BITS;
        if (prob_vector_number + 1 == state->number_of_prob_vectors) {
            current_prob_vector_number_of_bits = state->last_prob_vector_bit_count;
        }

        #ifdef INFO
        fprintf(stdout, "[INFO] > Requesting %d bits GPU memory for prob_vector %d\n", 
            current_prob_vector_number_of_bits, prob_vector_number);
        #endif

        size_t prob_vector_size = sizeof(float) * current_prob_vector_number_of_bits;
        error = hipMalloc((void**)&(state->gpu_prob_vectors[prob_vector_number]), prob_vector_size);
        
        if (error != hipSuccess) {
            fprintf(stderr, "[ERROR] Requesting GPU memory for prob_vector_number[%d]\n", prob_vector_number);
            exit(EXIT_FAILURE);
        }
    }
       
    #ifdef INFO
    fprintf(stdout, "[INFO] Requesting a size %d samples CPU memory\n", state->number_of_samples);
    #endif
    
    size_t samples_array_size = sizeof(float*) * state->number_of_samples;
    state->gpu_samples = (float***)malloc(samples_array_size);
    if (!state->gpu_samples) {
        fprintf(stderr, "[ERROR] Requesting samples_fitness CPU memory\n");
        exit(EXIT_FAILURE);
    }
   
    for (int sample_number = 0; sample_number < state->number_of_samples; sample_number++) {
        #ifdef INFO
        fprintf(stdout, "[INFO] > Requesting CPU memory for sample %d vectors array\n", sample_number);
        #endif

        size_t samples_vector_array_size = sizeof(float*) * state->number_of_prob_vectors;
        state->gpu_samples[sample_number] = (float**)malloc(samples_vector_array_size);
        if (!state->gpu_samples) {
            fprintf(stderr, "[ERROR] > Requesting CPU memory for sample_vector_array[%d]\n", sample_number);
            exit(EXIT_FAILURE);
        }
    
        for (int prob_vector_number = 0; prob_vector_number < state->number_of_prob_vectors; prob_vector_number++) {
            int current_prob_vector_number_of_bits = MAX_PROB_VECTOR_BITS;
            if (prob_vector_number + 1 == state->number_of_prob_vectors) {
                current_prob_vector_number_of_bits = state->last_prob_vector_bit_count;
            }
            size_t sample_vector_size = sizeof(char) * (current_prob_vector_number_of_bits / 8);
            assert(current_prob_vector_number_of_bits % 8 == 0);

            #ifdef INFO
            fprintf(stdout, "[INFO] > Requesting sample %d GPU memory for vector %d\n", sample_number, prob_vector_number);
            #endif

            error = hipMalloc((void**)&(state->gpu_samples[sample_number][prob_vector_number]), sample_vector_size);
            if (error != hipSuccess) {
                fprintf(stderr, "[ERROR] > Requesting GPU memory for sample_number[%d]\n", sample_number);
                exit(EXIT_FAILURE);
            }
        }
    }
    
    #ifdef INFO
    fprintf(stdout, "[INFO] Requesting samples_fitness CPU memory\n");
    #endif
    
    size_t samples_fitness_size = sizeof(long*) * state->number_of_samples;
    state->gpu_samples_fitness = (long**)malloc(samples_fitness_size);
    if (!state->gpu_samples_fitness) {
        fprintf(stderr, "[ERROR] > Requesting CPU memory for samples_fitness_size\n");
        exit(EXIT_FAILURE);
    }
        
    for (int sample_number = 0; sample_number < state->number_of_samples; sample_number++) {
        #ifdef INFO
        fprintf(stdout, "[INFO] > Requesting GPU memory for sample %d fitness vector array\n", sample_number);
        #endif

        size_t samples_fitness_vector_size = sizeof(long) * state->number_of_prob_vectors;
        error = hipMalloc((void**)&(state->gpu_samples_fitness[sample_number]), samples_fitness_vector_size);
        if (error != hipSuccess) {
            fprintf(stderr, "[ERROR] Requesting memory for samples_fitness_vector_size[%d]\n", sample_number);
            exit(EXIT_FAILURE);
        }
    }
    
    #if defined(DEBUG)
    ccudaEventRecord(end, 0);
    ccudaEventSynchronize(end);
    ccudaEventElapsedTime(&gputime, start, end);
    fprintf(stdout, "TIME] Processing time: %f (ms)\n", gputime);
    #endif
    
    // === Inicializo el vector de probabilidades ============================================
    #if defined(INFO) || defined(DEBUG)
    fprintf(stdout, "[INFO] === Inicializando memoria =======================\n");
    #endif

    #if defined(DEBUG)   
    ccudaEventCreate(&start);
    ccudaEventCreate(&end);

    ccudaEventRecord(start, 0);
    #endif
    
    for (int prob_vector_number = 0; prob_vector_number < state->number_of_prob_vectors; prob_vector_number++) {
        int current_prob_vector_number_of_bits = MAX_PROB_VECTOR_BITS;
        if (prob_vector_number + 1 == state->number_of_prob_vectors) {
            current_prob_vector_number_of_bits = state->last_prob_vector_bit_count;
        }

        #ifdef INFO
        fprintf(stdout, "[INFO] Inicializando GPU memory of prob_vector %d (%d bits)\n", 
            prob_vector_number, current_prob_vector_number_of_bits);
        #endif

        kern_vector_set<<< VECTOR_SET_BLOCKS, VECTOR_SET_THREADS >>>(
            state->gpu_prob_vectors[prob_vector_number], 
            current_prob_vector_number_of_bits, INIT_PROB_VECTOR_VALUE);
    }
    
    #if defined(DEBUG)
    ccudaEventRecord(end, 0);
    ccudaEventSynchronize(end);
    ccudaEventElapsedTime(&gputime, start, end);
    fprintf(stdout, "[TIME] Processing time: %f (ms)\n", gputime);
        
    ccudaEventDestroy(start);
    ccudaEventDestroy(end);
    #endif
}

void bga_show_prob_vector_state(struct bga_state *state) {
    #if defined(DEBUG)
    float gputime;
    hipEvent_t start;
    hipEvent_t end;
    
    ccudaEventCreate(&start);
    ccudaEventCreate(&end);

    ccudaEventRecord(start, 0);
    #endif

    fprintf(stdout, "[INFO] === Probability vector status =======================\n");

    float *partial_sum;
    vector_sum_init(&partial_sum);

    fprintf(stdout, "[INFO] Prob. vector sample:");
    
    for (int prob_vector_number = 0; prob_vector_number < state->number_of_prob_vectors; prob_vector_number++) {
        int current_prob_vector_number_of_bits = MAX_PROB_VECTOR_BITS;
        
        if (prob_vector_number + 1 == state->number_of_prob_vectors) {
            if (prob_vector_number != 0) {
                fprintf(stdout, "...");
            }
            
            current_prob_vector_number_of_bits = state->last_prob_vector_bit_count;
            
            int probs_to_show_count = SHOW_PROB_VECTOR_STARTING_BITS;
            if (current_prob_vector_number_of_bits < SHOW_PROB_VECTOR_STARTING_BITS) 
                probs_to_show_count = current_prob_vector_number_of_bits;
            
            float *probs_to_show = (float*)malloc(sizeof(float) * probs_to_show_count);
            ccudaMemcpy(probs_to_show, state->gpu_prob_vectors[prob_vector_number], 
                sizeof(uint32_t) * probs_to_show_count, hipMemcpyDeviceToHost);
            
            for (int i = 0; i < probs_to_show_count; i++) {
                fprintf(stdout, " %.4f", probs_to_show[i]);
            }
        } if (prob_vector_number == 0) {
            int probs_to_show_count = SHOW_PROB_VECTOR_STARTING_BITS;
            if (MAX_PROB_VECTOR_BITS < SHOW_PROB_VECTOR_STARTING_BITS) 
                probs_to_show_count = MAX_PROB_VECTOR_BITS;
            
            float *probs_to_show = (float*)malloc(sizeof(float) * probs_to_show_count);
            ccudaMemcpy(probs_to_show, state->gpu_prob_vectors[prob_vector_number], 
                sizeof(uint32_t) * probs_to_show_count, hipMemcpyDeviceToHost);
            
            for (int i = 0; i < probs_to_show_count; i++) {
                fprintf(stdout, " %.4f", probs_to_show[i]);
            }
        }
        
        kern_vector_sum<<< VECTOR_SUM_BLOCKS, VECTOR_SUM_THREADS >>>( 
            state->gpu_prob_vectors[prob_vector_number], partial_sum,
            current_prob_vector_number_of_bits);
    }
    fprintf(stdout, "\n");

    double accumulated_probability = 0.0;
    accumulated_probability = vector_sum_free(partial_sum);
    fprintf(stdout, "[INFO] Prob. vector accumulated probability: %f\n", accumulated_probability);
    
    #if defined(DEBUG)
    ccudaEventRecord(end, 0);
    ccudaEventSynchronize(end);
    ccudaEventElapsedTime(&gputime, start, end);
    fprintf(stdout, "[TIME] Processing time: %f (ms)\n", gputime);
        
    ccudaEventDestroy(start);
    ccudaEventDestroy(end);
    #endif
}

// Paso 2 del algoritmo.
void bga_model_sampling_mt(struct bga_state *state) {
    //mtgp32_generate_float(&mt_status);
    //mtgp32_print_generated_floats(&mt_status);

    //mtgp32_generate_uint32(&mt_status);
    //mtgp32_print_generated_uint32(&mt_status);
}

// Paso 3 del algoritmo.
void bga_evaluation(struct bga_state *state) {
}

// Paso 4 y 5 del algoritmo.
void bga_model_update(struct bga_state *state) {
}

// Libera la memoria pedida para de estado.
void bga_free(struct bga_state *state) {
    #ifdef INFO
    fprintf(stdout, "[INFO] Freeing memory\n");
    #endif

    for (int vector_number = 0; vector_number < state->number_of_prob_vectors; vector_number++) {
        hipFree(state->gpu_prob_vectors[vector_number]);
    }
    
    free(state->gpu_prob_vectors);
    
    for (int vector_number = 0; vector_number < state->number_of_prob_vectors; vector_number++) {
        hipFree(state->gpu_prob_vectors[vector_number]);
    }

    for (int sample_number = 0; sample_number < state->number_of_samples; sample_number++) {
        for (int vector_number = 0; vector_number < state->number_of_prob_vectors; vector_number++) {
            hipFree(state->gpu_samples[sample_number][vector_number]);
        }
        free(state->gpu_samples[sample_number]);
    }
    free(state->gpu_samples);

    for (int sample_number = 0; sample_number < state->number_of_samples; sample_number++) {
        hipFree(state->gpu_samples_fitness[sample_number]);
    }
    free(state->gpu_samples_fitness);   
}
